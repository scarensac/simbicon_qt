#include "hip/hip_runtime.h"
#include "DFSPH_rendering_cuda.h"

#include "DFSPH_define_cuda.h"
#include "DFSPH_macro_cuda.h"

#include "DFSPH_c_arrays_structure.h"
#include "SPH_other_systems_cuda.h"

#include <iostream>

namespace RenderingCuda
{
	__global__ void init_buffer_kernel(Vector3d* buff, unsigned int size,Vector3d val) {
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		if (i >= size) { return; }

		buff[i] = (i>(size/2))?val+val:val;

	}
}





void cuda_opengl_initParticleRendering(ParticleSetRenderingData& renderingData, unsigned int numParticles,
	Vector3d** pos, Vector3d** vel, bool need_color_buffer, Vector3d** color) {

	//read_last_error_cuda("before alloc rendering on gpu: ");

	glGenVertexArrays(1, &renderingData.vao); // Cr�er le VAO
	glBindVertexArray(renderingData.vao); // Lier le VAO pour l'utiliser


	glGenBuffers(1, &renderingData.pos_buffer);
	// selectionne le buffer pour l'initialiser
	glBindBuffer(GL_ARRAY_BUFFER, renderingData.pos_buffer);
	// dimensionne le buffer actif sur array_buffer, l'alloue et l'initialise avec les positions des sommets de l'objet
	glBufferData(GL_ARRAY_BUFFER,
		/* length */	numParticles * sizeof(Vector3d),
		/* data */      NULL,
		/* usage */     GL_DYNAMIC_DRAW);
	//set it to the attribute
	glEnableVertexAttribArray(0);
	glVertexAttribPointer(0, 3, GL_FORMAT, GL_FALSE, 0, 0);

	glGenBuffers(1, &renderingData.vel_buffer);
	// selectionne le buffer pour l'initialiser
	glBindBuffer(GL_ARRAY_BUFFER, renderingData.vel_buffer);
	// dimensionne le buffer actif sur array_buffer, l'alloue et l'initialise avec les positions des sommets de l'objet
	glBufferData(GL_ARRAY_BUFFER,
		/* length */	numParticles * sizeof(Vector3d),
		/* data */      NULL,
		/* usage */     GL_DYNAMIC_DRAW);
	//set it to the attribute
	glEnableVertexAttribArray(1);
	glVertexAttribPointer(1, 3, GL_FORMAT, GL_FALSE, 0, 0);

	if (need_color_buffer) {
		glGenBuffers(1, &renderingData.color_buffer);
		// selectionne le buffer pour l'initialiser
		glBindBuffer(GL_ARRAY_BUFFER, renderingData.color_buffer);
		// dimensionne le buffer actif sur array_buffer, l'alloue et l'initialise avec les positions des sommets de l'objet
		glBufferData(GL_ARRAY_BUFFER,
			/* length */	(numParticles) * sizeof(Vector3d),
			/* data */      NULL,
			/* usage */     GL_DYNAMIC_DRAW);
		//set it to the attribute
		glEnableVertexAttribArray(2);
		glVertexAttribPointer(2, 3, GL_FORMAT, GL_FALSE, 0, 0);
	}

	// nettoyage
	glBindVertexArray(0);
	glBindBuffer(GL_ARRAY_BUFFER, 0);

	// Registration with CUDA.
	gpuErrchk(hipGraphicsGLRegisterBuffer(&renderingData.pos, renderingData.pos_buffer, hipGraphicsRegisterFlagsNone));
	gpuErrchk(hipGraphicsGLRegisterBuffer(&renderingData.vel, renderingData.vel_buffer, hipGraphicsRegisterFlagsNone));
	if (need_color_buffer) {
		gpuErrchk(hipGraphicsGLRegisterBuffer(&renderingData.color, renderingData.color_buffer, hipGraphicsRegisterFlagsNone));
	}

	//link the pos and vel buffer to cuda
	gpuErrchk(hipGraphicsMapResources(1, &renderingData.pos, 0));
	gpuErrchk(hipGraphicsMapResources(1, &renderingData.vel, 0));
	if (need_color_buffer) {
		gpuErrchk(hipGraphicsMapResources(1, &renderingData.color, 0));
	}

	//set the openglbuffer for direct use in cuda
	Vector3d* vboPtr = NULL;
	size_t size = 0;

	// pos
	gpuErrchk(hipGraphicsResourceGetMappedPointer((void**)&vboPtr, &size, renderingData.pos));//get cuda ptr
	*pos = vboPtr;

	// vel
	gpuErrchk(hipGraphicsResourceGetMappedPointer((void**)&vboPtr, &size, renderingData.vel));//get cuda ptr
	*vel = vboPtr;

	if (need_color_buffer) {
		gpuErrchk(hipGraphicsResourceGetMappedPointer((void**)&vboPtr, &size, renderingData.color));//get cuda ptr
		*color = vboPtr;
	}
}

void cuda_opengl_releaseParticleRendering(ParticleSetRenderingData& renderingData) {
	//unlink the pos and vel buffer from cuda
	gpuErrchk(hipGraphicsUnmapResources(1, &(renderingData.pos), 0));
	gpuErrchk(hipGraphicsUnmapResources(1, &(renderingData.vel), 0));
	if (renderingData.color_buffer <= 100000) {
		gpuErrchk(hipGraphicsUnmapResources(1, &renderingData.color, 0));
	}

	//delete the opengl buffers
	glDeleteBuffers(1, &renderingData.vel_buffer);
	glDeleteBuffers(1, &renderingData.pos_buffer);
	if (renderingData.color_buffer <= 100000) {
		glDeleteBuffers(1, &renderingData.color_buffer);
	}
	glDeleteVertexArrays(1, &renderingData.vao);
}

void cuda_opengl_renderParticleSet(ParticleSetRenderingData& renderingData, unsigned int numParticles) {


	//unlink the pos and vel buffer from cuda
	gpuErrchk(hipGraphicsUnmapResources(1, &(renderingData.pos), 0));
	gpuErrchk(hipGraphicsUnmapResources(1, &(renderingData.vel), 0));
	if (renderingData.color_buffer<=100000) {
		gpuErrchk(hipGraphicsUnmapResources(1, &(renderingData.color), 0));
	}


	//Actual opengl rendering
	// link the vao
	glBindVertexArray(renderingData.vao);

	glBindBuffer(GL_ARRAY_BUFFER, 0);

	//show it
	glDrawArrays(GL_POINTS, 0, numParticles);

	// unlink the vao
	glBindVertexArray(0);

	//link the pos and vel buffer to cuda
	gpuErrchk(hipGraphicsMapResources(1, &renderingData.pos, 0));
	gpuErrchk(hipGraphicsMapResources(1, &renderingData.vel, 0));
	if (renderingData.color_buffer <= 100000) {
		gpuErrchk(hipGraphicsMapResources(1, &renderingData.color, 0));
	}

}

void cuda_renderFluid(SPH::DFSPHCData* data) {
	cuda_opengl_renderParticleSet(*data->fluid_data->renderingData, data->fluid_data[0].numParticles);
}



void cuda_renderBoundaries(SPH::DFSPHCData* data, bool renderWalls) {
	if (renderWalls) {
		cuda_opengl_renderParticleSet(*(data->boundaries_data->renderingData), data->boundaries_data->numParticles);
	}

	for (int i = 0; i < data->numDynamicBodies; ++i) {
		SPH::UnifiedParticleSet& body = data->vector_dynamic_bodies_data[i];
		cuda_opengl_renderParticleSet(*body.renderingData, body.numParticles);
	}
}



void cuda_reset_color(SPH::UnifiedParticleSet* particleSet) {
	if (particleSet->has_color_buffer) {
		int numBlocks = calculateNumBlocks(particleSet->numParticles);
		RenderingCuda::init_buffer_kernel << <numBlocks, BLOCKSIZE >> > (particleSet->color, particleSet->numParticles, Vector3d(-1));
		gpuErrchk(hipDeviceSynchronize());
	}
}