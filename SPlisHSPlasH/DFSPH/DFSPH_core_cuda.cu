#include "hip/hip_runtime.h"
#include "DFSPH_core_cuda.h"


#include <stdio.h>
#include <chrono>
#include <iostream>
#include <thread>
#include <sstream>
#include <fstream>

#include "DFSPH_define_cuda.h"
#include "DFSPH_macro_cuda.h"
#include "DFSPH_static_variables_structure_cuda.h"


#include "hip/hip_runtime.h"
#include ""
#include "DFSPH_c_arrays_structure.h"
#include "cub.cuh"



#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "basic_kernels_cuda.cuh"
#include "SPH_other_systems_cuda.h"

//#include "SPH_memory_storage_precomp_kernels.cuh"



////////////////////////////////////////////////////
/////////       constant memory kernel /////////////
////////////////////////////////////////////////////

#include "SPlisHSPlasH\BasicTypes.h"
#include <string>
#include <vector>

#include "SPlisHSPlasH\Vector.h"
#include "SPlisHSPlasH\Quaternion.h"

#include "DFSPH_define_c.h"
#include "hip/hip_runtime.h"

namespace CoreCuda
{
	__global__ void init_buffer_kernel(Vector3d* buff, unsigned int size, Vector3d val) {
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		if (i >= size) { return; }

		buff[i] = val;
	}
}

//#include "SPH_memory_storage_precomp_kernels.cuh"

#ifdef PRECOMPUTED_KERNELS_USE_CONSTANT_MEMORY

__constant__ RealCuda m_W[PRECOMPUTED_KERNELS_SAMPLE_COUNT];
__constant__ RealCuda m_gradW[PRECOMPUTED_KERNELS_SAMPLE_COUNT];
__constant__ RealCuda m_radius;
__constant__ RealCuda m_radius2;
__constant__ RealCuda m_invStepSize;





__device__  RealCuda get_constant_W_cuda(const SPH::Vector3d &r)
{
	RealCuda res = 0.0;
	const RealCuda r2 = r.squaredNorm();
	if (r2 <= m_radius2)
	{
		const RealCuda r = sqrt(r2);
		const unsigned int pos = (unsigned int)(r * m_invStepSize);
		res = m_W[pos];
	}
	return res;
}

//*
__device__  RealCuda get_constant_W_cuda(const RealCuda r)
{
	RealCuda res = 0.0;
	if (r <= m_radius)
	{
		const unsigned int pos = (unsigned int)(r * m_invStepSize);
		res = m_W[pos];
	}
	return res;
}
__device__  SPH::Vector3d get_constant_grad_W_cuda(const SPH::Vector3d &r)
{
	SPH::Vector3d res;
	const RealCuda r2 = r.squaredNorm();
	if (r2 <= m_radius2)
	{
		const RealCuda rl = sqrt(r2);
		const unsigned int pos = (unsigned int)(rl * m_invStepSize);
		res = m_gradW[pos] * r;
	}
	else
		res.setZero();

	return res;
}
//*/


#include "SPH_memory_storage_precomp_kernels.h"
#include "SPH_other_systems_cuda.h"
#include <iostream>

void writte_to_precomp_kernel(RealCuda* W_i, RealCuda* gradW_i, RealCuda radius, RealCuda radius2, RealCuda invStepSize) {
	hipMemcpyToSymbol(HIP_SYMBOL(m_W), W_i, sizeof(RealCuda) * PRECOMPUTED_KERNELS_SAMPLE_COUNT);
	read_last_error_cuda("test");
	hipMemcpyToSymbol(HIP_SYMBOL(m_gradW), gradW_i, sizeof(RealCuda) * PRECOMPUTED_KERNELS_SAMPLE_COUNT);
	read_last_error_cuda("test");
	hipMemcpyToSymbol(HIP_SYMBOL(m_radius), &radius, sizeof(RealCuda));
	read_last_error_cuda("test");
	hipMemcpyToSymbol(HIP_SYMBOL(m_radius2), &radius2, sizeof(RealCuda));
	read_last_error_cuda("test");
	hipMemcpyToSymbol(HIP_SYMBOL(m_invStepSize), &invStepSize, sizeof(RealCuda));
	read_last_error_cuda("test");
	hipDeviceSynchronize();
	/*
	for (int i = 0; i < PRECOMPUTED_KERNELS_SAMPLE_COUNT; ++i) {
	std::cout << "kernel values: " << W_i[i] << "  " << gradW_i[i] << std::endl;
	}

	test_constant_mem_precomp_kernel_cuda();
	//*/
}
#include "DFSPH_macro_cuda.h"

__global__ void test_constant_mem_precomp_kernel_kernel(RealCuda* W, SPH::Vector3d* gradW, RealCuda* r, RealCuda* r2, RealCuda* invd) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= PRECOMPUTED_KERNELS_SAMPLE_COUNT) { return; }

	const RealCuda posX = 1.0 / m_invStepSize * (RealCuda)i;
	SPH::Vector3d distance = Vector3d(posX, 0.0, 0.0);

	W[i] = KERNEL_W("data", distance);
	gradW[i] = KERNEL_GRAD_W("data", distance);

	if (i == 0) {
		*r = m_radius;
		*r2 = m_radius2;
		*invd = m_invStepSize;
	}
}

void test_constant_mem_precomp_kernel_cuda() {
	RealCuda* W;
	SPH::Vector3d* gradW;
	RealCuda* r;
	RealCuda* r2;
	RealCuda* invd;

	hipMallocManaged(&(W), sizeof(RealCuda) * PRECOMPUTED_KERNELS_SAMPLE_COUNT);
	hipMallocManaged(&(gradW), sizeof(SPH::Vector3d) * PRECOMPUTED_KERNELS_SAMPLE_COUNT);
	hipMallocManaged(&(r), sizeof(RealCuda));
	hipMallocManaged(&(r2), sizeof(RealCuda));
	hipMallocManaged(&(invd), sizeof(RealCuda));

	{//fluid
		int numBlocks = (PRECOMPUTED_KERNELS_SAMPLE_COUNT + BLOCKSIZE - 1) / BLOCKSIZE;
		test_constant_mem_precomp_kernel_kernel << <numBlocks, BLOCKSIZE >> > (W, gradW, r, r2, invd);
	}
	hipDeviceSynchronize();

	for (int i = 0; i < PRECOMPUTED_KERNELS_SAMPLE_COUNT; ++i) {
		std::cout << "kernel values: " << W[i] << "  " << gradW[i].x << std::endl;
	}

	std::cout << "end_values: " << *r << "  " << *r2 << "  " << *invd << std::endl;

	CUDA_FREE_PTR(W);
	CUDA_FREE_PTR(gradW);
	CUDA_FREE_PTR(r);
	CUDA_FREE_PTR(r2);
	CUDA_FREE_PTR(invd);
}
#endif // !BLOCKER//see the macro_cuda_file_for an explanaitions







////////////////////////////////////////////////////
/////////       DIVERGENCE SOLVER      /////////////
////////////////////////////////////////////////////



template <bool ignore_when_no_fluid_near>
__global__ void DFSPH_divergence_warmstart_init_kernel(SPH::DFSPHCData m_data, SPH::UnifiedParticleSet* particleSet) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= particleSet->numParticles) { return; }

	if (ignore_when_no_fluid_near) {
		if (particleSet->getNumberOfNeighbourgs(i) == 0) {
			return;
		}
	}

	particleSet->kappaV[i] = MAX_MACRO_CUDA(particleSet->kappaV[i] * m_data.h_ratio_to_past / 2, -0.5);
	//computeDensityChange(m_data, i);


	//I can actually make the factor and density computation here
	{
#ifndef STORE_PARTICLE_NEIGHBORS
		unsigned int numNeighbors = 0;
	#define computeDensityChange_additional numNeighbors++;
#else
	#define computeDensityChange_additional  
#endif // !STORE_PARTICLE_NEIGHBORS


		//////////////////////////////////////////////////////////////////////////
		// Compute gradient dp_i/dx_j * (1/k)  and dp_j/dx_j * (1/k)
		//////////////////////////////////////////////////////////////////////////
		const Vector3d &xi = particleSet->pos[i];
		const Vector3d &vi = particleSet->vel[i];
		RealCuda sum_grad_p_k = 0;
		Vector3d grad_p_i;
		grad_p_i.setZero();

		RealCuda density =  particleSet->mass[i] * m_data.W_zero;
		RealCuda densityAdv = 0;

		//////////////////////////////////////////////////////////////////////////
		// Fluid
		//////////////////////////////////////////////////////////////////////////
		ITER_NEIGHBORS_INIT(m_data,particleSet, i);

		ITER_NEIGHBORS_FLUID(m_data, particleSet,
			i,
			const Vector3d &xj = body.pos[neighborIndex];
		density += body.mass[neighborIndex] * KERNEL_W(m_data,xi - xj);
		const Vector3d grad_p_j = body.mass[neighborIndex] * KERNEL_GRAD_W(m_data,xi - xj);
		sum_grad_p_k += grad_p_j.squaredNorm();
		grad_p_i += grad_p_j;
		densityAdv += (vi - body.vel[neighborIndex]).dot(grad_p_j);
		computeDensityChange_additional
		);


		//////////////////////////////////////////////////////////////////////////
		// Boundary
		//////////////////////////////////////////////////////////////////////////

#ifdef BENDER2019_BOUNDARIES

		const Vector3d& xj = particleSet->X_rigids[i];
		const RealCuda mass = particleSet->V_rigids[i] * particleSet->density0;
		density += mass * KERNEL_W(m_data, xi - xj);
		const Vector3d grad_p_j = mass * KERNEL_GRAD_W(m_data, xi - xj);
		sum_grad_p_k += grad_p_j.squaredNorm();
		grad_p_i += grad_p_j;
		//No Vj for statics boundaries
		densityAdv += (vi).dot(grad_p_j);


#else

		ITER_NEIGHBORS_BOUNDARIES(m_data, particleSet,
			i,
			const Vector3d &xj = body.pos[neighborIndex];
		density += body.mass[neighborIndex] * KERNEL_W(m_data,xi - xj);
		const Vector3d grad_p_j = body.mass[neighborIndex] * KERNEL_GRAD_W(m_data,xi - xj);
		sum_grad_p_k += grad_p_j.squaredNorm();
		grad_p_i += grad_p_j;
		densityAdv += (vi - body.vel[neighborIndex]).dot(grad_p_j);
		computeDensityChange_additional
		);

#endif

		//////////////////////////////////////////////////////////////////////////
		// Dynamic bodies
		//////////////////////////////////////////////////////////////////////////
		//*
		ITER_NEIGHBORS_SOLIDS(m_data, particleSet,
			i,
			const Vector3d &xj = body.pos[neighborIndex];
		density += body.mass[neighborIndex] * KERNEL_W(m_data,xi - xj);
		const Vector3d grad_p_j = body.mass[neighborIndex] * KERNEL_GRAD_W(m_data,xi - xj);
		sum_grad_p_k += grad_p_j.squaredNorm();
		grad_p_i += grad_p_j;
		densityAdv += (vi - body.vel[neighborIndex]).dot(grad_p_j);
		computeDensityChange_additional
		);
		//*/


		sum_grad_p_k += grad_p_i.squaredNorm();

		//////////////////////////////////////////////////////////////////////////
		// Compute pressure stiffness denominator
		//////////////////////////////////////////////////////////////////////////
		particleSet->factor[i] = (-m_data.invH / (MAX_MACRO_CUDA(sum_grad_p_k, m_eps)));
		particleSet->density[i] = density;

		if (density > 1050) {
			particleSet->color[i].y = 1;
		}

		//end the density adv computation
#ifdef STORE_PARTICLE_NEIGHBORS
		unsigned int numNeighbors = particleSet->getNumberOfNeighbourgs(i);
#endif //STORE_PARTICLE_NEIGHBORS
		// in case of particle deficiency do not perform a divergence solve
		if (numNeighbors < 20) {
			for (unsigned int pid = 1; pid < 3; pid++)
			{
				numNeighbors += particleSet->getNumberOfNeighbourgs(i, pid);
			}
		}
		if (numNeighbors < 20) {
			particleSet->densityAdv[i] = 0;
		}
		else {
			particleSet->densityAdv[i] = MAX_MACRO_CUDA(densityAdv, 0.0);

		}

	}

}

void cuda_divergence_warmstart_init(SPH::DFSPHCData& data) {
	{//fluid
		int numBlocks = calculateNumBlocks(data.fluid_data[0].numParticles);
		DFSPH_divergence_warmstart_init_kernel<false> << <numBlocks, BLOCKSIZE >> > (data, data.fluid_data[0].gpu_ptr);
	}

	//*
	if (data.boundaries_data[0].has_factor_computation) {//boundaries
		int numBlocks = calculateNumBlocks(data.boundaries_data[0].numParticles);
		DFSPH_divergence_warmstart_init_kernel<false> << <numBlocks, BLOCKSIZE >> > (data, data.boundaries_data[0].gpu_ptr);
	}
	//*/


	hipError_t cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cuda_divergence_warmstart_init failed: %d\n", (int)cudaStatus);
		exit(1598);
	}
}


template <bool warm_start> __device__ void divergenceSolveParticle(SPH::DFSPHCData& m_data, SPH::UnifiedParticleSet* particleSet, const unsigned int i) {
	Vector3d v_i = Vector3d(0, 0, 0);
	//////////////////////////////////////////////////////////////////////////
	// Evaluate rhs
	//////////////////////////////////////////////////////////////////////////
	const RealCuda ki = (warm_start) ? particleSet->kappaV[i] : (particleSet->densityAdv[i])*particleSet->factor[i];

#ifdef USE_WARMSTART_V
	if (!warm_start) { particleSet->kappaV[i] += ki; }
#endif

	const Vector3d &xi = particleSet->pos[i];


	//////////////////////////////////////////////////////////////////////////
	// Fluid
	//////////////////////////////////////////////////////////////////////////
	ITER_NEIGHBORS_INIT(m_data, particleSet, i);

	ITER_NEIGHBORS_FLUID(m_data, particleSet,
		i,
		const RealCuda kSum = (ki + ((warm_start) ? body.kappaV[neighborIndex] : (body.densityAdv[neighborIndex])*body.factor[neighborIndex]));
	if (fabs(kSum) > m_eps)
	{
		// ki, kj already contain inverse density
		v_i += kSum *  body.mass[neighborIndex] * KERNEL_GRAD_W(m_data,xi - body.pos[neighborIndex]);
	}
	);

#ifdef USE_BOUNDARIES_DYNAMIC_PROPERTiES
	ITER_NEIGHBORS_BOUNDARIES(m_data, particleSet,
		i,
		const RealCuda kSum = (ki + ((warm_start) ? body.kappaV[neighborIndex] : (body.densityAdv[neighborIndex])*body.factor[neighborIndex]));
	if (fabs(kSum) > m_eps)
	{
		// ki, kj already contain inverse density
		v_i += kSum *  body.mass[neighborIndex] * KERNEL_GRAD_W(m_data,xi - body.pos[neighborIndex]);
	}
	);
#endif


	if (fabs(ki) > m_eps)
	{
		//////////////////////////////////////////////////////////////////////////
		// Boundary
		//////////////////////////////////////////////////////////////////////////
#ifndef USE_BOUNDARIES_DYNAMIC_PROPERTiES

#ifdef BENDER2019_BOUNDARIES
		const Vector3d& xj = particleSet->X_rigids[i];
		const RealCuda mass = particleSet->V_rigids[i] * particleSet->density0;
		const Vector3d delta = ki * mass * KERNEL_GRAD_W(m_data, xi - xj);
		v_i += delta;// ki already contains inverse density
#else
		ITER_NEIGHBORS_BOUNDARIES(m_data, particleSet,
			i,
			const Vector3d delta = ki * body.mass[neighborIndex] * KERNEL_GRAD_W(m_data,xi - body.pos[neighborIndex]);
		v_i += delta;// ki already contains inverse density
		);
#endif

#endif

		//////////////////////////////////////////////////////////////////////////
		// Dynamic bodies
		//////////////////////////////////////////////////////////////////////////

		ITER_NEIGHBORS_SOLIDS(m_data, particleSet,
			i,
			Vector3d delta = ki * body.mass[neighborIndex] * KERNEL_GRAD_W(m_data,xi - body.pos[neighborIndex]);
		v_i += delta;// ki already contains inverse density

					 //we apply the force to the body particle (no invH since it has been fatorized at the end)
		delta *= -particleSet->mass[i];
		atomicAdd(&(body.F[neighborIndex].x), delta.x);
		atomicAdd(&(body.F[neighborIndex].y), delta.y);
		atomicAdd(&(body.F[neighborIndex].z), delta.z);
		);
	}

	particleSet->vel[i] += v_i*m_data.h;
}


//WARNING !!! this is not suposed to be called for the fluid this function is used for boundaries and object for witch 
//doing the velocity variation computation makes no sense but still need the accumulation of kappa for the warm start
__global__ void DFSPH_divergence_accumulate_kappaV_kernel(SPH::UnifiedParticleSet* particleSet) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= particleSet->numParticles) { return; }

	const RealCuda ki = (particleSet->densityAdv[i])*particleSet->factor[i];
	particleSet->kappaV[i] += ki;
}


template<bool warmstart> __global__ void DFSPH_divergence_compute_kernel(SPH::DFSPHCData m_data, SPH::UnifiedParticleSet* particleSet) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= particleSet->numParticles) { return; }

	if (warmstart) {
		if (particleSet->densityAdv[i] > 0.0) {
			divergenceSolveParticle<warmstart>(m_data, particleSet, i);
		}
	}
	else {
		divergenceSolveParticle<warmstart>(m_data, particleSet, i);
	}

}

template<bool warmstart> void cuda_divergence_compute(SPH::DFSPHCData& data) {
	{//fluid
		int numBlocks = calculateNumBlocks(data.fluid_data[0].numParticles);
		DFSPH_divergence_compute_kernel<warmstart> << <numBlocks, BLOCKSIZE >> > (data, data.fluid_data[0].gpu_ptr);
	}

	//*

	if (data.boundaries_data[0].has_factor_computation) {//boundaries 
		if (!warmstart) {
			int numBlocks = calculateNumBlocks(data.boundaries_data[0].numParticles);
			DFSPH_divergence_accumulate_kappaV_kernel << <numBlocks, BLOCKSIZE >> > (data.boundaries_data[0].gpu_ptr);
		}
	}
	//*/

	hipError_t cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cuda_divergence_compute failed: %d\n", (int)cudaStatus);
		exit(1598);
	}
}
template void cuda_divergence_compute<true>(SPH::DFSPHCData& data);
template void cuda_divergence_compute<false>(SPH::DFSPHCData& data);



__device__ void computeDensityChange(const SPH::DFSPHCData& m_data, SPH::UnifiedParticleSet* particleSet, const unsigned int index) {
#ifdef STORE_PARTICLE_NEIGHBORS
#define computeDensityChange_additional  
	unsigned int numNeighbors = particleSet->getNumberOfNeighbourgs(index);
	// in case of particle deficiency do not perform a divergence solve
	if (numNeighbors < 20) {
		for (unsigned int pid = 1; pid < 3; pid++)
		{
			numNeighbors += particleSet->getNumberOfNeighbourgs(index, pid);
		}
	}
	if (numNeighbors < 20) {
		particleSet->densityAdv[index] = 0;
	}
	else 
#endif //STORE_PARTICLE_NEIGHBORS
	{
#ifndef STORE_PARTICLE_NEIGHBORS
		unsigned int numNeighbors = 0;
#define computeDensityChange_additional numNeighbors++;
#endif //STORE_PARTICLE_NEIGHBORS

		RealCuda densityAdv = 0;
		const Vector3d &xi = particleSet->pos[index];
		const Vector3d &vi = particleSet->vel[index];
		//////////////////////////////////////////////////////////////////////////
		// Fluid
		//////////////////////////////////////////////////////////////////////////
		ITER_NEIGHBORS_INIT(m_data, particleSet, index);

		ITER_NEIGHBORS_FLUID(m_data, particleSet,
			index,
			densityAdv += body.mass[neighborIndex] * (vi - body.vel[neighborIndex]).dot(KERNEL_GRAD_W(m_data,xi - body.pos[neighborIndex]));
			computeDensityChange_additional
		);
		//////////////////////////////////////////////////////////////////////////
		// Boundary
		//////////////////////////////////////////////////////////////////////////
#ifdef BENDER2019_BOUNDARIES
		const Vector3d& xj = particleSet->X_rigids[index];
		const RealCuda mass = particleSet->V_rigids[index] * particleSet->density0;
		densityAdv += mass* (vi).dot(KERNEL_GRAD_W(m_data, xi - xj));
#else
		ITER_NEIGHBORS_BOUNDARIES(m_data, particleSet,
			index,
			densityAdv += body.mass[neighborIndex] * (vi - body.vel[neighborIndex]).dot(KERNEL_GRAD_W(m_data,xi - body.pos[neighborIndex]));
			computeDensityChange_additional
		);
#endif

		//////////////////////////////////////////////////////////////////////////
		// Dynamic Bodies
		//////////////////////////////////////////////////////////////////////////
		ITER_NEIGHBORS_SOLIDS(m_data, particleSet,
			index,
			densityAdv += body.mass[neighborIndex] * (vi - body.vel[neighborIndex]).dot(KERNEL_GRAD_W(m_data,xi - body.pos[neighborIndex]));
			computeDensityChange_additional
		);

#ifndef STORE_PARTICLE_NEIGHBORS
		if (numNeighbors < 20) {
			densityAdv = 0;
		}
#endif //STORE_PARTICLE_NEIGHBORS

		// only correct positive divergence
		particleSet->densityAdv[index] = MAX_MACRO_CUDA(densityAdv, 0.0);
	}
}



__global__ void DFSPH_divergence_init_kernel(SPH::DFSPHCData m_data, SPH::UnifiedParticleSet* particleSet) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= particleSet->numParticles) { return; }

	{
#ifdef USE_WARMSTART_V
		particleSet->kappaV[i] = 0;
#endif

		///TODO when doing this kernel I can actually fuse the code for all those computation to limit the number
		///of time I read the particles positions
		computeDensityChange(m_data, particleSet, i);

#ifndef USE_WARMSTART_V
		//I can actually make the factor and desity computation here
		{
			//////////////////////////////////////////////////////////////////////////
			// Compute gradient dp_i/dx_j * (1/k)  and dp_j/dx_j * (1/k)
			//////////////////////////////////////////////////////////////////////////
			const Vector3d &xi = particleSet->pos[i];
			RealCuda sum_grad_p_k = 0;
			Vector3d grad_p_i;
			grad_p_i.setZero();

			RealCuda density = particleSet->mass[i] * m_data.W_zero;

			//////////////////////////////////////////////////////////////////////////
			// Fluid
			//////////////////////////////////////////////////////////////////////////
			ITER_NEIGHBORS_INIT(m_data, particleSet, i);

			ITER_NEIGHBORS_FLUID(m_data, particleSet,
				i,
				const Vector3d &xj = body.pos[neighborIndex];
			density += body.mass[neighborIndex] * KERNEL_W(m_data,xi - xj);
			const Vector3d grad_p_j = body.mass[neighborIndex] * KERNEL_GRAD_W(m_data,xi - xj);
			sum_grad_p_k += grad_p_j.squaredNorm();
			grad_p_i += grad_p_j;
			);

			//////////////////////////////////////////////////////////////////////////
			// Boundary
			//////////////////////////////////////////////////////////////////////////
			ITER_NEIGHBORS_BOUNDARIES(m_data, particleSet,
				i,
				const Vector3d &xj = body.pos[neighborIndex];
			density += body.mass[neighborIndex] * KERNEL_W(m_data,xi - xj);
			const Vector3d grad_p_j = body.mass[neighborIndex] * KERNEL_GRAD_W(m_data,xi - xj);
			sum_grad_p_k += grad_p_j.squaredNorm();
			grad_p_i += grad_p_j;
			);

			//////////////////////////////////////////////////////////////////////////
			// Dynamic bodies
			//////////////////////////////////////////////////////////////////////////
			//*
			ITER_NEIGHBORS_SOLIDS(m_data, particleSet,
				i,
				const Vector3d &xj = body.pos[neighborIndex];
			density += body.mass[neighborIndex] * KERNEL_W(m_data,xi - xj);
			const Vector3d grad_p_j = body.mass[neighborIndex] * KERNEL_GRAD_W(m_data,xi - xj);
			sum_grad_p_k += grad_p_j.squaredNorm();
			grad_p_i += grad_p_j;
			);
			//*/


			sum_grad_p_k += grad_p_i.squaredNorm();

			//////////////////////////////////////////////////////////////////////////
			// Compute pressure stiffness denominator
			//////////////////////////////////////////////////////////////////////////
			particleSet->factor[i] = (-m_data.invH / (MAX_MACRO_CUDA(sum_grad_p_k, m_eps)));
			particleSet->density[i] = density;

		}
#endif


	}

}

void cuda_divergence_init(SPH::DFSPHCData& data) {
	{//fluid
		int numBlocks = calculateNumBlocks(data.fluid_data[0].numParticles);
		DFSPH_divergence_init_kernel << <numBlocks, BLOCKSIZE >> > (data, data.fluid_data[0].gpu_ptr);
	}

	//*
	if (data.boundaries_data[0].has_factor_computation) {//boundaries 
		int numBlocks = calculateNumBlocks(data.boundaries_data[0].numParticles);
		DFSPH_divergence_init_kernel << <numBlocks, BLOCKSIZE >> > (data, data.boundaries_data[0].gpu_ptr);
	}
	//*/

	hipError_t cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cuda_divergence_init failed: %d\n", (int)cudaStatus);
		exit(1598);
	}
}


__global__ void DFSPH_divergence_loop_end_kernel(SPH::DFSPHCData m_data, SPH::UnifiedParticleSet* particleSet, RealCuda* avg_density_err) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= particleSet->numParticles) { return; }

	computeDensityChange(m_data, particleSet, i);
	//atomicAdd(avg_density_err, m_data.densityAdv[i]);
}

RealCuda cuda_divergence_loop_end(SPH::DFSPHCData& data) {
	RealCuda* avg_density_err = SVS_CU::get()->avg_density_err;

	{//fluid
		int numBlocks = calculateNumBlocks(data.fluid_data[0].numParticles );
		DFSPH_divergence_loop_end_kernel << <numBlocks, BLOCKSIZE >> > (data, data.fluid_data[0].gpu_ptr, avg_density_err);
	}

	//*
	if (data.boundaries_data[0].has_factor_computation) {//boundaries 
		int numBlocks = calculateNumBlocks(data.boundaries_data[0].numParticles );
		DFSPH_divergence_loop_end_kernel << <numBlocks, BLOCKSIZE >> > (data, data.boundaries_data[0].gpu_ptr, avg_density_err);
	}
	//*/


	hipError_t cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cuda_divergence_loop_end failed: %d\n", (int)cudaStatus);
		exit(1598);
	}


	// Run sum-reduction
	hipcub::DeviceReduce::Sum(data.fluid_data->d_temp_storage, data.fluid_data->temp_storage_bytes, data.fluid_data->densityAdv, avg_density_err, data.fluid_data[0].numParticles);
	gpuErrchk(hipDeviceSynchronize());


	RealCuda result = 0;
	gpuErrchk(hipMemcpy(&result, avg_density_err, sizeof(RealCuda), hipMemcpyDeviceToHost));

	return result;
}


int cuda_divergenceSolve(SPH::DFSPHCData& m_data, const unsigned int maxIter, const RealCuda maxError) {
	//////////////////////////////////////////////////////////////////////////
	// Init parameters
	//////////////////////////////////////////////////////////////////////////

	const RealCuda h = m_data.h;
	const int numParticles = m_data.fluid_data[0].numParticles;
	const RealCuda density0 = m_data.density0;

	std::chrono::steady_clock::time_point start = std::chrono::steady_clock::now();

#ifdef USE_WARMSTART_V
	cuda_divergence_warmstart_init(m_data);

	std::chrono::steady_clock::time_point m0 = std::chrono::steady_clock::now();
	cuda_divergence_compute<true>(m_data);
#endif

	std::chrono::steady_clock::time_point m1 = std::chrono::steady_clock::now();
	//////////////////////////////////////////////////////////////////////////
	// Compute velocity of density change
	//////////////////////////////////////////////////////////////////////////
	cuda_divergence_init(m_data);

	std::chrono::steady_clock::time_point m2 = std::chrono::steady_clock::now();

	unsigned int m_iterationsV = 0;

	//////////////////////////////////////////////////////////////////////////
	// Start solver
	//////////////////////////////////////////////////////////////////////////

    /*
    double avg_density=0;
    double avg_mass=0;
    for (int i=0;i<numParticles;++i){
        avg_mass+= m_data.fluid_data->mass[i];
        avg_density+= m_data.fluid_data->density[i];
        printf("avg density %f  %f\n", m_data.fluid_data->density[i], m_data.fluid_data->mass[i]);
    }
    avg_density/=numParticles;
    avg_mass/=numParticles;
    printf("avg density %f  %f\n", avg_density, avg_mass);
    //*/



	// Maximal allowed density fluctuation
	// use maximal density error divided by time step size
	const RealCuda eta = maxError * 0.01 * density0 / h;  // maxError is given in percent

	float time_3_1 = 0;
	float time_3_2 = 0;
	RealCuda avg_density_err = 0.0;
	while (((avg_density_err > eta) || (m_iterationsV < 3)) && (m_iterationsV < maxIter))
	{

		//////////////////////////////////////////////////////////////////////////
		// Perform Jacobi iteration over all blocks
		//////////////////////////////////////////////////////////////////////////
		std::chrono::steady_clock::time_point p0 = std::chrono::steady_clock::now();
		cuda_divergence_compute<false>(m_data);
		std::chrono::steady_clock::time_point p1 = std::chrono::steady_clock::now();

		avg_density_err = cuda_divergence_loop_end(m_data);
		std::chrono::steady_clock::time_point p2 = std::chrono::steady_clock::now();

		avg_density_err /= numParticles;
		m_iterationsV++;

		time_3_1 += std::chrono::duration_cast<std::chrono::nanoseconds> (p1 - p0).count() / 1000000.0f;
		time_3_2 += std::chrono::duration_cast<std::chrono::nanoseconds> (p2 - p1).count() / 1000000.0f;
	}




	/*
	std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();

	float time_0 = std::chrono::duration_cast<std::chrono::nanoseconds> (m0 - start).count() / 1000000.0f;
	float time_1 = std::chrono::duration_cast<std::chrono::nanoseconds> (m1 - m0).count() / 1000000.0f;
	float time_2 = std::chrono::duration_cast<std::chrono::nanoseconds> (m2 - m1).count() / 1000000.0f;
	float time_3 = std::chrono::duration_cast<std::chrono::nanoseconds> (end - m2).count() / 1000000.0f;

	std::cout << "detail pressure solve (iter total (varible_comp warm_comp init actual_comp (t1 t2))): " << m_iterationsV << "  " << time_0+ time_1 + time_2 + time_3 <<
	"  (" << time_0 << "  " << time_1 << "  " << time_2 << "  " << time_3 << "(" << time_3_1 << " " << time_3_2 << ") )" << std::endl;

	//*/
	return m_iterationsV;
}

////////////////////////////////////////////////////
/////////          DENSITY SOLVER      /////////////
////////////////////////////////////////////////////

template <bool warm_start> __device__ void pressureSolveParticle(SPH::DFSPHCData& m_data, SPH::UnifiedParticleSet* particleSet, const unsigned int i) {
	//////////////////////////////////////////////////////////////////////////
	// Evaluate rhs
	//////////////////////////////////////////////////////////////////////////
	const RealCuda ki = (warm_start) ? particleSet->kappa[i] : (particleSet->densityAdv[i])*particleSet->factor[i];

#ifdef USE_WARMSTART
	//if (!warm_start) { particleSet->kappa[i] += ki; } //moved to the evaluation
#endif


	Vector3d v_i = Vector3d(0, 0, 0);
	const Vector3d &xi = particleSet->pos[i];

	//////////////////////////////////////////////////////////////////////////
	// Fluid
	//////////////////////////////////////////////////////////////////////////
	ITER_NEIGHBORS_INIT(m_data, particleSet, i);

	ITER_NEIGHBORS_FLUID(m_data, particleSet,
		i,
		const RealCuda kSum = (ki + ((warm_start) ? body.kappa[neighborIndex] : (body.densityAdv[neighborIndex])*body.factor[neighborIndex]));
	if (fabs(kSum) > m_eps)
	{
		// ki, kj already contain inverse density
		v_i += kSum * body.mass[neighborIndex] * KERNEL_GRAD_W(m_data,xi - body.pos[neighborIndex]);
	}
	);

#ifdef USE_BOUNDARIES_DYNAMIC_PROPERTiES
	ITER_NEIGHBORS_BOUNDARIES(m_data, particleSet,
		i,
		const RealCuda kSum = (ki + ((warm_start) ? body.kappa[neighborIndex] : (body.densityAdv[neighborIndex])*body.factor[neighborIndex]));
	if (fabs(kSum) > m_eps)
	{
		// ki, kj already contain inverse density
		v_i += kSum * body.mass[neighborIndex] * KERNEL_GRAD_W(m_data,xi - body.pos[neighborIndex]);
	}
	);
#endif

	if (fabs(ki) > m_eps)
	{
		//////////////////////////////////////////////////////////////////////////
		// Boundary
		//////////////////////////////////////////////////////////////////////////

#ifndef USE_BOUNDARIES_DYNAMIC_PROPERTiES

#ifdef BENDER2019_BOUNDARIES
		const Vector3d& xj = particleSet->X_rigids[i];
		const RealCuda mass = particleSet->V_rigids[i] * particleSet->density0;

		v_i += ki * mass * KERNEL_GRAD_W(m_data, xi - xj);

#else
		ITER_NEIGHBORS_BOUNDARIES(m_data, particleSet,
			i,
			v_i += ki * body.mass[neighborIndex] * KERNEL_GRAD_W(m_data,xi - body.pos[neighborIndex]);
		);
#endif


#endif


		//////////////////////////////////////////////////////////////////////////
		// Dynamic bodies
		//////////////////////////////////////////////////////////////////////////
		ITER_NEIGHBORS_SOLIDS(m_data, particleSet,
			i,
			Vector3d delta = ki * body.mass[neighborIndex] * KERNEL_GRAD_W(m_data,xi - body.pos[neighborIndex]);
		v_i += delta;// ki already contains inverse density

					 //we apply the force to the body particle (no invH since it has been fatorized at the end)
		delta *= -particleSet->mass[i];
		atomicAdd(&(body.F[neighborIndex].x), delta.x);
		atomicAdd(&(body.F[neighborIndex].y), delta.y);
		atomicAdd(&(body.F[neighborIndex].z), delta.z);
		);
	}

	// Directly update velocities instead of storing pressure accelerations
	particleSet->vel[i] += v_i*m_data.h_future;
}



//WARNING !!! this is not suposed to be called for the fluid this function is used for boundaries and object for witch 
//doing the velocity variation computation makes no sense but still need the accumulation of kappa for the warm start
__global__ void DFSPH_density_accumulate_kappa_kernel(SPH::UnifiedParticleSet* particleSet) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= particleSet->numParticles) { return; }

	const RealCuda ki = (particleSet->densityAdv[i])*particleSet->factor[i];
	particleSet->kappa[i] += ki;
}

template<bool warmstart> __global__ void DFSPH_pressure_compute_kernel(SPH::DFSPHCData m_data, SPH::UnifiedParticleSet* particleSet) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= particleSet->numParticles) { return; }

	pressureSolveParticle<warmstart>(m_data, particleSet, i);

}

template<bool warmstart> void cuda_pressure_compute(SPH::DFSPHCData& data) {
	{//fluid
		int numBlocks = calculateNumBlocks(data.fluid_data[0].numParticles);
		DFSPH_pressure_compute_kernel<warmstart> << <numBlocks, BLOCKSIZE >> > (data, data.fluid_data[0].gpu_ptr);
	}

	if (data.boundaries_data[0].has_factor_computation) {//boundaries 
		if (!warmstart) {
			int numBlocks = calculateNumBlocks(data.boundaries_data[0].numParticles);
			DFSPH_density_accumulate_kappa_kernel << <numBlocks, BLOCKSIZE >> > (data.boundaries_data[0].gpu_ptr);
		}
	}

	hipError_t cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cuda_pressure_compute failed: %d\n", (int)cudaStatus);
		exit(1598);
	}
}
template void cuda_pressure_compute<true>(SPH::DFSPHCData& data);
template void cuda_pressure_compute<false>(SPH::DFSPHCData& data);


__device__ void computeDensityAdv(SPH::DFSPHCData& m_data, SPH::UnifiedParticleSet* particleSet, const unsigned int index) {
	const Vector3d xi = particleSet->pos[index];
	const Vector3d vi = particleSet->vel[index];
	RealCuda delta = 0;


	//////////////////////////////////////////////////////////////////////////
	// Fluid
	//////////////////////////////////////////////////////////////////////////
	ITER_NEIGHBORS_INIT(m_data, particleSet, index);

	ITER_NEIGHBORS_FLUID(m_data, particleSet,
		index,
		delta += body.mass[neighborIndex] * (vi - body.vel[neighborIndex]).dot(KERNEL_GRAD_W(m_data,xi - body.pos[neighborIndex]));
	);

	//////////////////////////////////////////////////////////////////////////
	// Boundary
	//////////////////////////////////////////////////////////////////////////

#ifdef BENDER2019_BOUNDARIES
	const Vector3d& xj = particleSet->X_rigids[index];
	const RealCuda mass = particleSet->V_rigids[index] * particleSet->density0;

	delta += mass * (vi - xj).dot(KERNEL_GRAD_W(m_data, xi - xj));

#else
    //*
	ITER_NEIGHBORS_BOUNDARIES(m_data, particleSet,
		index,
		delta += body.mass[neighborIndex] * (vi - body.vel[neighborIndex]).dot(KERNEL_GRAD_W(m_data,xi - body.pos[neighborIndex]));
    );//*/
#endif

	//////////////////////////////////////////////////////////////////////////
	// Dynamic bodies
    //////////////////////////////////////////////////////////////////////////
    /// \brief ITER_NEIGHBORS_SOLIDS

    //*
	ITER_NEIGHBORS_SOLIDS(m_data, particleSet,
		index,
		delta += body.mass[neighborIndex] * (vi - body.vel[neighborIndex]).dot(KERNEL_GRAD_W(m_data,xi - body.pos[neighborIndex]));
	)
//*/
		particleSet->densityAdv[index] = MAX_MACRO_CUDA(particleSet->density[index] + m_data.h_future*delta - m_data.density0, 0.0);


#ifdef USE_WARMSTART
	particleSet->kappa[index] += (particleSet->densityAdv[index])*particleSet->factor[index];

#endif
}


__global__ void DFSPH_pressure_init_kernel(SPH::DFSPHCData m_data, SPH::UnifiedParticleSet* particleSet) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= particleSet->numParticles) { return; }

#ifdef USE_WARMSTART
	particleSet->kappa[i] = 0;
#endif

	particleSet->factor[i] *= m_data.invH_future;

	computeDensityAdv(m_data, particleSet, i);


}

void cuda_pressure_init(SPH::DFSPHCData& data) {
	{//fluid
		int numBlocks = calculateNumBlocks(data.fluid_data[0].numParticles);
		DFSPH_pressure_init_kernel << <numBlocks, BLOCKSIZE >> > (data, data.fluid_data[0].gpu_ptr);
	}

	if (data.boundaries_data[0].has_factor_computation) {//boundaries 
		int numBlocks = calculateNumBlocks(data.boundaries_data[0].numParticles);
		DFSPH_pressure_init_kernel << <numBlocks, BLOCKSIZE >> > (data, data.boundaries_data[0].gpu_ptr);
	}


	hipError_t cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cuda_pressure_init failed: %d\n", (int)cudaStatus);
		exit(1598);
	}
}

__global__ void DFSPH_pressure_loop_end_kernel(SPH::DFSPHCData m_data, SPH::UnifiedParticleSet* particleSet, RealCuda* avg_density_err) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= particleSet->numParticles) { return; }


	computeDensityAdv(m_data, particleSet, i);
	//atomicAdd(avg_density_err, m_data.densityAdv[i]);
}

RealCuda cuda_pressure_loop_end(SPH::DFSPHCData& data) {

	std::chrono::steady_clock::time_point p0 = std::chrono::steady_clock::now();

	RealCuda* avg_density_err = SVS_CU::get()->avg_density_err;

	{//fluid
		int numBlocks = calculateNumBlocks(data.fluid_data[0].numParticles);
		DFSPH_pressure_loop_end_kernel << <numBlocks, BLOCKSIZE >> > (data, data.fluid_data[0].gpu_ptr, avg_density_err);
	}

	if (data.boundaries_data[0].has_factor_computation) {//boundaries 
		int numBlocks = calculateNumBlocks(data.boundaries_data[0].numParticles);
		DFSPH_pressure_loop_end_kernel << <numBlocks, BLOCKSIZE >> > (data, data.boundaries_data[0].gpu_ptr, avg_density_err);
	}

	/*
	///LOL the detailed implementation is slower so no need to even think about developping data
	DFSPH_pressure_loop_end_kernel << <numBlocks, BLOCKSIZE >> > (data.numFluidParticles, data.posFluid, data.velFluid,
	data.neighbourgs, data.numberOfNeighbourgs,
	data.mass, data.m_kernel_precomp, data.boundaryPsi, data.posBoundary, data.velBoundary,
	data.vector_dynamic_bodies_data_cuda, data.densityAdv, data.density, data.h_future, data.density0);
	//*/

	hipError_t cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cuda_pressure_loop_end failed: %d\n", (int)cudaStatus);
		exit(1598);
	}

    std::chrono::steady_clock::time_point p1 = std::chrono::steady_clock::now();


	// Run sum-reduction
	hipcub::DeviceReduce::Sum(data.fluid_data->d_temp_storage, data.fluid_data->temp_storage_bytes, data.fluid_data->densityAdv, avg_density_err, data.fluid_data[0].numParticles);


	RealCuda result = 0;
	gpuErrchk(hipMemcpy(&result, avg_density_err, sizeof(RealCuda), hipMemcpyDeviceToHost));


	std::chrono::steady_clock::time_point p2 = std::chrono::steady_clock::now();
	float time1 = std::chrono::duration_cast<std::chrono::nanoseconds> (p1 - p0).count() / 1000000.0f;
	float time2 = std::chrono::duration_cast<std::chrono::nanoseconds> (p2 - p1).count() / 1000000.0f;

	//std::cout << "pressure loop end details: " << time1 << "  " << time2 << std::endl;

	return result;
}


int cuda_pressureSolve(SPH::DFSPHCData& m_data, const unsigned int m_maxIterations, const RealCuda m_maxError) {
	const RealCuda density0 = m_data.density0;
	const int numParticles = (int)m_data.fluid_data[0].numParticles;
	RealCuda avg_density_err = 0.0;


	std::chrono::steady_clock::time_point start = std::chrono::steady_clock::now();




#ifdef USE_WARMSTART		
	cuda_pressure_compute<true>(m_data);
#endif

	std::chrono::steady_clock::time_point m1 = std::chrono::steady_clock::now();

	//////////////////////////////////////////////////////////////////////////
	// Compute rho_adv
	//////////////////////////////////////////////////////////////////////////
	cuda_pressure_init(m_data);


	std::chrono::steady_clock::time_point m2 = std::chrono::steady_clock::now();


	unsigned int m_iterations = 0;

	//////////////////////////////////////////////////////////////////////////
	// Start solver
	//////////////////////////////////////////////////////////////////////////

	// Maximal allowed density fluctuation
	const RealCuda eta = m_maxError * 0.01 * density0;  // maxError is given in percent

	float time_3_1 = 0;
	float time_3_2 = 0;
	while (((avg_density_err > eta) || (m_iterations < 2)) && (m_iterations < m_maxIterations))
    {
		std::chrono::steady_clock::time_point p0 = std::chrono::steady_clock::now();
		cuda_pressure_compute<false>(m_data);
		std::chrono::steady_clock::time_point p1 = std::chrono::steady_clock::now();
		avg_density_err = cuda_pressure_loop_end(m_data);
		std::chrono::steady_clock::time_point p2 = std::chrono::steady_clock::now();
		avg_density_err /= numParticles;

		m_iterations++;

		time_3_1 += std::chrono::duration_cast<std::chrono::nanoseconds> (p1 - p0).count() / 1000000.0f;
		time_3_2 += std::chrono::duration_cast<std::chrono::nanoseconds> (p2 - p1).count() / 1000000.0f;


	}
	/*
	std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();

	float time_1 = std::chrono::duration_cast<std::chrono::nanoseconds> (m1 - start).count() / 1000000.0f;
	float time_2 = std::chrono::duration_cast<std::chrono::nanoseconds> (m2 - m1).count() / 1000000.0f;
	float time_3 = std::chrono::duration_cast<std::chrono::nanoseconds> (end - m2).count() / 1000000.0f;

	std::cout << "detail pressure solve (iter total (warm init actual_comp (t1 t2))): " <<m_iterations <<"  "<< time_1 + time_2 +time_3 <<
	"  (" << time_1 << "  " << time_2<< "  "<< time_3 <<"("<< time_3_1<<" "<< time_3_2<<") )" << std::endl;

	//*/

	return m_iterations;

}


// also prepare the normals for the adhesion force
__global__ void DFSPH_viscosityXSPH_kernel(SPH::DFSPHCData m_data, SPH::UnifiedParticleSet* particleSet) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= particleSet->numParticles) { return; }

	//I set the gravitation directly here to lover the number of kernels
	Vector3d ai = Vector3d(0, 0, 0);
	Vector3d ni = Vector3d(0, 0, 0);
	const Vector3d &xi = particleSet->pos[i];
	const Vector3d &vi = particleSet->vel[i];

	//////////////////////////////////////////////////////////////////////////
	// Fluid
	//////////////////////////////////////////////////////////////////////////
//*	
	ITER_NEIGHBORS_INIT(m_data, particleSet, i);

    //*
	//*
	ITER_NEIGHBORS_FLUID(m_data, particleSet,
		i,
		Vector3d xixj = xi - body.pos[neighborIndex];
	RealCuda mass_div_density = body.mass[neighborIndex] / body.density[neighborIndex];
	ai -= m_data.invH * m_data.viscosity * (mass_div_density) * (vi - body.vel[neighborIndex]) * KERNEL_W(m_data,xixj);
	ni += mass_div_density * KERNEL_GRAD_W(m_data,xixj);
	)
    //*/
		//*/
		/*
		//viscosity only
		ITER_NEIGHBORS_FLUID(
		i,
		ai -= m_data.invH * m_data.viscosity * (body.mass[neighborIndex] / body.density[neighborIndex]) *
		(vi - body.vel[neighborIndex]) * KERNEL_W(m_data,xi - body.pos[neighborIndex]);

		)//*/

        particleSet->acc[i] = m_data.gravitation + ai;

    ///TODO WARNING THERE IS A PROLEM WIHT THAT, it make another buffer crahs at some poin in the simulation
    /// meaning this might be writting in a random position inthe simulation
    /*
	//I'm gona use the vector3D used for the agglomerated neigbor search to store the normals
	ni *= m_data.getKernelRadius();
	m_data.posBufferGroupedDynamicBodies[i] = ni;
	//*/
}


__global__ void DFSPH_applySurfaceAkinci2013SurfaceTension_kernel(SPH::DFSPHCData m_data, SPH::UnifiedParticleSet* particleSet) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= particleSet->numParticles) { return; }

	//for more lisability of the code
	Vector3d* normals = m_data.posBufferGroupedDynamicBodies;
	RealCuda supportRadius = m_data.getKernelRadius();
	RealCuda k = m_data.getSurfaceTension();
	RealCuda density0 = m_data.density0;

	//I set the gravitation directly here to lover the number of kernels
	Vector3d ai = Vector3d(0, 0, 0);
	Vector3d ni = normals[i];
	RealCuda rhoi = particleSet->density[i];
	const Vector3d &xi = particleSet->pos[i];

	ITER_NEIGHBORS_INIT(m_data, particleSet, i);

	//////////////////////////////////////////////////////////////////////////
	// Fluid
	//////////////////////////////////////////////////////////////////////////

	ITER_NEIGHBORS_FLUID(m_data, particleSet,
		i,
		RealCuda K_ij = 2.0*density0 / (rhoi + body.density[neighborIndex]);

	Vector3d accel = Vector3d(0, 0, 0);


	// Cohesion force
	Vector3d xixj = xi - body.pos[neighborIndex];
	const Real length2 = xixj.squaredNorm();
	if (length2 > 1.0e-9)
	{
		xixj = ((Real) 1.0 / sqrt(length2)) * xixj;
		accel -= k * body.mass[neighborIndex] * xixj * m_data.WCohesion(xixj);
	}

	// Curvature
	accel -= k * supportRadius* (ni - normals[neighborIndex]);

	ai += K_ij * accel;
	//*/
	);
	//////////////////////////////////////////////////////////////////////////
	// Boundary
	//////////////////////////////////////////////////////////////////////////

#ifdef BENDER2019_BOUNDARIES

	const Vector3d& xj = particleSet->X_rigids[i];
	const RealCuda mass = particleSet->V_rigids[i] * particleSet->density0;
	
	Vector3d xixj = (xi - xj);
	const Real length2 = xixj.squaredNorm();
	if (length2 > 1.0e-9)
	{
		xixj = ((Real)1.0 / sqrt(length2)) * xixj;
		ai -= k * mass * xixj * m_data.WAdhesion(xixj);
	}

#else

	ITER_NEIGHBORS_BOUNDARIES(m_data, particleSet,
		i,
		// adhesion force
		Vector3d xixj = (xi - body.pos[neighborIndex]);
	const Real length2 = xixj.squaredNorm();
	if (length2 > 1.0e-9)
	{
		xixj = ((Real) 1.0 / sqrt(length2)) * xixj;
		ai -= k * body.mass[neighborIndex] * xixj * m_data.WAdhesion(xixj);
	}
	);


#endif


	//////////////////////////////////////////////////////////////////////////
	// Dynamic Bodies
	//////////////////////////////////////////////////////////////////////////
	ITER_NEIGHBORS_SOLIDS(m_data, particleSet,
		i,
		// adhesion force
		Vector3d xixj = (xi - body.pos[neighborIndex]);
	const Real length2 = xixj.squaredNorm();
	if (length2 > 1.0e-9)
	{
		xixj = ((Real) 1.0 / sqrt(length2)) * xixj;
		ai -= k * body.mass[neighborIndex] * xixj * m_data.WAdhesion(xixj);
	}
	);

	particleSet->acc[i] += ai;
}


void cuda_externalForces(SPH::DFSPHCData& data) {
	int numBlocks = calculateNumBlocks(data.fluid_data[0].numParticles);
    DFSPH_viscosityXSPH_kernel << <numBlocks, BLOCKSIZE >> > (data, data.fluid_data[0].gpu_ptr);

	hipError_t cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cuda_viscosityXSPH failed: %d\n", (int)cudaStatus);
		exit(1598);
	}


	//end the computations for the surface tension

    //DFSPH_applySurfaceAkinci2013SurfaceTension_kernel << <numBlocks, BLOCKSIZE >> > (data, data.fluid_data[0].gpu_ptr);
	gpuErrchk(hipDeviceSynchronize());




}




////////////////////////////////////////////////////
/////////         NEIGHBORS SEARCH     /////////////
////////////////////////////////////////////////////


__global__ void DFSPH_fill_aggregated_pos_buffer_kernel(SPH::DFSPHCData data, unsigned int num_particles) {

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= num_particles) { return; }

	if (data.is_fluid_aggregated) {
		if (i<data.fluid_data_cuda->numParticles) {

			//writte de pos
			data.posBufferGroupedDynamicBodies[i] = data.fluid_data_cuda->pos[i];

			return;
		}
	}

	//find the current dynamic body
	int count_particles_previous_bodies = (data.is_fluid_aggregated) ? data.fluid_data_cuda->numParticles : 0;
	int body_id = 0;
	while ((count_particles_previous_bodies + data.vector_dynamic_bodies_data_cuda[body_id].numParticles)<i) {
		count_particles_previous_bodies += data.vector_dynamic_bodies_data_cuda[body_id].numParticles;
		body_id++;
	}

	//writte de pos
	data.posBufferGroupedDynamicBodies[i] = data.vector_dynamic_bodies_data_cuda[body_id].pos[i - count_particles_previous_bodies];
}




template<unsigned int grid_size, bool z_curve>
__global__ void DFSPH_computeGridIdx_kernel(Vector3d* in, unsigned int* out, RealCuda kernel_radius, unsigned int num_particles,
	Vector3i gridOffset) {

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	//i *= 4;
	if (i >= num_particles) { return; }

	if (z_curve) {

	}
	else {
		//the offset is used to be able to use a small grid bu placing the simulation correctly inside it
		Vector3d pos = (in[i] / kernel_radius) + gridOffset;
		pos.toFloor();
		out[i] = COMPUTE_CELL_INDEX(pos.x, pos.y, pos.z);
		/*
		pos = (in[i + 1] / kernel_radius) + gridOffset;
		pos.toFloor();
		out[i + 1] = COMPUTE_CELL_INDEX(pos.x, pos.y, pos.z);

		pos = (in[i + 2] / kernel_radius) + gridOffset;
		pos.toFloor();
		out[i + 2] = COMPUTE_CELL_INDEX(pos.x, pos.y, pos.z);

		pos = (in[i + 3] / kernel_radius) + gridOffset;
		pos.toFloor();
		out[i + 3] = COMPUTE_CELL_INDEX(pos.x, pos.y, pos.z);
		//*/
	}
}




void cuda_neighborsSearchInternal_sortParticlesId(Vector3d* pos, RealCuda kernel_radius, Vector3i gridOffset, int numParticles,
	void **d_temp_storage_pair_sort, size_t   &temp_storage_bytes_pair_sort,
	unsigned int* cell_id, unsigned int* cell_id_sorted,
	unsigned int* p_id, unsigned int* p_id_sorted) {
	hipError_t cudaStatus;


	/*
	//some test for the definition domain (it is just for debugging purposes)
	//check for negatives values
	for (int i = 0; i < numParticles; ++i) {
	Vector3d temp = (pos[i] / kernel_radius) + 2;
	if (temp.x <= 0 || temp.y <= 0 || temp.z <= 0 ) {
	fprintf(stderr, "negative coordinates: %d\n", (int)i);
	exit(1598);
	}
	}


	//find the bounding box of the particles
	Vector3d min = pos[0];
	Vector3d max = pos[0];
	for (int i = 0; i < numParticles; ++i) {

	if (pos[i].x < min.x) { min.x = pos[i].x; }
	if (pos[i].y < min.y) { min.y = pos[i].y; }
	if (pos[i].z < min.z) { min.z = pos[i].z; }

	if (pos[i].x > max.x) { max.x = pos[i].x; }
	if (pos[i].y > max.y) { max.y = pos[i].y; }
	if (pos[i].z > max.z) { max.z = pos[i].z; }

	}
	fprintf(stderr, "min: %f // %f // %f\n", min.x, min.y, min.z);
	fprintf(stderr, "max: %f // %f // %f\n", max.x, max.y, max.z);
	fprintf(stderr, "description: %f\n", CELL_ROW_LENGTH*kernel_radius);
	exit(1598);
	//*/
	std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();

	int numBlocks = calculateNumBlocks(numParticles);


	//compute the idx of the cell for each particles
	DFSPH_computeGridIdx_kernel<CELL_ROW_LENGTH, false> << <numBlocks, BLOCKSIZE >> > (pos, cell_id,
		kernel_radius, numParticles, gridOffset);

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "idxs failed: %d\n", (int)cudaStatus);
		exit(1598);
	}

	//std::chrono::steady_clock::time_point middle = std::chrono::steady_clock::now();

	// Run sorting operation
	hipcub::DeviceRadixSort::SortPairs(*d_temp_storage_pair_sort, temp_storage_bytes_pair_sort,
		cell_id, cell_id_sorted, p_id, p_id_sorted, numParticles);
	//*/

	hipGetLastError();
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "sort failed: %d\n", (int)cudaStatus);
		exit(1598);
	}

	/*
	std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
	float time0;
	float time1;
	static float time_avg = 0;
	static int time_count = 0;
	time_count++;

	time0 = std::chrono::duration_cast<std::chrono::nanoseconds> (middle - begin).count() / 1000000.0f;
	time1 = std::chrono::duration_cast<std::chrono::nanoseconds> (end - middle).count() / 1000000.0f;

	time_avg += time0 + time1;
	printf("cuda_neighborsSearchInternal_sortParticlesId: %f ms (%f,%f)   avg: %f ms \n", time0 + time1, time0, time1, time_avg / time_count);
	//*/

}


void cuda_neighborsSearchInternal_computeCellStartEnd(int numParticles, unsigned int* cell_id_sorted,
	unsigned int* hist, void **d_temp_storage_cumul_hist, size_t   &temp_storage_bytes_cumul_hist, unsigned int* cell_start_end) {
	hipError_t cudaStatus;
	int numBlocks = calculateNumBlocks(numParticles);


	//Now we need to determine the start and end of each cell
	//init the histogram values. Maybe doing it wiith thrust fill is faster.
	//the doc is not realy clear
	hipMemset(hist, 0, (CELL_COUNT + 1) * sizeof(unsigned int));
	gpuErrchk(hipDeviceSynchronize());

	//compute the actual histogram (done here with atomic adds)
	//*
	DFSPH_Histogram_kernel << <numBlocks, BLOCKSIZE >> > (cell_id_sorted, hist, numParticles);

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		std::cerr << "histogram failed: " << (int)cudaStatus << std::endl;
		exit(1598);
	}//*/

	//transformour histogram to a cumulative histogram to have  the start and end of each cell
	//note: the exlusive sum make so that each cell will contains it's start value
	// Run exclusive prefix sum
	hipcub::DeviceScan::ExclusiveSum(*d_temp_storage_cumul_hist, temp_storage_bytes_cumul_hist, hist, cell_start_end, (CELL_COUNT + 1));

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cumulative histogram failed: %d\n", (int)cudaStatus);
		exit(1598);
	}
}



__global__ void DFSPH_computeGridIdx_kernel(SPH::UnifiedParticleSet* particleSet, RealCuda kernel_radius,
	Vector3i gridOffset) {

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= particleSet->numParticles) { return; }


	//the offset is used to be able to use a small grid bu placing the simulation correctly inside it
	Vector3d pos = (particleSet->pos[i] / kernel_radius) + gridOffset;
	pos.toFloor();
	particleSet->neighborsDataSet->cell_id[i] = COMPUTE_CELL_INDEX(pos.x, pos.y, pos.z);
		
	//we can accumulate directly here
	particleSet->neighborsDataSet->cell_id_sorted[i] =
		atomicAdd(&(particleSet->neighborsDataSet->hist[particleSet->neighborsDataSet->cell_id[i]]), 1);

}


__global__ void DFSPH_CountingSortIds_kernel(SPH::UnifiedParticleSet* particleSet) {

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= particleSet->numParticles) { return; }

	int new_pos=particleSet->neighborsDataSet->cell_start_end[particleSet->neighborsDataSet->cell_id[i]] + particleSet->neighborsDataSet->cell_id_sorted[i];
	particleSet->neighborsDataSet->p_id_sorted[new_pos] = i;
}

void cuda_neighborsSearchInternal_sortParticlesId(SPH::UnifiedParticleSet& particleSet, SPH::NeighborsSearchDataSet& dataSet, SPH::DFSPHCData& data) {
	hipError_t cudaStatus;


	/*
	//some test for the definition domain (it is just for debugging purposes)
	//check for negatives values
	for (int i = 0; i < numParticles; ++i) {
	Vector3d temp = (pos[i] / kernel_radius) + 2;
	if (temp.x <= 0 || temp.y <= 0 || temp.z <= 0 ) {
	fprintf(stderr, "negative coordinates: %d\n", (int)i);
	exit(1598);
	}
	}


	//find the bounding box of the particles
	Vector3d min = pos[0];
	Vector3d max = pos[0];
	for (int i = 0; i < numParticles; ++i) {

	if (pos[i].x < min.x) { min.x = pos[i].x; }
	if (pos[i].y < min.y) { min.y = pos[i].y; }
	if (pos[i].z < min.z) { min.z = pos[i].z; }

	if (pos[i].x > max.x) { max.x = pos[i].x; }
	if (pos[i].y > max.y) { max.y = pos[i].y; }
	if (pos[i].z > max.z) { max.z = pos[i].z; }

	}
	fprintf(stderr, "min: %f // %f // %f\n", min.x, min.y, min.z);
	fprintf(stderr, "max: %f // %f // %f\n", max.x, max.y, max.z);
	fprintf(stderr, "description: %f\n", CELL_ROW_LENGTH*kernel_radius);
	exit(1598);
	//*/
	
	std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();
	
	//Now we need to determine the start and end of each cell
	//init the histogram values. Maybe doing it wiith thrust fill is faster.
	//the doc is not realy clear
	hipMemset(particleSet.neighborsDataSet->hist, 0, (CELL_COUNT + 1) * sizeof(unsigned int));
	gpuErrchk(hipDeviceSynchronize());

	std::chrono::steady_clock::time_point middle1 = std::chrono::steady_clock::now();

	int numBlocks = calculateNumBlocks(particleSet.numParticles);


	//compute the idx of the cell for each particles
	DFSPH_computeGridIdx_kernel << <numBlocks, BLOCKSIZE >> > (particleSet.gpu_ptr,
		data.getKernelRadius(), data.gridOffset);
	gpuErrchk(hipDeviceSynchronize());

	std::chrono::steady_clock::time_point middle2 = std::chrono::steady_clock::now();

	//transformour histogram to a cumulative histogram to have  the start and end of each cell
	//note: the exlusive sum make so that each cell will contains it's start value
	// Run exclusive prefix sum
	hipcub::DeviceScan::ExclusiveSum(particleSet.neighborsDataSet->d_temp_storage_cumul_hist, particleSet.neighborsDataSet->temp_storage_bytes_cumul_hist, 
		particleSet.neighborsDataSet->hist, particleSet.neighborsDataSet->cell_start_end, (CELL_COUNT + 1));
	gpuErrchk(hipDeviceSynchronize());

	DFSPH_CountingSortIds_kernel << <numBlocks, BLOCKSIZE >> > (particleSet.gpu_ptr);
	gpuErrchk(hipDeviceSynchronize());

	/*
	std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
	float time0;
	float time1;
	float time2;
	static float time_avg = 0;
	static int time_count = 0;
	time_count++;

	time0 = std::chrono::duration_cast<std::chrono::nanoseconds> (middle1 - begin).count() / 1000000.0f;
	time1 = std::chrono::duration_cast<std::chrono::nanoseconds> (middle2 - middle1).count() / 1000000.0f;
	time2 = std::chrono::duration_cast<std::chrono::nanoseconds> (end - middle2).count() / 1000000.0f;

	time_avg += time0 + time1 + time2;
	printf("cuda_neighborsSearchInternal_sortParticlesId: %f ms (%f,%f,%f)   avg: %f ms \n", time0 + time1 + time2, time0, time1, time2, time_avg / time_count);
	//*/

}




void cuda_initNeighborsSearchDataSet(SPH::UnifiedParticleSet& particleSet, SPH::NeighborsSearchDataSet& dataSet,
	SPH::DFSPHCData& data, bool sortBuffers) {



	std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();
	//com the id
	cuda_neighborsSearchInternal_sortParticlesId(particleSet, dataSet, data);

	//cuda_neighborsSearchInternal_sortParticlesId(particleSet.pos, data.getKernelRadius(), data.gridOffset, dataSet.numParticles,
	//	&dataSet.d_temp_storage_pair_sort, dataSet.temp_storage_bytes_pair_sort, dataSet.cell_id, dataSet.cell_id_sorted,
	//	dataSet.p_id, dataSet.p_id_sorted);
	std::chrono::steady_clock::time_point middle1 = std::chrono::steady_clock::now();

	//since it the init iter I'll sort both even if it's the boundaries
	if (sortBuffers) {
		cuda_sortData(particleSet, dataSet.p_id_sorted);
	}

	std::chrono::steady_clock::time_point middle2 = std::chrono::steady_clock::now();


	//and now I cna compute the start and end of each cell :)
	//cuda_neighborsSearchInternal_computeCellStartEnd(dataSet.numParticles, dataSet.cell_id_sorted, dataSet.hist,
	//	&dataSet.d_temp_storage_cumul_hist, dataSet.temp_storage_bytes_cumul_hist, dataSet.cell_start_end);





	/*
	std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
	float time0;
	float time1;
	float time2;
	static float time_avg = 0;
	static int time_count = 0;
	time_count++;

	time0 = std::chrono::duration_cast<std::chrono::nanoseconds> (middle1 - begin).count() / 1000000.0f;
	time1 = std::chrono::duration_cast<std::chrono::nanoseconds> (middle2 - middle1).count() / 1000000.0f;
	time2 = std::chrono::duration_cast<std::chrono::nanoseconds> (end - middle2).count() / 1000000.0f;

	time_avg += time0 + time1 + time2;
	printf("Time to generate cell start end internal: %f ms (%f,%f,%f)   avg: %f ms \n", time0 + time1 + time2, time0, time1, time2, time_avg / time_count);
	//*/

}

void cuda_initNeighborsSearchDataSetGroupedDynamicBodies(SPH::DFSPHCData& data) {
	if (data.numDynamicBodies<1) {
		return;
	}

	SPH::NeighborsSearchDataSet& dataSet = *(data.neighborsDataSetGroupedDynamicBodies);


	//before anything I need to update the number of active particles
	int numParticles = (data.is_fluid_aggregated) ? data.fluid_data[0].numParticles : 0;
	for (int i = 0; i<data.numDynamicBodies; ++i) {
		numParticles += data.vector_dynamic_bodies_data[i].numParticles;
	}

	if (dataSet.numParticles != numParticles) {
		if (numParticles <= (int)dataSet.numParticlesMax) {
			dataSet.updateActiveParticleNumber(numParticles);
		}
		else {
			std::ostringstream oss;
			oss << "TODO::I need to add particles to the grouped data struct when the number of particle goes above the max" <<
				" current max: " << dataSet.numParticlesMax << "  number of particles: " << numParticles << std::endl;
			throw(oss.str());
		}
	}

	// now fill itr
	int numBlocks = calculateNumBlocks(dataSet.numParticles);
	DFSPH_fill_aggregated_pos_buffer_kernel << <numBlocks, BLOCKSIZE >> > (data, dataSet.numParticles);
	gpuErrchk(hipDeviceSynchronize());

	//and now we can do the neighbor search
	//com the id
	cuda_neighborsSearchInternal_sortParticlesId(data.posBufferGroupedDynamicBodies, data.getKernelRadius(), data.gridOffset, dataSet.numParticles,
		&dataSet.d_temp_storage_pair_sort, dataSet.temp_storage_bytes_pair_sort, dataSet.cell_id, dataSet.cell_id_sorted,
		dataSet.p_id, dataSet.p_id_sorted);



	//and now I cna compute the start and end of each cell :)
	cuda_neighborsSearchInternal_computeCellStartEnd(dataSet.numParticles, dataSet.cell_id_sorted, dataSet.hist,
		&dataSet.d_temp_storage_cumul_hist, dataSet.temp_storage_bytes_cumul_hist, dataSet.cell_start_end);



}

template<typename T>
__global__ void DFSPH_sortFromIndex_kernel(T* in, T* out, unsigned int* index, unsigned int nbElements) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= nbElements) { return; }

	out[i] = in[index[i]];
}



void cuda_sortData(SPH::UnifiedParticleSet& particleSet, unsigned int * sort_id) {
	//*
	unsigned int numParticles = particleSet.neighborsDataSet->numParticles;
	int numBlocks = calculateNumBlocks(numParticles);
	unsigned int *p_id_sorted = sort_id;

	Vector3d* intermediate_buffer_v3d = particleSet.neighborsDataSet->intermediate_buffer_v3d;
	RealCuda* intermediate_buffer_real = particleSet.neighborsDataSet->intermediate_buffer_real;

	//pos
	DFSPH_sortFromIndex_kernel<Vector3d> << <numBlocks, BLOCKSIZE >> > (particleSet.pos, intermediate_buffer_v3d, p_id_sorted, numParticles);
	gpuErrchk(hipDeviceSynchronize());
	gpuErrchk(hipMemcpy(particleSet.pos, intermediate_buffer_v3d, numParticles * sizeof(Vector3d), hipMemcpyDeviceToDevice));

	//vel
	DFSPH_sortFromIndex_kernel<Vector3d> << <numBlocks, BLOCKSIZE >> > (particleSet.vel, intermediate_buffer_v3d, p_id_sorted, numParticles);
	gpuErrchk(hipDeviceSynchronize());
	gpuErrchk(hipMemcpy(particleSet.vel, intermediate_buffer_v3d, numParticles * sizeof(Vector3d), hipMemcpyDeviceToDevice));

	//color
	if (particleSet.has_color_buffer) {
		DFSPH_sortFromIndex_kernel<Vector3d> << <numBlocks, BLOCKSIZE >> > (particleSet.color, intermediate_buffer_v3d, p_id_sorted, numParticles);
		gpuErrchk(hipDeviceSynchronize());
		gpuErrchk(hipMemcpy(particleSet.color, intermediate_buffer_v3d, numParticles * sizeof(Vector3d), hipMemcpyDeviceToDevice));
	}
	
	//mass
	DFSPH_sortFromIndex_kernel<RealCuda> << <numBlocks, BLOCKSIZE >> > (particleSet.mass, intermediate_buffer_real, p_id_sorted, numParticles);
	gpuErrchk(hipDeviceSynchronize());
	gpuErrchk(hipMemcpy(particleSet.mass, intermediate_buffer_real, numParticles * sizeof(RealCuda), hipMemcpyDeviceToDevice));


	if (particleSet.has_factor_computation) {
		//kappa
		DFSPH_sortFromIndex_kernel<RealCuda> << <numBlocks, BLOCKSIZE >> > (particleSet.kappa, intermediate_buffer_real, p_id_sorted, numParticles);
		gpuErrchk(hipDeviceSynchronize());
		gpuErrchk(hipMemcpy(particleSet.kappa, intermediate_buffer_real, numParticles * sizeof(RealCuda), hipMemcpyDeviceToDevice));

		//kappav
		DFSPH_sortFromIndex_kernel<RealCuda> << <numBlocks, BLOCKSIZE >> > (particleSet.kappaV, intermediate_buffer_real, p_id_sorted, numParticles);
		gpuErrchk(hipDeviceSynchronize());
		gpuErrchk(hipMemcpy(particleSet.kappaV, intermediate_buffer_real, numParticles * sizeof(RealCuda), hipMemcpyDeviceToDevice));
	}



	//now that everything is sorted we can set each particle index to itself
	gpuErrchk(hipMemcpy(p_id_sorted, particleSet.neighborsDataSet->p_id, numParticles * sizeof(unsigned int), hipMemcpyDeviceToDevice));

}



__global__ void generateShuffleIndex_kernel(unsigned int *shuffle_index, unsigned int nbElements, hiprandState *state) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= 1) { return; }

	for (int j = 0; j < nbElements; ++j) {
		shuffle_index[j] = j;
	}



	hiprandState localState = *state;
	for (int j = 0; j < nbElements; ++j) {
		float x = hiprand_uniform(&localState);
		x *= nbElements;
		unsigned int idx = x;
		if (x < nbElements) {
			unsigned int temp = shuffle_index[idx];
			shuffle_index[idx] = shuffle_index[i];
			shuffle_index[i] = temp;
		}
	}
	*state = localState;
}


void cuda_shuffleData(SPH::UnifiedParticleSet& particleSet) {
	unsigned int numParticles = particleSet.numParticles;
	int numBlocks = calculateNumBlocks(numParticles);

	//create a random sorting index
	unsigned int* shuffle_index = SVS_CU::get()->shuffle_index;
	hiprandState *state = SVS_CU::get()->curand_state;
	if (shuffle_index == NULL) {
		hipMallocManaged(&(SVS_CU::get()->shuffle_index), particleSet.numParticlesMax * sizeof(unsigned int));
		shuffle_index = SVS_CU::get()->shuffle_index;
		gpuErrchk(hipDeviceSynchronize());
	}


	generateShuffleIndex_kernel << <1, 1 >> > (shuffle_index, numParticles, state);
	gpuErrchk(hipDeviceSynchronize());


	unsigned int *p_id_sorted = shuffle_index;

	Vector3d* intermediate_buffer_v3d = particleSet.neighborsDataSet->intermediate_buffer_v3d;
	RealCuda* intermediate_buffer_real = particleSet.neighborsDataSet->intermediate_buffer_real;

	//pos
	DFSPH_sortFromIndex_kernel<Vector3d> << <numBlocks, BLOCKSIZE >> > (particleSet.pos, intermediate_buffer_v3d, p_id_sorted, numParticles);
	gpuErrchk(hipDeviceSynchronize());
	gpuErrchk(hipMemcpy(particleSet.pos, intermediate_buffer_v3d, numParticles * sizeof(Vector3d), hipMemcpyDeviceToDevice));

	//vel
	DFSPH_sortFromIndex_kernel<Vector3d> << <numBlocks, BLOCKSIZE >> > (particleSet.vel, intermediate_buffer_v3d, p_id_sorted, numParticles);
	gpuErrchk(hipDeviceSynchronize());
	gpuErrchk(hipMemcpy(particleSet.vel, intermediate_buffer_v3d, numParticles * sizeof(Vector3d), hipMemcpyDeviceToDevice));

	//mass
	DFSPH_sortFromIndex_kernel<RealCuda> << <numBlocks, BLOCKSIZE >> > (particleSet.mass, intermediate_buffer_real, p_id_sorted, numParticles);
	gpuErrchk(hipDeviceSynchronize());
	gpuErrchk(hipMemcpy(particleSet.mass, intermediate_buffer_real, numParticles * sizeof(RealCuda), hipMemcpyDeviceToDevice));

	if (particleSet.has_factor_computation) {
		//kappa
		DFSPH_sortFromIndex_kernel<RealCuda> << <numBlocks, BLOCKSIZE >> > (particleSet.kappa, intermediate_buffer_real, p_id_sorted, numParticles);
		gpuErrchk(hipDeviceSynchronize());
		gpuErrchk(hipMemcpy(particleSet.kappa, intermediate_buffer_real, numParticles * sizeof(RealCuda), hipMemcpyDeviceToDevice));

		//kappav
		DFSPH_sortFromIndex_kernel<RealCuda> << <numBlocks, BLOCKSIZE >> > (particleSet.kappaV, intermediate_buffer_real, p_id_sorted, numParticles);
		gpuErrchk(hipDeviceSynchronize());
		gpuErrchk(hipMemcpy(particleSet.kappaV, intermediate_buffer_real, numParticles * sizeof(RealCuda), hipMemcpyDeviceToDevice));
	}



}






template <bool is_fluid_container>
__global__ void DFSPH_neighborsSearch_kernel(SPH::DFSPHCData data, SPH::UnifiedParticleSet* particleSet) {

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= particleSet->numParticles) { return; }


	ITER_NEIGHBORS_INIT_FROM_STRUCTURE(data, particleSet, i);

	//this variable is need for the interleave but I'm not modifying all the macro for only a single case
#ifdef INTERLEAVE_NEIGHBORS
	int numParticles = particleSet->numParticles;
#endif

	unsigned int nb_neighbors_fluid = 0;
	unsigned int nb_neighbors_boundary = 0;
	unsigned int nb_neighbors_dynamic_objects = 0;
	int* cur_neighbor_ptr= particleSet->getNeighboursPtr(i);
	//int neighbors_fluid[MAX_NEIGHBOURS];//doing it with local buffer was not faster
	//int neighbors_boundary[MAX_NEIGHBOURS];


	
	if (data.is_fluid_aggregated) {
		int neighbors_solids[MAX_NEIGHBOURS];

		//dynamic bodies
		if (data.numDynamicBodies >0) {

#ifdef GROUP_DYNAMIC_BODIES_NEIGHBORS_SEARCH
			ITER_NEIGHBORS_FROM_STRUCTURE(data.neighborsDataSetGroupedDynamicBodies_cuda, data.posBufferGroupedDynamicBodies,
				if (j<data.fluid_data_cuda->numParticles) {
					if (i != j) { WRITE_AND_ADVANCE_NEIGHBORS(cur_neighbor_ptr, j);	nb_neighbors_fluid++; }
				}
				else {
					int body_id = 0; int count_particles_previous_bodies = data.fluid_data_cuda->numParticles;
					while ((count_particles_previous_bodies + data.vector_dynamic_bodies_data_cuda[body_id].numParticles)<j) {
						count_particles_previous_bodies += data.vector_dynamic_bodies_data_cuda[body_id].numParticles;
						body_id++;
					}
					//*cur_neighbor_ptr++ = WRITE_DYNAMIC_BODIES_PARTICLES_INDEX(body_id, j-count_particles_previous_bodies);
					neighbors_solids[nb_neighbors_dynamic_objects] = WRITE_DYNAMIC_BODIES_PARTICLES_INDEX(body_id, j - count_particles_previous_bodies);
					nb_neighbors_dynamic_objects++;
				})
#else
			for (int id_body = 0; id_body < data.numDynamicBodies; ++id_body) {
				ITER_NEIGHBORS_FROM_STRUCTURE(data.vector_dynamic_bodies_data_cuda[id_body].neighborsDataSet, data.vector_dynamic_bodies_data_cuda[id_body].pos,
					*cur_neighbor_ptr++ = WRITE_DYNAMIC_BODIES_PARTICLES_INDEX(id_body, j); nb_neighbors_dynamic_objects++; )
			}
#endif

		}
		else {
			//fluid
			ITER_NEIGHBORS_FROM_STRUCTURE(data.fluid_data_cuda[0].neighborsDataSet, data.fluid_data_cuda[0].pos,
				if (i != j) { WRITE_AND_ADVANCE_NEIGHBORS(cur_neighbor_ptr, j);	nb_neighbors_fluid++; });
		}

		//boundaries
#ifndef BENDER2019_BOUNDARIES
		ITER_NEIGHBORS_FROM_STRUCTURE(data.boundaries_data_cuda[0].neighborsDataSet, data.boundaries_data_cuda[0].pos,
			WRITE_AND_ADVANCE_NEIGHBORS(cur_neighbor_ptr, j); nb_neighbors_boundary++; );
#endif

		//copy the dynamic bodies at the end
		for (int j = 0; j<nb_neighbors_dynamic_objects; ++j) {
			WRITE_AND_ADVANCE_NEIGHBORS(cur_neighbor_ptr, neighbors_solids[j]);
		}

	}
	else {

		//uses the standart version
		//fluid
		if (is_fluid_container) {

			ITER_NEIGHBORS_FROM_STRUCTURE(data.fluid_data_cuda[0].neighborsDataSet, data.fluid_data_cuda[0].pos,
				if (!is_fluid_container || i != j) { WRITE_AND_ADVANCE_NEIGHBORS(cur_neighbor_ptr, j);	nb_neighbors_fluid++; });

		}

		//boundaries
#ifndef BENDER2019_BOUNDARIES
		ITER_NEIGHBORS_FROM_STRUCTURE(data.boundaries_data_cuda[0].neighborsDataSet, data.boundaries_data_cuda[0].pos,
			if (is_fluid_container || i != j) { WRITE_AND_ADVANCE_NEIGHBORS(cur_neighbor_ptr, j); nb_neighbors_boundary++; });
#endif

		if (data.numDynamicBodies > 0) {

#ifdef GROUP_DYNAMIC_BODIES_NEIGHBORS_SEARCH
			ITER_NEIGHBORS_FROM_STRUCTURE(data.neighborsDataSetGroupedDynamicBodies_cuda, data.posBufferGroupedDynamicBodies,
			{ int body_id = 0; int count_particles_previous_bodies = 0;
			while ((count_particles_previous_bodies + data.vector_dynamic_bodies_data_cuda[body_id].numParticles)<j) {
				count_particles_previous_bodies += data.vector_dynamic_bodies_data_cuda[body_id].numParticles;
				body_id++;
			}
			int neighbor_idx= WRITE_DYNAMIC_BODIES_PARTICLES_INDEX(body_id, j - count_particles_previous_bodies);
			WRITE_AND_ADVANCE_NEIGHBORS(cur_neighbor_ptr, neighbor_idx);
			nb_neighbors_dynamic_objects++; })
#else
			for (int id_body = 0; id_body < data.numDynamicBodies; ++id_body) {
				ITER_NEIGHBORS_FROM_STRUCTURE(data.vector_dynamic_bodies_data_cuda[id_body].neighborsDataSet, data.vector_dynamic_bodies_data_cuda[id_body].pos,
                    int neighbor_idx = WRITE_DYNAMIC_BODIES_PARTICLES_INDEX(id_body, j);
				WRITE_AND_ADVANCE_NEIGHBORS(cur_neighbor_ptr, neighbor_idx); )
			}
#endif

		}

	}



	particleSet->numberOfNeighbourgs[3 * i] =  nb_neighbors_fluid;
	particleSet->numberOfNeighbourgs[3 * i + 1] = nb_neighbors_boundary;
	particleSet->numberOfNeighbourgs[3 * i + 2] = nb_neighbors_dynamic_objects;
	
	/*
	//simple splashless surface detection
	if (((nb_neighbors_fluid+nb_neighbors_boundary) < 35)&& (nb_neighbors_fluid + nb_neighbors_boundary) >15) {
		particleSet->color[i] = Vector3d(0, 1, 0);
	}
	//*/

	//memcpy((neighbors_buff + i*MAX_NEIGHBOURS*2), neighbors_fluid, sizeof(int)*nb_neighbors_fluid);
	//memcpy((neighbors_buff + i*MAX_NEIGHBOURS * 2 + MAX_NEIGHBOURS), neighbors_boundary, sizeof(int)*nb_neighbors_boundary);


}

__global__ void DFSPH_neighborsSearchBasic_kernel(unsigned int numFluidParticles, RealCuda radius,
	SPH::UnifiedParticleSet* fluid_data,
	SPH::UnifiedParticleSet* boundaries_data,
	SPH::UnifiedParticleSet* vect_dynamic_bodies, int nb_dynamic_bodies) {

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= numFluidParticles) { return; }


	RealCuda radius_sq = radius;
	Vector3d pos = fluid_data->pos[i];
	radius_sq *= radius_sq;

	unsigned int nb_neighbors_fluid = 0;
	unsigned int nb_neighbors_boundary = 0;
	unsigned int nb_neighbors_dynamic_objects = 0;
	int* cur_neighbor_ptr = fluid_data->neighbourgs + i*MAX_NEIGHBOURS;

	for (int k = 0; k < fluid_data->numParticles; ++k) {
		if (i != k) {
			if ((fluid_data->pos[k] - pos).squaredNorm() < radius_sq) {
				*cur_neighbor_ptr++ = k;	nb_neighbors_fluid++;
			}
		}
	}

	/*
	for (int k = 0; k < boundaries_data->numParticles; ++k) {
	if ((boundaries_data->pos[k] - pos).squaredNorm() < radius_sq) {
	*cur_neighbor_ptr++ = k; nb_neighbors_boundary++;
	}
	}
	//*/

	/*
	for (int id_body = 0; id_body < nb_dynamic_bodies; ++id_body) {
	for (int k = 0; k < vect_dynamic_bodies[id_body].numParticles; ++k) {
	if ((vect_dynamic_bodies[id_body].pos[k] - pos).squaredNorm() < radius_sq) {
	*cur_neighbor_ptr++ = WRITTE_DYNAMIC_BODIES_PARTICLES_INDEX(id_body, k); nb_neighbors_dynamic_objects++;
	}
	}
	}
	//*/


	fluid_data->numberOfNeighbourgs[3 * i] = nb_neighbors_fluid;
	fluid_data->numberOfNeighbourgs[3 * i + 1] = nb_neighbors_boundary;
	fluid_data->numberOfNeighbourgs[3 * i + 2] = nb_neighbors_dynamic_objects;

}




void cuda_neighborsSearch(SPH::DFSPHCData& data) {

	//std::chrono::steady_clock::time_point begin_global = std::chrono::steady_clock::now();
	static unsigned int time_count = 0;
	float time_global;
	static float time_avg_global = 0;
	time_count++;

	/*
	if (time_count<5) {
	cuda_shuffleData(data.fluid_data[0]);
	std::cout << "randomizing particle order" << std::endl;
	}
	//*/

	bool need_sort = true;// ((time_count % 15) == 0);

	if (need_sort) {
		//std::cout<<"doing full neighbor search"<<std::endl;
	}

	bool old_fluid_aggregated = data.is_fluid_aggregated;
	hipError_t cudaStatus;
	if (true) {
		if (need_sort&&data.is_fluid_aggregated) {
			data.is_fluid_aggregated = false;
		}


		//*
		std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();
		//*/

		//first let's generate the cell start end for the dynamic bodies
#ifdef GROUP_DYNAMIC_BODIES_NEIGHBORS_SEARCH
		cuda_initNeighborsSearchDataSetGroupedDynamicBodies(data);
#else
        for (int i = 0; i < data.numDynamicBodies; ++i) {
            data.vector_dynamic_bodies_data[i].initNeighborsSearchData(data, false);
		}
#endif
		std::chrono::steady_clock::time_point middle = std::chrono::steady_clock::now();

		//no need to ever do it forthe boundaries since they don't ever move

		//now update the cell start end of the fluid particles
		if ((!data.is_fluid_aggregated) || data.numDynamicBodies<1) {

			//since it the init iter I'll sort both even if it's the boundaries
			static int step_count = 0;
			step_count++;

			data.fluid_data->initNeighborsSearchData(data, need_sort);


			cudaStatus = hipDeviceSynchronize();
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "before neighbors search: %d\n", (int)cudaStatus);
				exit(1598);
			}


		}

		/*

		std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
		float time0;
		float time1;
		static float time_avg = 0;
		time0 = std::chrono::duration_cast<std::chrono::nanoseconds> (middle - begin).count() / 1000000.0f;
		time1 = std::chrono::duration_cast<std::chrono::nanoseconds> (end - middle).count() / 1000000.0f;

		time_avg += time0+time1;
		printf("Time to generate cell start end: %f ms (%f,%f)   avg: %f ms \n", time0+time1,time0,time1, time_avg / time_count);

		if (time_count > 150) {
		time_avg = 0;
		}
		//*/


	}
	//and we can now do the actual search of the neaighbor for eahc fluid particle
#ifdef STORE_PARTICLE_NEIGHBORS

	if (true)
	{
		//*
		float time;
		static float time_avg = 0;
		std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();
		//*/

		//cuda way
		int numBlocks = calculateNumBlocks(data.fluid_data[0].numParticles);

		//*
		DFSPH_neighborsSearch_kernel<true> << <numBlocks, BLOCKSIZE >> > (data, data.fluid_data_cuda);

		//*
		if (data.boundaries_data->has_factor_computation) {
			DFSPH_neighborsSearch_kernel<false> << <numBlocks, BLOCKSIZE >> > (data, data.boundaries_data_cuda);
		}
		//*/

		//*/
		/*
		//this test show that even just computing the neighbors for the fluid particle
		//with a basic method take more time than building the whole structure
		DFSPH_neighborsSearchBasic_kernel << <numBlocks, BLOCKSIZE >> > (data.numFluidParticles,
		data.getKernelRadius(),
		data.fluid_data_cuda,
		data.boundaries_data_cuda,
		data.vector_dynamic_bodies_data_cuda, data.numDynamicBodies);
		//*/

		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			std::cerr << "cuda neighbors search failed: " << (int)cudaStatus << std::endl;
			exit(1598);
		}

		/*
		std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
		time = std::chrono::duration_cast<std::chrono::nanoseconds> (end - begin).count() / 1000000.0f;

		time_avg += time;
		printf("Time to generate neighbors buffers: %f ms   avg: %f ms \n", time, time_avg / time_count);

		if (time_count > 150) {
		time_avg = 0;
		time_count = 0;
		}
		//*/



        /*
		{
            if(false){
			std::cout << "test: " << data.fluid_data->neighborsDataSet->cell_id_sorted[0] << "   " <<
				data.fluid_data->neighborsDataSet->cell_id_sorted[10] << "   " <<
				data.fluid_data->neighborsDataSet->cell_id_sorted[50] << "   " << std::endl;

			int count_valid = 0;
			for (int i = 0; i < data.fluid_data->numParticles; ++i) {
				if (data.fluid_data->neighborsDataSet->cell_id_sorted[i] != 0) {
					count_valid++;
				}
			}
			std::cout << "test2: " << count_valid << std::endl;
			}


		//a simple check to know the max nbr of neighbors
		static int absolute_max = 0;
		int max = 0;

		static int absolute_max_d[3] = { 0 };
		int max_d[3] = { 0 };



		for (int j = 0; j < data.fluid_data->getNumberOfNeighbourgs(j); j++)
		{
		//check the global value
		int count_neighbors = 0;
		for (int k = 0; k < 3; ++k) {
		count_neighbors += data.fluid_data->getNumberOfNeighbourgs(j, k);
		}
		if (count_neighbors > max)max = count_neighbors;

		//chekc the max for each category
		for (unsigned int k = 0; k < 3; ++k) {
		if ((int)data.fluid_data->getNumberOfNeighbourgs(j,k) > max_d[k])max_d[k] = data.fluid_data->getNumberOfNeighbourgs(j,k);
		}

		}
		if (max>absolute_max)absolute_max = max;
		for (unsigned int k = 0; k < 3; ++k) {
		if (max_d[k]>absolute_max_d[k])absolute_max_d[k] = max_d[k];
		}
		printf("max nbr of neighbors %d  (%d) \n", absolute_max, max);
		printf("max nbr of neighbors %d  (%d)      absolute max  fluid // boundaries // bodies   %d // %d // %d\n",
		absolute_max, max, absolute_max_d[0], absolute_max_d[1], absolute_max_d[2]);
		}


		//*/
		/*
		{
		//another test ot be sure the contruction of the boundries neighbors works orrectly
		if (data.boundaries_data->has_factor_computation) {
		//a simple check to know the  nbr of neighbors of the first boundries particle

		int nb_neighbors[3] = { 0 };

		for (int k = 0; k < 3; ++k) {
		nb_neighbors[k] = data.boundaries_data->getNumberOfNeighbourgs(0, k);
		}

		printf(" nbr of neighbors %d     fluid // boundaries // bodies   %d // %d // %d\n",
		nb_neighbors[0] + nb_neighbors[1] + nb_neighbors[2], nb_neighbors[0], nb_neighbors[1], nb_neighbors[2]);
		}
		}

		//*/

		
	}
#endif //STORE_PARTICLE_NEIGHBORS

	//reactive the aggragation if we desactivated it because a sort was required
	if (need_sort&&old_fluid_aggregated) {
		data.is_fluid_aggregated = true;
	}



	/*
	std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
	time_global = std::chrono::duration_cast<std::chrono::nanoseconds> (end - begin_global).count() / 1000000.0f;

	time_avg_global += time_global;
	printf("time taken by the neighbor function: %f ms   avg: %f ms \n", time_global, time_avg_global / time_count);
	//*/
}



////////////////////////////////////////////////////
/////////             OTHERS           /////////////
////////////////////////////////////////////////////

__global__ void DFSPH_update_vel_kernel(SPH::DFSPHCData m_data, SPH::UnifiedParticleSet* particleSet) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= particleSet->numParticles) { return; }

	particleSet->vel[i] += m_data.h * particleSet->acc[i];

#ifdef USE_WARMSTART	
	//done here to have one less kernel
	particleSet->kappa[i] = MAX_MACRO_CUDA(particleSet->kappa[i] * m_data.h_ratio_to_past2, -0.5);
#endif
}




void cuda_update_vel(SPH::DFSPHCData& data) {
	int numBlocks = calculateNumBlocks(data.fluid_data[0].numParticles);
	DFSPH_update_vel_kernel << <numBlocks, BLOCKSIZE >> > (data, data.fluid_data[0].gpu_ptr);

	hipError_t cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cuda_update_vel failed: %d\n", (int)cudaStatus);
		exit(1598);
    }

}


__global__ void DFSPH_update_pos_kernel(SPH::DFSPHCData data, SPH::UnifiedParticleSet* particleSet) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= particleSet->numParticles) { return; }

	if (data.damp_borders) {
		/*
		RealCuda max_vel_sq = (data.particleRadius / 2.0f) / data.h;
		max_vel_sq *= max_vel_sq;
		RealCuda cur_vel_sq = particleSet->vel[i].squaredNorm();
		if (cur_vel_sq> max_vel_sq)
		{
		particleSet->vel[i] *= max_vel_sq / cur_vel_sq;
		}//*/

		RealCuda affected_distance_sq = data.particleRadius * 6;
		affected_distance_sq *= affected_distance_sq;

		for (int k = 0; k < data.damp_planes_count; ++k) {
			Vector3d plane = data.damp_planes[k];
			if ((particleSet->pos[i] * plane.abs() / plane.norm() - plane).squaredNorm() < affected_distance_sq) {
				if (data.damp_borders_steps_count>1) {
					RealCuda max_vel_sq = (data.particleRadius / 25.0f) / data.h;
					max_vel_sq *= max_vel_sq;
					RealCuda cur_vel_sq = particleSet->vel[i].squaredNorm();
					if (cur_vel_sq> max_vel_sq)
					{
						particleSet->vel[i] *= max_vel_sq / cur_vel_sq;
					}
					//if we triggered once no need to check for the other planes
					break;
				}
				else {
					particleSet->vel[i] *= 0.1;
				}
			}
		}
	}

	if (data.cancel_wave) {
		RealCuda affected_distance_sq = data.getKernelRadius();
		affected_distance_sq *= affected_distance_sq;
		for (int k = 0; k < 2; ++k) {
			Vector3i plane = data.cancel_wave_planes[k];
			if ((particleSet->pos[i] * plane.abs() / plane.norm() - plane).squaredNorm() < affected_distance_sq) {
				//particleSet->vel[i]=Vector3d(0,1,0);
			}
		}
		Vector3d axis = data.cancel_wave_planes[0].abs() / data.cancel_wave_planes[0].norm();
		if (particleSet->pos[i].y>data.cancel_wave_lowest_point) {
			if ((particleSet->pos[i].dot(axis))<(data.cancel_wave_planes[0].dot(axis))) {
				if ((particleSet->vel[i].dot(axis))<0) {
					particleSet->vel[i] -= particleSet->vel[i] * axis;
				}
			}

			if ((particleSet->pos[i].dot(axis))>(data.cancel_wave_planes[1].dot(axis))) {
				if ((particleSet->vel[i].dot(axis))>0) {
					particleSet->vel[i] -= particleSet->vel[i] * axis;
				}
			}
		}

	}


	particleSet->pos[i] += data.h * particleSet->vel[i];
}



void cuda_update_pos(SPH::DFSPHCData& data) {


	int numBlocks = calculateNumBlocks(data.fluid_data[0].numParticles);
	DFSPH_update_pos_kernel << <numBlocks, BLOCKSIZE >> > (data, data.fluid_data[0].gpu_ptr);


	hipError_t cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cuda_update_pos failed: %d\n", (int)cudaStatus);
		exit(1598);
	}

	if (data.damp_borders) {
		for (int k = 0; k < data.damp_planes_count; ++k) {
			Vector3d plane = data.damp_planes[k];
			//std::cout << "damping plane: " << plane.x << "  " << plane.y << "  " << plane.z << std::endl;
		}
		data.damp_borders_steps_count--;
		if (data.damp_borders_steps_count == 0) {
			data.damp_borders = false;
			data.damp_planes_count = 0;
		}
	}
	if (data.cancel_wave) {
		//*

		for (int k = 0; k < 2; ++k) {
			Vector3d plane = data.cancel_wave_planes[k];
			std::cout << "cancel wave plane: " << plane.x << "  " << plane.y << "  " << plane.z << std::endl;
		}
		data.cancel_wave_steps_count--;
		if (data.cancel_wave_steps_count == 0) {
			data.cancel_wave = false;
		}
		//*/
	}


}




__global__ void DFSPH_CFL_kernel(SPH::DFSPHCData m_data, SPH::UnifiedParticleSet particleSet, RealCuda* maxVel) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= m_data.fluid_data[0].numParticles) { return; }

	for (unsigned int i = 0; i < m_data.fluid_data[0].numParticles; i++)
	{
		const RealCuda velMag = (particleSet.vel[i] + particleSet.acc[i] * m_data.h).squaredNorm();
		if (velMag > *maxVel)
			*maxVel = velMag;
	}
}

__global__ void DFSPH_CFLVelSquaredNorm_kernel(SPH::DFSPHCData m_data, SPH::UnifiedParticleSet* particleSet, RealCuda* sqaredNorm) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= particleSet->numParticles) { return; }

	sqaredNorm[i] = (particleSet->vel[i] + particleSet->acc[i] * m_data.h).squaredNorm();
}

__global__ void DFSPH_CFLAdvanced_kernel(SPH::DFSPHCData m_data, RealCuda *max, int *mutex, unsigned int n)
{
	unsigned int index = threadIdx.x + blockIdx.x*blockDim.x;
	unsigned int stride = gridDim.x*blockDim.x;
	unsigned int offset = 0;

	__shared__ RealCuda cache[256];


	RealCuda temp = 0;
	while (index + offset < n) {
		int i = index + offset;
		const RealCuda velMag = (m_data.fluid_data_cuda->vel[i] + m_data.fluid_data_cuda->acc[i] * m_data.h).squaredNorm();
		temp = fmaxf(temp, velMag);

		offset += stride;
	}

	cache[threadIdx.x] = temp;

	__syncthreads();


	// reduction
	unsigned int i = blockDim.x / 2;
	while (i != 0) {
		if (threadIdx.x < i) {
			cache[threadIdx.x] = MAX_MACRO_CUDA(cache[threadIdx.x], cache[threadIdx.x + i]);
		}

		__syncthreads();
		i /= 2;
	}

	if (threadIdx.x == 0) {
		while (atomicCAS(mutex, 0, 1) != 0);  //lock
		*max = MAX_MACRO_CUDA(*max, cache[0]);
		atomicExch(mutex, 0);  //unlock
	}
}

void cuda_CFL(SPH::DFSPHCData& m_data, const RealCuda minTimeStepSize, RealCuda m_cflFactor, RealCuda m_cflMaxTimeStepSize) {

	//we compute the square norm

	std::chrono::steady_clock::time_point t0 = std::chrono::steady_clock::now();

	RealCuda* out_buff;
	hipMalloc(&(out_buff), sizeof(RealCuda));

	if (true) {

		//cub version
		static RealCuda* temp_buff = NULL;
		if (temp_buff == NULL) {
			hipMallocManaged(&(temp_buff), m_data.fluid_data[0].numParticles * sizeof(RealCuda));
		}
		int numBlocks = calculateNumBlocks(m_data.fluid_data[0].numParticles);
		DFSPH_CFLVelSquaredNorm_kernel << <numBlocks, BLOCKSIZE >> > (m_data, m_data.fluid_data[0].gpu_ptr, temp_buff);

		hipError_t cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "cuda_cfl squared norm failed: %d\n", (int)cudaStatus);
			exit(1598);
		}

		// Determine temporary device storage requirements
		static void     *d_temp_storage = NULL;
		static size_t   temp_storage_bytes = 0;
		if (d_temp_storage == NULL) {
			hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, temp_buff, out_buff, m_data.fluid_data[0].numParticles);
			// Allocate temporary storage
			hipMalloc(&d_temp_storage, temp_storage_bytes);
		}
		// Run max-reduction
		hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, temp_buff, out_buff, m_data.fluid_data[0].numParticles);

	}
	else {
		//manual
		int *d_mutex;
		hipMalloc((void**)&d_mutex, sizeof(int));
		hipMemset(d_mutex, 0, sizeof(float));

		int numBlocks = calculateNumBlocks(m_data.fluid_data[0].numParticles);
		DFSPH_CFLAdvanced_kernel << < numBlocks, BLOCKSIZE >> > (m_data, out_buff, d_mutex, m_data.fluid_data[0].numParticles);

		hipError_t cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "cuda_cfl failed: %d\n", (int)cudaStatus);
			exit(1598);
		}
		hipFree(d_mutex);
	}
	RealCuda maxVel;
	hipMemcpy(&maxVel, out_buff, sizeof(RealCuda), hipMemcpyDeviceToHost);
	hipFree(out_buff);

	std::chrono::steady_clock::time_point t1 = std::chrono::steady_clock::now();

	RealCuda h = m_data.h;

	// Approximate max. time step size
	h = m_cflFactor * .4 * (2.0*m_data.particleRadius / (sqrt(maxVel)));

	h = min(h, m_cflMaxTimeStepSize);
	h = max(h, minTimeStepSize);

	m_data.updateTimeStep(h);//*/


	std::chrono::steady_clock::time_point t2 = std::chrono::steady_clock::now();



	float time_search = std::chrono::duration_cast<std::chrono::nanoseconds> (t1 - t0).count() / 1000000.0f;
	float time_comp = std::chrono::duration_cast<std::chrono::nanoseconds> (t2 - t1).count() / 1000000.0f;

	printf("Time to do cfl (search,comp): %f    %f\n", time_search, time_comp);
}




//this is the bases for all kernels based function
//I also use that kernel to reset the force

__global__ void DFSPH_updateDynamicObjectParticles_kernel(int numParticles, Vector3d* pos, Vector3d* vel, Vector3d* pos0,
	Vector3d position, Vector3d velocity, Quaternion q, Vector3d angular_vel, Vector3d* F) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= numParticles) { return; }

	//reset the force
	F[i] = Vector3d(0, 0, 0);

	//update location and velocity
	pos[i] = q.rotate(pos0[i]) + position;
	vel[i] = angular_vel.cross(pos[i] - position) + velocity;

}

void update_dynamicObject_UnifiedParticleSet_cuda(SPH::UnifiedParticleSet& particle_set) {


	if (particle_set.is_dynamic_object) {
		int numBlocks = calculateNumBlocks(particle_set.numParticles);


		//update the particle location and velocity
		DFSPH_updateDynamicObjectParticles_kernel << <numBlocks, BLOCKSIZE >> > (particle_set.numParticles,
			particle_set.pos, particle_set.vel, particle_set.pos0,
			particle_set.rigidBody_cpu->position, particle_set.rigidBody_cpu->velocity,
			particle_set.rigidBody_cpu->q, particle_set.rigidBody_cpu->angular_vel,
			particle_set.F);

		//also we can use that time to reset the force buffer
		//directly done in the other kernel
		//DFSPH_setVector3dBufferToZero_kernel << <numBlocks, BLOCKSIZE >> > (container.F, container.numParticles);

		hipError_t cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "update_dynamicObject_UnifiedParticleSet_cuda failed: %d\n", (int)cudaStatus);
			exit(1369);
		}
	}


}



__global__ void compute_dynamic_body_particle_mass_kernel(SPH::DFSPHCData data, SPH::UnifiedParticleSet* particleSet) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= particleSet->numParticles) { return; }

	Real delta = data.W_zero;

	RealCuda radius_sq = data.getKernelRadius();
	Vector3d pos = particleSet->pos[i];
	Vector3d pos_cell = (pos / radius_sq) + data.gridOffset; //on that line the radius is not yet squared
	pos_cell.toFloor();
	int x = pos_cell.x;
	int y = pos_cell.y;
	int z = pos_cell.z;
	radius_sq *= radius_sq;


	//since this version use the std index to be able to iterate on 3 successive cells
	//I can do the -1 at the start on x.
	//one thing: it x=0 then we can only iterate 2 cells at a time
	unsigned int successive_cells_count = (x > 0) ? 3 : 2;
	x = (x > 0) ? x - 1 : x;


	const SPH::UnifiedParticleSet& body = *particleSet;
	for (int k = -1; k < 2; ++k) {
		for (int m = -1; m < 2; ++m) {
			unsigned int cur_cell_id = COMPUTE_CELL_INDEX(x, y + k, z + m);
			unsigned int end = body.neighborsDataSet->cell_start_end[cur_cell_id + successive_cells_count];
			for (unsigned int cur_particle = body.neighborsDataSet->cell_start_end[cur_cell_id]; cur_particle < end; ++cur_particle) {
				unsigned int j = body.neighborsDataSet->p_id_sorted[cur_particle];
				if ((pos - body.pos[j]).squaredNorm() < radius_sq) {
					if (i != j) { delta += KERNEL_W(data,pos - body.pos[j]); }
				}
			}
		}
	}


	const Real volume = 1.0 / delta;
	particleSet->mass[i] = particleSet->density0 * volume;
	particleSet->mass[i] = data.fluid_data_cuda->mass[0];
}



__global__ void refine_dynamic_body_particle_mass_kernel(SPH::DFSPHCData data, SPH::UnifiedParticleSet* particleSet) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= particleSet->numParticles) { return; }

	//the factor is due to the fact that we compensate only a part of the error (proportional to the importance of the mass in the density
	//particleSet->mass[i] += (particleSet->mass[i] * data.W_zero / particleSet->density[i])*(data.density0 - particleSet->density[i]) / (data.W_zero);
	particleSet->mass[i] += (0.3)*(data.density0 - particleSet->density[i]) / (data.W_zero);
}


__global__ void compute_boundaries_density_error_kernel(SPH::DFSPHCData data, SPH::UnifiedParticleSet* particleSet, RealCuda* err, int* err_max) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= particleSet->numParticles) { return; }

	RealCuda density = particleSet->mass[i] * data.W_zero;

	//*
	ITER_NEIGHBORS_INIT(data, particleSet, i);

	//////////////////////////////////////////////////////////////////////////
	// Boundary
	//////////////////////////////////////////////////////////////////////////
	ITER_NEIGHBORS_BOUNDARIES(data, particleSet,
		i,
		density += body.mass[neighborIndex] * KERNEL_W(data,particleSet->pos[i] - body.pos[neighborIndex]);
	);
	//*/
	/*
	RealCuda radius_sq = data.getKernelRadius();
	Vector3d pos_cell = (pos / radius_sq) + data.gridOffset; //on that line the radius is not yet squared
	pos_cell.toFloor();
	int x = pos_cell.x;
	int y = pos_cell.y;
	int z = pos_cell.z;
	radius_sq *= radius_sq;


	//since this version use the std index to be able to iterate on 3 successive cells
	//I can do the -1 at the start on x.
	//one thing: it x=0 then we can only iterate 2 cells at a time
	unsigned int successive_cells_count = (x > 0) ? 3 : 2;
	x = (x > 0) ? x - 1 : x;


	const SPH::UnifiedParticleSet& body = *particleSet;
	for (int k = -1; k < 2; ++k) {
		for (int m = -1; m < 2; ++m) {
			unsigned int cur_cell_id = COMPUTE_CELL_INDEX(x, y + k, z + m);
			unsigned int end = body.neighborsDataSet->cell_start_end[cur_cell_id + successive_cells_count];
			for (unsigned int cur_particle = body.neighborsDataSet->cell_start_end[cur_cell_id]; cur_particle < end; ++cur_particle) {
				unsigned int j = body.neighborsDataSet->p_id_sorted[cur_particle];
				if ((pos - body.pos[j]).squaredNorm() < radius_sq) {
					if (i != j) { density += particleSet->mass[j] * KERNEL_W(data,pos - body.pos[j]); }
				}
			}
		}
	}
	//*/

	particleSet->density[i] = density;
	const RealCuda error =  abs(data.density0-density);
	atomicAdd(err, error);
	atomicMax(err_max, (int)(error*10000));
}

void compute_UnifiedParticleSet_particles_mass_cuda(SPH::DFSPHCData& data, SPH::UnifiedParticleSet& container) {
	int numBlocks = calculateNumBlocks(container.numParticles);
	
	data.destructor_activated = false;

	container.initNeighborsSearchData(data, false, false);
	//init the neighbors
	bool fluid_agg = data.is_fluid_aggregated;
	data.is_fluid_aggregated=false;
	//DFSPH_neighborsSearch_kernel<false> << <numBlocks, BLOCKSIZE >> > (data, data.boundaries_data_cuda);
	data.is_fluid_aggregated = fluid_agg;

	//compute_dynamic_body_particle_mass_kernel << <numBlocks, BLOCKSIZE >> > (data, container.gpu_ptr);
	//gpuErrchk(hipDeviceSynchronize());


	bool refine_masses = false;//This is a test using relaxed jacobi to calculate the true mass of the particle
	if (refine_masses) {


		RealCuda* err;
		int* err_max;
		hipMallocManaged(&(err), sizeof(RealCuda));
		hipMallocManaged(&(err_max), sizeof(int));
		*err = 0.0;
		*err_max = 0;

		//calc the error on the density
		compute_boundaries_density_error_kernel << <numBlocks, BLOCKSIZE >> > (data, container.gpu_ptr, err, err_max);
		gpuErrchk(hipDeviceSynchronize());


		RealCuda target_error = data.density0 / 100.0*0.1;
		RealCuda target_error_max = data.density0 / 100.0*0.1;
		RealCuda avg_err = (*err) / container.numParticles;
		RealCuda err_max_float = (*err_max)/ 10000.0;
		*err = 0;
		*err_max = 0;
		std::cout << "current density error: " << avg_err << " // " << err_max_float << "  target error: " << target_error << std::endl;

		std::chrono::steady_clock::time_point begin= std::chrono::steady_clock::now();

		//and refine it
		//while (avg_err>(target_error)|| target_error_max>15)
		for (int i =0;i<100;++i )
		{
			//refine the values
			refine_dynamic_body_particle_mass_kernel << <numBlocks, BLOCKSIZE >> > (data, container.gpu_ptr);
			gpuErrchk(hipDeviceSynchronize());

			//compute the new error
			compute_boundaries_density_error_kernel << <numBlocks, BLOCKSIZE >> > (data, container.gpu_ptr, err, err_max);
			gpuErrchk(hipDeviceSynchronize());

			avg_err = (*err) / container.numParticles;
			err_max_float = (*err_max)/ 10000.0;
			*err = 0;
			*err_max = 0;
			//std::cout << "current density error: " << avg_err << " // " << err_max_float << "  target error: " << target_error << std::endl;
		}

		std::chrono::steady_clock::time_point end= std::chrono::steady_clock::now();
		float time = std::chrono::duration_cast<std::chrono::nanoseconds> (end - begin).count() / 1000000.0f;
		std::cout << "current density error: " << avg_err << " // " << err_max_float << "  computation_time: " << time << std::endl;


		if (true ) {
			std::string filename = "boundaries density adv.csv";
			std::remove(filename.c_str());
			std::ofstream myfile;
			myfile.open(filename, std::ios_base::app);
			if (myfile.is_open()) {
				for (int i = 0; i < data.boundaries_data->numParticles; ++i) {
					myfile << i << ", " << container.getNumberOfNeighbourgs(i, 0)
						<< ", " << container.getNumberOfNeighbourgs(i, 1)
						<< ", " << container.getNumberOfNeighbourgs(i, 2)
						<< ", " << container.density[i] << std::endl;
				}
				//myfile << total_time / (count_steps + 1) << ", " << m_iterations << ", " << m_iterationsV << std::endl;;
				myfile.close();
			}
			else {
				std::cout << "failed to open file: " << filename << "   reason: " << std::strerror(errno) << std::endl;
			}
		}

		CUDA_FREE_PTR(err);
		CUDA_FREE_PTR(err_max);
	}
	data.destructor_activated = true;
}


