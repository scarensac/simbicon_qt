#include "hip/hip_runtime.h"
#include "DFSPH_memory_management_cuda.h"

#include <stdio.h>
#include <chrono>
#include <iostream>
#include <thread>

#include "DFSPH_define_cuda.h"
#include "DFSPH_macro_cuda.h"
#include "DFSPH_static_variables_structure_cuda.h"


#include "hip/hip_runtime.h"
#include ""
#include "DFSPH_c_arrays_structure.h"
#include "cub.cuh"



#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>


#include "basic_kernels_cuda.cuh"
#include "SPH_other_systems_cuda.h"

namespace MemoryManagementCuda
{
	__global__ void init_buffer_kernel(Vector3d* buff, unsigned int size, Vector3d val) {
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		if (i >= size) { return; }

		buff[i] = val;
	}
}


void allocate_DFSPHCData_base_cuda(SPH::DFSPHCData& data) {
	if (data.damp_planes == NULL) {
		hipMallocManaged(&(data.damp_planes), sizeof(Vector3d) * 10);
	}
	if (data.cancel_wave_planes == NULL) {
		hipMallocManaged(&(data.cancel_wave_planes), sizeof(Vector3i) * 2);
	}

	if (data.bmin == NULL) {

		hipMallocManaged(&(data.bmin), sizeof(Vector3d));
		hipMallocManaged(&(data.bmax), sizeof(Vector3d));
	}

	//alloc static variables
	SVS_CU::get();
}

void free_DFSPHCData_base_cuda(SPH::DFSPHCData& data) {
	CUDA_FREE_PTR(data.damp_planes);
	CUDA_FREE_PTR(data.cancel_wave_planes);


	CUDA_FREE_PTR(data.bmin);
	CUDA_FREE_PTR(data.bmax);

	//free static variables
	SVS_CU::get(true);
}



void allocate_UnifiedParticleSet_cuda(SPH::UnifiedParticleSet& container) {

	read_last_error_cuda("before alloc unified on gpu: ");

	//hipMalloc(&(container.pos), container.numParticles * sizeof(Vector3d)); //use opengl buffer with cuda interop
	//hipMalloc(&(container.vel), container.numParticles * sizeof(Vector3d)); //use opengl buffer with cuda interop
    hipMallocManaged(&(container.mass), container.numParticlesMax * sizeof(RealCuda));


	if (container.has_factor_computation) {
		//*
		hipMallocManaged(&(container.numberOfNeighbourgs), container.numParticlesMax * 3 * sizeof(int));
		hipMallocManaged(&(container.neighbourgs), container.numParticlesMax * MAX_NEIGHBOURS * sizeof(int));

		hipMallocManaged(&(container.density), container.numParticlesMax * sizeof(RealCuda));
		hipMalloc(&(container.factor), container.numParticlesMax * sizeof(RealCuda));
		hipMallocManaged(&(container.densityAdv), container.numParticlesMax * sizeof(RealCuda));

		hipMalloc(&(container.kappa), container.numParticlesMax * sizeof(RealCuda));
		hipMalloc(&(container.kappaV), container.numParticlesMax * sizeof(RealCuda));

		if (container.velocity_impacted_by_fluid_solver) {
            hipMallocManaged(&(container.acc), container.numParticlesMax * sizeof(Vector3d));

#ifdef BENDER2019_BOUNDARIES
			hipMallocManaged(&(container.X_rigids), container.numParticlesMax * sizeof(Vector3d));
			hipMallocManaged(&(container.V_rigids), container.numParticlesMax * sizeof(RealCuda));
#endif

			//I need the allocate the memory cub need to compute the reduction
			//I need the avg pointer because cub require it (but i'll clear after the cub call)
			RealCuda* avg_density_err = SVS_CU::get()->avg_density_err;

			container.d_temp_storage = NULL;
			container.temp_storage_bytes = 0;
			hipcub::DeviceReduce::Sum(container.d_temp_storage, container.temp_storage_bytes,
				container.densityAdv, avg_density_err, container.numParticlesMax);
			// Allocate temporary storage
			hipMalloc(&(container.d_temp_storage), container.temp_storage_bytes);

		}
		//*/

	}

	if (container.is_dynamic_object) {
		hipMalloc(&(container.pos0), container.numParticlesMax * sizeof(Vector3d));
		hipMalloc(&(container.F), container.numParticlesMax * sizeof(Vector3d));
	}

	

	gpuErrchk(hipDeviceSynchronize());
}

void release_UnifiedParticleSet_cuda(SPH::UnifiedParticleSet& container) {
	CUDA_FREE_PTR(container.mass);

	if (container.has_factor_computation) {
		//*
		CUDA_FREE_PTR(container.numberOfNeighbourgs);
		CUDA_FREE_PTR(container.neighbourgs);

		CUDA_FREE_PTR(container.density);
		CUDA_FREE_PTR(container.factor);
		CUDA_FREE_PTR(container.densityAdv);

		CUDA_FREE_PTR(container.kappa);
		CUDA_FREE_PTR(container.kappaV);
		if (container.velocity_impacted_by_fluid_solver) {
			CUDA_FREE_PTR(container.acc);

#ifdef BENDER2019_BOUNDARIES
			CUDA_FREE_PTR(container.V_rigids);
			CUDA_FREE_PTR(container.X_rigids);
#endif

			CUDA_FREE_PTR(container.d_temp_storage);
			container.temp_storage_bytes = 0;
		}
		//*/

	}

	if (container.is_dynamic_object) {
		CUDA_FREE_PTR(container.F);
	}

	//delete the cpu buffers if there are some
	if (container.is_dynamic_object) {
		FREE_PTR(container.F_cpu);
	}
}




void load_UnifiedParticleSet_cuda(SPH::UnifiedParticleSet& container, Vector3d* pos, Vector3d* vel, RealCuda* mass) {

	gpuErrchk(hipMemcpy(container.pos, pos, container.numParticles * sizeof(Vector3d), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(container.vel, vel, container.numParticles * sizeof(Vector3d), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(container.mass, mass, container.numParticles * sizeof(RealCuda), hipMemcpyHostToDevice));

	if (container.is_dynamic_object) {
		int numBlocks = calculateNumBlocks(container.numParticles);
		gpuErrchk(hipMemcpy(container.pos0, pos, container.numParticles * sizeof(Vector3d), hipMemcpyHostToDevice));
		DFSPH_setVector3dBufferToZero_kernel << <numBlocks, BLOCKSIZE >> > (container.F, container.numParticles);
	}

	if (container.has_factor_computation) {

		gpuErrchk(hipMemset(container.kappa, 0, container.numParticles * sizeof(RealCuda)));
		gpuErrchk(hipMemset(container.kappaV, 0, container.numParticles * sizeof(RealCuda)));
		if (container.velocity_impacted_by_fluid_solver) {
		}
	}


}

void read_UnifiedParticleSet_cuda(SPH::UnifiedParticleSet& container, Vector3d* pos, Vector3d* vel, RealCuda* mass, Vector3d* pos0) {
	if (pos != NULL) {
		gpuErrchk(hipMemcpy(pos, container.pos, container.numParticles * sizeof(Vector3d), hipMemcpyDeviceToHost));
	}

	if (vel != NULL) {
		gpuErrchk(hipMemcpy(vel, container.vel, container.numParticles * sizeof(Vector3d), hipMemcpyDeviceToHost));
	}

	if (mass != NULL) {
		gpuErrchk(hipMemcpy(mass, container.mass, container.numParticles * sizeof(RealCuda), hipMemcpyDeviceToHost));
	}

	if (container.is_dynamic_object&&pos0 != NULL) {
		gpuErrchk(hipMemcpy(pos0, container.pos0, container.numParticles * sizeof(Vector3d), hipMemcpyDeviceToHost));
	}
}

void copy_UnifiedParticleSet_cuda(SPH::UnifiedParticleSet& dst, SPH::UnifiedParticleSet& src, bool copy_warmstart) {
	if (dst.numParticles != src.numParticles) {
		std::string err_mess("copy_UnifiedParticleSet_cuda: cannot copy data if the number is not the same in both structures");
		std::cout << err_mess << std::endl;
		throw(err_mess);
	}

	gpuErrchk(hipMemcpy(dst.pos, src.pos, dst.numParticles * sizeof(Vector3d), hipMemcpyDeviceToDevice));
	
	gpuErrchk(hipMemcpy(dst.vel, src.vel, dst.numParticles * sizeof(Vector3d), hipMemcpyDeviceToDevice));

	gpuErrchk(hipMemcpy(dst.mass, src.mass, dst.numParticles * sizeof(RealCuda), hipMemcpyDeviceToDevice));

	gpuErrchk(hipMemcpy(dst.color, src.color, dst.numParticles * sizeof(Vector3d), hipMemcpyDeviceToDevice));	
}

void read_rigid_body_force_cuda(SPH::UnifiedParticleSet& container) {
	if (container.is_dynamic_object) {
		if (container.F_cpu == NULL) {
			container.F_cpu = new Vector3d[container.numParticles];
		}

		gpuErrchk(hipMemcpy(container.F_cpu, container.F, container.numParticles * sizeof(Vector3d), hipMemcpyDeviceToHost));
	}
}



__global__ void compute_fluid_impact_on_dynamic_body_kernel(SPH::UnifiedParticleSet* container, Vector3d rb_position,
	Vector3d* force, Vector3d* moment, Vector3d reduction_factor) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= container->numParticles) { return; }

	Vector3d F, M;

	F = container->F[i];
	F *= reduction_factor;
	M = (container->pos[i] - rb_position).cross(F);

	atomicAdd(&(force->x), F.x);
	atomicAdd(&(force->y), F.y);
	atomicAdd(&(force->z), F.z);
	atomicAdd(&(moment->x), M.x);
	atomicAdd(&(moment->y), M.y);
	atomicAdd(&(moment->z), M.z);
}

void compute_fluid_impact_on_dynamic_body_cuda(SPH::UnifiedParticleSet& container, Vector3d& force, Vector3d& moment,
	const Vector3d& reduction_factor) {
	Vector3d* force_cuda = SVS_CU::get()->force_cuda;
	Vector3d* moment_cuda = SVS_CU::get()->moment_cuda;
	*force_cuda = Vector3d(0, 0, 0);
	*moment_cuda = Vector3d(0, 0, 0);


	int numBlocks = calculateNumBlocks(container.numParticles);
	compute_fluid_impact_on_dynamic_body_kernel << <numBlocks, BLOCKSIZE >> > (container.gpu_ptr,
		container.rigidBody_cpu->position, force_cuda,
		moment_cuda, reduction_factor);
	gpuErrchk(hipDeviceSynchronize());

	force = *force_cuda;
	moment = *moment_cuda;
}





__global__ void compute_fluid_boyancy_on_dynamic_body_kernel(SPH::UnifiedParticleSet* container, Vector3d* force, Vector3d* pt_appli) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= container->numParticles) { return; }



	//I use the abs just in case for some reason the vertical force is negative ...
	//By this I mena that the y component also contains the y component of the drag. but there
	//is no way to extract the actual boyancy, soand approximation will have to do
	RealCuda boyancy = container->F[i].y;
	RealCuda boyancy_abs = abs(boyancy);
	if (boyancy_abs>0) {
		Vector3d pt = container->pos[i] * boyancy_abs;

		//in the x componant I'll store the total abs
		atomicAdd(&(force->x), boyancy_abs);
		atomicAdd(&(force->y), boyancy);
		atomicAdd(&(pt_appli->x), pt.x);
		atomicAdd(&(pt_appli->y), pt.y);
		atomicAdd(&(pt_appli->z), pt.z);
	}
}

void compute_fluid_Boyancy_on_dynamic_body_cuda(SPH::UnifiedParticleSet& container, Vector3d& force, Vector3d& pt_appli) {
	Vector3d* force_cuda = SVS_CU::get()->force_cuda;
	Vector3d* pt_cuda = SVS_CU::get()->pt_cuda;
	*force_cuda = Vector3d(0, 0, 0);
	*pt_cuda = Vector3d(0, 0, 0);

	int numBlocks = calculateNumBlocks(container.numParticles);
	compute_fluid_boyancy_on_dynamic_body_kernel << <numBlocks, BLOCKSIZE >> > (container.gpu_ptr, force_cuda, pt_cuda);
	gpuErrchk(hipDeviceSynchronize());

	force = *force_cuda;
	//if the sum of the force is non zero
	if (abs(force.y)>0) {
		pt_appli = *pt_cuda;

		//now compute the avg to get the actual point
		pt_appli = pt_appli / force.x;
		//and clear the x component
		force.x = 0;
	}
	else {
		force = Vector3d(0, 0, 0);
		pt_appli = Vector3d(0, 0, 0);
	}
}




void allocate_and_copy_UnifiedParticleSet_vector_cuda(SPH::UnifiedParticleSet** out_vector, SPH::UnifiedParticleSet* in_vector, int numSets) {

	gpuErrchk(hipMalloc(out_vector, numSets * sizeof(SPH::UnifiedParticleSet)));

	//now set the gpu_ptr in eahc object so that it points to the right place
	for (int i = 0; i < numSets; ++i) {
		in_vector[i].gpu_ptr = *out_vector + i;
	}

	//before being able to fill the gpu array we need to make a copy of the data structure since
	//we will have to change the neighborsdataset from the cpu to the gpu
	//*
	SPH::UnifiedParticleSet* temp;
	temp = new SPH::UnifiedParticleSet[numSets];
	std::copy(in_vector, in_vector + numSets, temp);

	for (int i = 0; i < numSets; ++i) {
		SPH::UnifiedParticleSet& body = temp[i];

		//we need to toggle the flag that prevent the destructor from beeing called on release
		//since it's the cpu version that clear the memory buffers that are common to the two structures
		body.releaseDataOnDestruction = false;

		//the gpu unified particle set has a irect pointer to the gpu neughbor dataset
		body.neighborsDataSet = body.neighborsDataSet->gpu_ptr;
	}
	//*/

	gpuErrchk(hipMemcpy(*out_vector, temp, numSets * sizeof(SPH::UnifiedParticleSet), hipMemcpyHostToDevice));

	//Now I have to update the pointer of the cpu set so that it point to the gpu structure
	delete[] temp;

}




void allocate_grouped_neighbors_struct_cuda(SPH::DFSPHCData& data) {
	std::cout << "initialising aggregated structure" << std::endl;

	if (data.numDynamicBodies < 1) {
		std::cout << "no dynamic bodies detected" << std::endl;

		//ok so I need the grouped buffer because I reuse it for external forces computation
		//note if I want to rmv that restriction I only have to use the one 
		//TODO apply that modification
		hipMalloc(&(data.posBufferGroupedDynamicBodies), data.fluid_data->numParticlesMax * sizeof(Vector3d));

		return;
	}

	if (data.neighborsDataSetGroupedDynamicBodies != NULL || data.posBufferGroupedDynamicBodies != NULL) {
		throw("allocate_grouped_neighbors_struct_cuda already allocated");
	}

	int numParticles = 0;
	int numParticlesMax = 0;
	if (data.is_fluid_aggregated) {
		numParticles += data.fluid_data->numParticles;
		numParticlesMax += data.fluid_data->numParticlesMax;
	}

	for (int i = 0; i<data.numDynamicBodies; ++i) {
		numParticles += data.vector_dynamic_bodies_data[i].numParticles;
		numParticlesMax += data.vector_dynamic_bodies_data[i].numParticlesMax;
	}

	//allocate the dataset
	data.neighborsDataSetGroupedDynamicBodies = new SPH::NeighborsSearchDataSet(numParticles, numParticlesMax);

	//read gpu ptr
	data.neighborsDataSetGroupedDynamicBodies_cuda = data.neighborsDataSetGroupedDynamicBodies->gpu_ptr;


	//now it's like the normal neighbor search excapt that we have to iterate on all the solid particles
	//instead of just one buffer
	//the easiest way is to build a new pos array that contains all the dynamic particles
	hipMalloc(&(data.posBufferGroupedDynamicBodies), numParticlesMax * sizeof(Vector3d));

}


void free_grouped_neighbors_struct_cuda(SPH::DFSPHCData& data) {
	FREE_PTR(data.neighborsDataSetGroupedDynamicBodies);

	CUDA_FREE_PTR(data.posBufferGroupedDynamicBodies);
}


void update_neighborsSearchBuffers_UnifiedParticleSet_vector_cuda(SPH::UnifiedParticleSet** out_vector, SPH::UnifiedParticleSet* in_vector, int numSets) {
	SPH::UnifiedParticleSet* temp;
	temp = new SPH::UnifiedParticleSet[numSets];

	gpuErrchk(hipMemcpy(temp, *out_vector, numSets * sizeof(SPH::UnifiedParticleSet), hipMemcpyDeviceToHost));

	for (int i = 0; i < numSets; ++i) {
		SPH::UnifiedParticleSet& body = temp[i];

		//we need to toggle the flag that prevent the destructor from beeing called on release
		//since it's the cpu version that clear the memory buffers that are common to the two structures
		body.releaseDataOnDestruction = false;

		//update the neighbor dataset to the cpu
		gpuErrchk(hipMemcpy(body.neighborsDataSet, in_vector[i].neighborsDataSet,
			sizeof(SPH::NeighborsSearchDataSet), hipMemcpyHostToDevice));

	}

	gpuErrchk(hipMemcpy(*out_vector, temp, numSets * sizeof(SPH::UnifiedParticleSet), hipMemcpyHostToDevice));


	delete[] temp;
}




void release_UnifiedParticleSet_vector_cuda(SPH::UnifiedParticleSet** vector, int numSets) {
	//each stucture properly clear itself currently so here i just need to destroy the high level array
	CUDA_FREE_PTR((*vector));

	hipDeviceSynchronize();

}


template<class T> __global__ void cuda_updateParticleCount_kernel(T* container, unsigned int numParticles) {
	//that kernel wil only ever use one thread so I sould noteven need that
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= 1) { return; }

	container->numParticles = numParticles;
}


template<class T> void update_active_particle_number_cuda(T& container) {
	//And now I need to update the particle count in the gpu structures
	//the easiest way is to use a kernel with just one thread used
	//the other way would be to copy the data back to the cpu then update the value before sending it back to the cpu
	cuda_updateParticleCount_kernel<T> << <1, 1 >> > (container.gpu_ptr, container.numParticles);

	gpuErrchk(hipDeviceSynchronize());
}
template void update_active_particle_number_cuda<SPH::UnifiedParticleSet>(SPH::UnifiedParticleSet& container);
template void update_active_particle_number_cuda<SPH::NeighborsSearchDataSet>(SPH::NeighborsSearchDataSet& container);

void change_fluid_max_particle_number(SPH::DFSPHCData& data, int numParticlesMax) {
	//update the fluid dataset
	data.fluid_data[0].changeMaxParticleNumber(numParticlesMax);

	//and update the aggregated neighbor search if it is used
#ifdef GROUP_DYNAMIC_BODIES_NEIGHBORS_SEARCH
	free_grouped_neighbors_struct_cuda(data);

	allocate_grouped_neighbors_struct_cuda(data);
#endif


}


void change_max_particle_number(SPH::UnifiedParticleSet& container, int numParticlesMax) {
	//we need to copy the existing data so let's start in order.
	//first disactivate the destructor
	bool old_release_on_destruct = container.releaseDataOnDestruction;
	container.releaseDataOnDestruction = false;

	//the easy way is to create a dummy, backup the existing buffers inside
	//alloc the new buffers
	//do the copy
	//delete the temps storage

	SPH::UnifiedParticleSet dummy;
	dummy = container;
	//remove the pointer on the neighbor search data since it is handled separately
	dummy.neighborsDataSet = NULL;
	dummy.gpu_ptr = NULL;


	//*
	//now change the number of particle
	container.numParticlesMax = numParticlesMax;

	//the rendering data

	//allocate new
	container.renderingData = new ParticleSetRenderingData();
	cuda_opengl_initParticleRendering(*container.renderingData, numParticlesMax, &container.pos, &container.vel, container.has_color_buffer, &container.color);

	//now we need to copy the data
	gpuErrchk(hipMemcpy(container.pos, dummy.pos, dummy.numParticles * sizeof(Vector3d), hipMemcpyDeviceToDevice));
	gpuErrchk(hipMemcpy(container.vel, dummy.vel, dummy.numParticles * sizeof(Vector3d), hipMemcpyDeviceToDevice));

	//and the rest of the data
	allocate_UnifiedParticleSet_cuda(container);

	//and fill it with the old data for buffers that need to be kept
	gpuErrchk(hipMemcpy(container.mass, dummy.mass, dummy.numParticles * sizeof(RealCuda), hipMemcpyDeviceToDevice));
	gpuErrchk(hipMemcpy(container.kappa, dummy.kappa, dummy.numParticles * sizeof(RealCuda), hipMemcpyDeviceToDevice));
	gpuErrchk(hipMemcpy(container.kappaV, dummy.kappaV, dummy.numParticles * sizeof(RealCuda), hipMemcpyDeviceToDevice));

	//*/
	//finaly we need to update the data that is stored on the gpu
	//hopefully ne need for more allocation I can just copy the new ptr addresses
	SPH::NeighborsSearchDataSet* back_up = container.neighborsDataSet;
	container.neighborsDataSet = container.neighborsDataSet->gpu_ptr;
	gpuErrchk(hipMemcpy(container.gpu_ptr, &container, sizeof(SPH::UnifiedParticleSet), hipMemcpyHostToDevice));
	container.neighborsDataSet = back_up;
	//and free the data from the dummy
	dummy.clear();


	container.releaseDataOnDestruction = old_release_on_destruct;
}


void change_max_particle_number(SPH::NeighborsSearchDataSet& dataSet, int numParticlesMax) {
	if (!dataSet.internal_buffers_allocated) {
		throw("only consider fully allocated dataset for now");
	}


	release_neighbors_search_data_set(dataSet, false, true, true);

	//and allocate it back
	dataSet.numParticlesMax = numParticlesMax;

	allocate_neighbors_search_data_set(dataSet, false, true, false);
}



void add_particles_cuda(SPH::UnifiedParticleSet& container, int num_additional_particles, const Vector3d* pos, const Vector3d* vel) {
	//can't use memeset for the mass so I have to make a kernel for the set
	int numBlocks = calculateNumBlocks(num_additional_particles);
	cuda_setBufferToValue_kernel<RealCuda> << <numBlocks, BLOCKSIZE >> > (container.mass,
		container.m_V*container.density0, container.numParticles + num_additional_particles);



	gpuErrchk(hipMemcpy(container.pos + container.numParticles, pos, num_additional_particles * sizeof(Vector3d), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(container.vel + container.numParticles, vel, num_additional_particles * sizeof(Vector3d), hipMemcpyHostToDevice));


	gpuErrchk(hipMemset(container.kappa + container.numParticles, 0, num_additional_particles * sizeof(RealCuda)));
	gpuErrchk(hipMemset(container.kappaV + container.numParticles, 0, num_additional_particles * sizeof(RealCuda)));

	//update the particle count
	container.updateActiveParticleNumber(container.numParticles + num_additional_particles);


	hipError_t cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		std::cerr << "add_particles_cuda failed: " << (int)cudaStatus << std::endl;
		exit(1598);
	}


}

template<class T> void set_buffer_to_value(T* buff, T val, int size) {
	//can't use memeset for the mass so I have to make a kernel for the  set
	int numBlocks = calculateNumBlocks(size);
	cuda_setBufferToValue_kernel<T> << <numBlocks, BLOCKSIZE >> > (buff, val, size);

	hipError_t cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		std::cerr << "set_buffer_to_value failed: " << (int)cudaStatus << std::endl;
		exit(1598);
	}
}
template void set_buffer_to_value<Vector3d>(Vector3d* buff, Vector3d val, int size);
template void set_buffer_to_value<int>(int* buff, int val, int size);


void allocate_precomputed_kernel_managed(SPH::PrecomputedCubicKernelPerso& kernel, bool minimize_managed) {

	if (minimize_managed) {
		hipMalloc(&(kernel.m_W), kernel.m_resolution * sizeof(RealCuda));
		hipMalloc(&(kernel.m_gradW), (kernel.m_resolution + 1) * sizeof(RealCuda));
	}
	else {
		fprintf(stderr, "trying to use managed buffers for the kernels\n");
		exit(1256);
		//hipMallocManaged(&(kernel.m_W), kernel.m_resolution * sizeof(RealCuda));
		//hipMallocManaged(&(kernel.m_gradW), (kernel.m_resolution + 1) * sizeof(RealCuda));
	}
}

void free_precomputed_kernel_managed(SPH::PrecomputedCubicKernelPerso& kernel) {
	CUDA_FREE_PTR(kernel.m_W);
	CUDA_FREE_PTR(kernel.m_gradW);
}


void init_precomputed_kernel_from_values(SPH::PrecomputedCubicKernelPerso& kernel, RealCuda* w, RealCuda* grad_W) {
	hipError_t cudaStatus;
	//W
	cudaStatus = hipMemcpy(kernel.m_W,
		w,
		kernel.m_resolution * sizeof(RealCuda),
		hipMemcpyHostToDevice);

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "precomputed initialization of W from data failed: %d\n", (int)cudaStatus);
		exit(1598);
	}

	//grad W
	cudaStatus = hipMemcpy(kernel.m_gradW,
		grad_W,
		(kernel.m_resolution + 1) * sizeof(RealCuda),
		hipMemcpyHostToDevice);

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "precomputed initialization of grad W from data failed: %d\n", (int)cudaStatus);
		exit(1598);
	}

}




void allocate_neighbors_search_data_set(SPH::NeighborsSearchDataSet& dataSet, bool result_buffers_only, bool particle_related_only,
	bool allocate_gpu) {

	//result buffers
	hipMalloc(&(dataSet.p_id_sorted), dataSet.numParticlesMax * sizeof(unsigned int));
	if (!particle_related_only) {
		hipMalloc(&(dataSet.cell_start_end), (CELL_COUNT + 1) * sizeof(unsigned int));
	}

	//allocate the mem for fluid particles
	if (!result_buffers_only) {
		hipMallocManaged(&(dataSet.cell_id), dataSet.numParticlesMax * sizeof(unsigned int));
		hipMalloc(&(dataSet.cell_id_sorted), dataSet.numParticlesMax * sizeof(unsigned int));
		hipMalloc(&(dataSet.local_id), dataSet.numParticlesMax * sizeof(unsigned int));
		hipMalloc(&(dataSet.p_id), dataSet.numParticlesMax * sizeof(unsigned int));

		hipMalloc(&(dataSet.intermediate_buffer_v3d), dataSet.numParticlesMax * sizeof(Vector3d));
		hipMalloc(&(dataSet.intermediate_buffer_real), dataSet.numParticlesMax * sizeof(RealCuda));

		//reset the particle id
		int numBlocks = calculateNumBlocks(dataSet.numParticlesMax);
		DFSPH_setBufferValueToItself_kernel << <numBlocks, BLOCKSIZE >> > (dataSet.p_id, dataSet.numParticlesMax);
		DFSPH_setBufferValueToItself_kernel << <numBlocks, BLOCKSIZE >> > (dataSet.p_id_sorted, dataSet.numParticlesMax);
		gpuErrchk(hipDeviceSynchronize());


		//cub pair sort
		dataSet.temp_storage_bytes_pair_sort = 0;
		dataSet.d_temp_storage_pair_sort = NULL;
		hipcub::DeviceRadixSort::SortPairs(dataSet.d_temp_storage_pair_sort, dataSet.temp_storage_bytes_pair_sort,
			dataSet.cell_id, dataSet.cell_id_sorted, dataSet.p_id, dataSet.p_id_sorted, dataSet.numParticlesMax);
		gpuErrchk(hipDeviceSynchronize());
		hipMalloc(&(dataSet.d_temp_storage_pair_sort), dataSet.temp_storage_bytes_pair_sort);


		if (!particle_related_only) {
			hipMallocManaged(&(dataSet.hist), (CELL_COUNT + 1) * sizeof(unsigned int));

			//cub histogram
			dataSet.temp_storage_bytes_cumul_hist = 0;
			dataSet.d_temp_storage_cumul_hist = NULL;
			hipcub::DeviceScan::ExclusiveSum(dataSet.d_temp_storage_cumul_hist, dataSet.temp_storage_bytes_cumul_hist,
				dataSet.hist, dataSet.cell_start_end, (CELL_COUNT + 1));
			gpuErrchk(hipDeviceSynchronize());
			hipMalloc(&(dataSet.d_temp_storage_cumul_hist), dataSet.temp_storage_bytes_cumul_hist);
		}
	}


	/*
	std::cout << "neighbors struct num byte allocated cub (numParticlesMax pair_sort cumul_hist)" << dataSet.numParticlesMax << "  " <<
	dataSet.temp_storage_bytes_pair_sort << "  " << dataSet.temp_storage_bytes_cumul_hist << std::endl;
	//*/


	dataSet.internal_buffers_allocated = true;

	if (allocate_gpu) {
		//duplicate the neighbor dataset to the gpu
		gpuErrchk(hipMalloc(&(dataSet.gpu_ptr), sizeof(SPH::NeighborsSearchDataSet)));
	}
	//and copy the gpu data if the buffer exists
	if (dataSet.gpu_ptr != NULL) {
		gpuErrchk(hipMemcpy(dataSet.gpu_ptr, &dataSet,
			sizeof(SPH::NeighborsSearchDataSet), hipMemcpyHostToDevice));
	}

	gpuErrchk(hipDeviceSynchronize());

}



void release_neighbors_search_data_set(SPH::NeighborsSearchDataSet& dataSet, bool keep_result_buffers, bool keep_grid_related,
	bool keep_gpu) {

	//allocatethe mme for fluid particles
	CUDA_FREE_PTR(dataSet.cell_id);
	CUDA_FREE_PTR(dataSet.local_id);
	CUDA_FREE_PTR(dataSet.p_id);
	CUDA_FREE_PTR(dataSet.cell_id_sorted);

	CUDA_FREE_PTR(dataSet.d_temp_storage_pair_sort);
	dataSet.temp_storage_bytes_pair_sort = 0;

	CUDA_FREE_PTR(dataSet.intermediate_buffer_v3d);
	CUDA_FREE_PTR(dataSet.intermediate_buffer_real);

	if (!keep_grid_related) {
		CUDA_FREE_PTR(dataSet.hist);
		CUDA_FREE_PTR(dataSet.d_temp_storage_cumul_hist);
		dataSet.temp_storage_bytes_cumul_hist = 0;
	}

	dataSet.internal_buffers_allocated = false;

	if (!keep_result_buffers) {
		CUDA_FREE_PTR(dataSet.p_id_sorted);

		if (!keep_grid_related) {
			CUDA_FREE_PTR(dataSet.cell_start_end);
		}
	}

	if (!keep_gpu) {
		CUDA_FREE_PTR(dataSet.gpu_ptr);
	}
}



void load_bender2019_boundaries_from_cpu(SPH::UnifiedParticleSet& container, RealCuda* V_rigids_i, Vector3d* X_rigids_i) {
	
	//for some reason his returns invalid argument
	gpuErrchk(hipMemcpy(container.X_rigids, X_rigids_i, container.numParticles * sizeof(Vector3d), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(container.V_rigids, V_rigids_i, container.numParticles * sizeof(RealCuda), hipMemcpyHostToDevice));
	
}
