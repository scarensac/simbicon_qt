#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "DFSPH_cuda_basic.h"
#include <stdio.h>
#include "DFSPH_c_arrays_structure.h"
#include "cub.cuh"
#include <chrono>
#include <iostream>
#include <thread>

//#define SHOW_MESSAGES_IN_CUDA_FUNCTIONS

#define BLOCKSIZE 256
#define m_eps 1.0e-5
#define CELL_ROW_LENGTH 256
#define CELL_COUNT CELL_ROW_LENGTH*CELL_ROW_LENGTH*CELL_ROW_LENGTH

#define USE_WARMSTART
#define USE_WARMSTART_V

#define BITSHIFT_INDEX_DYNAMIC_BODIES

#ifdef BITSHIFT_INDEX_DYNAMIC_BODIES
#define WRITTE_DYNAMIC_BODIES_PARTICLES_INDEX(body_index,particle_index) WRITTE_DYNAMIC_BODIES_PARTICLES_INDEX_BITSHIFT(body_index,particle_index)
#define READ_DYNAMIC_BODIES_PARTICLES_INDEX(neighbors_ptr,body_index,particle_index) READ_DYNAMIC_BODIES_PARTICLES_INDEX_BITSHIFT(neighbors_ptr,body_index,particle_index)
#else
#define WRITTE_DYNAMIC_BODIES_PARTICLES_INDEX(body_index,particle_index) WRITTE_DYNAMIC_BODIES_PARTICLES_INDEX_ADDITION(body_index,particle_index)
#define READ_DYNAMIC_BODIES_PARTICLES_INDEX(neighbors_ptr,body_index,particle_index) READ_DYNAMIC_BODIES_PARTICLES_INDEX_ADDITION(neighbors_ptr,body_index,particle_index)
#endif

//those defines are to create and read the dynamic bodies indexes
#define WRITTE_DYNAMIC_BODIES_PARTICLES_INDEX_BITSHIFT(body_index,particle_index)  body_index + (particle_index << 0x8)
#define WRITTE_DYNAMIC_BODIES_PARTICLES_INDEX_ADDITION(body_index,particle_index)  particle_index + (body_index * 1000000)

//WARNING his one declare the body/particle index by itself
//you just have to give it the variable name you want
#define READ_DYNAMIC_BODIES_PARTICLES_INDEX_BITSHIFT(neighbors_ptr, body_index,particle_index)  \
    const unsigned int identifier = *neighbors_ptr++;\
    const unsigned int particle_index = identifier >> 0x8;\
    const unsigned int body_index = identifier & 0xFF;

#define READ_DYNAMIC_BODIES_PARTICLES_INDEX_ADDITION(neighbors_ptr, body_index,particle_index)   \
    const unsigned int identifier = *neighbors_ptr++;\
    const unsigned int particle_index = identifier % (1000000);\
    const unsigned int body_index=identifier / 1000000;

#define ITER_NEIGHBORS_INIT(index) int* neighbors_ptr = particleSet->getNeighboursPtr(index); int* end_ptr = neighbors_ptr;

#define ITER_NEIGHBORS_FLUID(index,code){\
    end_ptr += particleSet->getNumberOfNeighbourgs(index);\
    const SPH::UnifiedParticleSet& body = *(m_data.fluid_data_cuda);\
    while (neighbors_ptr != end_ptr)\
{\
    const unsigned int neighborIndex = *neighbors_ptr++;\
    code;\
    }\
    }


#define ITER_NEIGHBORS_BOUNDARIES(index,code){\
    const SPH::UnifiedParticleSet& body = *(m_data.boundaries_data_cuda);\
    end_ptr += particleSet->getNumberOfNeighbourgs(index, 1);\
    while (neighbors_ptr != end_ptr)\
{\
    const unsigned int neighborIndex = *neighbors_ptr++;\
    code; \
    }\
    }


#define ITER_NEIGHBORS_SOLIDS(index,code){\
    end_ptr += particleSet->getNumberOfNeighbourgs(index, 2);\
    while (neighbors_ptr != end_ptr)\
{\
    READ_DYNAMIC_BODIES_PARTICLES_INDEX(neighbors_ptr, bodyIndex, neighborIndex);\
    const SPH::UnifiedParticleSet& body = m_data.vector_dynamic_bodies_data_cuda[bodyIndex];\
    code; \
    }\
    }

//using norton bitshift for the cells is slower than using a normal index, not that much though
//#define BITSHIFT_INDEX_NEIGHBORS_CELL
//#define USE_COMPLETE


#ifdef BITSHIFT_INDEX_NEIGHBORS_CELL

#ifndef USE_COMPLETE
#define USE_COMPLETE
#endif

__device__ void interleave_2_bits_magic_numbers(unsigned int& x) {
    x = (x | (x << 16)) & 0x030000FF;
    x = (x | (x << 8)) & 0x0300F00F;
    x = (x | (x << 4)) & 0x030C30C3;
    x = (x | (x << 2)) & 0x09249249;
}
__device__ unsigned int compute_morton_magic_numbers(unsigned int x, unsigned int y, unsigned int z) {
    interleave_2_bits_magic_numbers(x);
    interleave_2_bits_magic_numbers(y);
    interleave_2_bits_magic_numbers(z);

    return x | (y << 1) | (z << 2);
}

#define COMPUTE_CELL_INDEX(x,y,z) compute_morton_magic_numbers(x,y,z)

#else
#define COMPUTE_CELL_INDEX(x,y,z) (x)+(z)*CELL_ROW_LENGTH+(y)*CELL_ROW_LENGTH*CELL_ROW_LENGTH
#endif



//those two variables are the identifiers that  link the ongle buffers to cuda
//hipGraphicsResource_t vboRes_pos;
//hipGraphicsResource_t vboRes_vel;

//easy function to check errors
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: error %d: %s %s %d\n", (int)code, hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

/*
//this is the bases for all kernels based function
__global__ void DFSPH__kernel(SPH::DFSPHCData m_data) {
int i = blockIdx.x * blockDim.x + threadIdx.x;
if (i >= m_data.numFluidParticles) { return; }

}
void cuda_(SPH::DFSPHCData& data) {
int numBlocks = (data.numFluidParticles + BLOCKSIZE - 1) / BLOCKSIZE;
DFSPH__kernel << <numBlocks, BLOCKSIZE >> > (data);

hipError_t cudaStatus = hipDeviceSynchronize();
if (cudaStatus != hipSuccess) {
fprintf(stderr, "cuda_compute_density failed: %d\n", (int)cudaStatus);
exit(1598);
}
}
//*/

FUNCTION inline int* getNeighboursPtr(int * neighbourgs, int particle_id) {
    //	return neighbourgs + body_id*numFluidParticles*MAX_NEIGHBOURS + particle_id*MAX_NEIGHBOURS;
    return neighbourgs + particle_id*MAX_NEIGHBOURS;
}

FUNCTION inline unsigned int getNumberOfNeighbourgs(int* numberOfNeighbourgs, int particle_id, int body_id = 0) {
    //return numberOfNeighbourgs[body_id*numFluidParticles + particle_id];
    return numberOfNeighbourgs[particle_id * 3 + body_id];
}

__global__ void get_min_max_pos_kernel(SPH::UnifiedParticleSet* particleSet, Vector3d* min_o, Vector3d *max_o, RealCuda particle_radius) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= 1) { return; }

    //the problem I have is that there wont be a particle in the exact corner
    //I I'll iter on some particles to be sure to reach smth near the corner
    Vector3d min = particleSet->pos[0];
    Vector3d max = particleSet->pos[particleSet->numParticles - 1];

    for (int k = 0; k < 10; ++k) {
        Vector3d p_min = particleSet->pos[k];
        Vector3d p_max = particleSet->pos[particleSet->numParticles - (1+k)];

        if (min.x > p_min.x) { min.x = p_min.x; }
        if (min.y > p_min.y) { min.y = p_min.y; }
        if (min.z > p_min.z) { min.z = p_min.z; }

        if (max.x < p_max.x) { max.x = p_max.x; }
        if (max.y < p_max.y) { max.y = p_max.y; }
        if (max.z < p_max.z) { max.z = p_max.z; }
    }

    min += 2*particle_radius;
    max -= 2*particle_radius;

    *min_o = min;
    *max_o = max;
}

__device__ void computeDensityChange(const SPH::DFSPHCData& m_data, SPH::UnifiedParticleSet* particleSet, const unsigned int index) {
    unsigned int numNeighbors = particleSet->getNumberOfNeighbourgs(index);
    // in case of particle deficiency do not perform a divergence solve
    if (numNeighbors < 20) {
        for (unsigned int pid = 1; pid < 3; pid++)
        {
            numNeighbors += particleSet->getNumberOfNeighbourgs(index, pid);
        }
    }
    if (numNeighbors < 20) {
        particleSet->densityAdv[index] = 0;
    }
    else {
        RealCuda densityAdv = 0;
        const Vector3d &xi = particleSet->pos[index];
        const Vector3d &vi = particleSet->vel[index];
        //////////////////////////////////////////////////////////////////////////
        // Fluid
        //////////////////////////////////////////////////////////////////////////
        ITER_NEIGHBORS_INIT(index);

        ITER_NEIGHBORS_FLUID(
                    index,
                    densityAdv += body.mass[neighborIndex] * (vi - body.vel[neighborIndex]).dot(m_data.gradW(xi - body.pos[neighborIndex]));
                );
        //////////////////////////////////////////////////////////////////////////
        // Boundary
        //////////////////////////////////////////////////////////////////////////
        ITER_NEIGHBORS_BOUNDARIES(
                    index,
                    densityAdv += body.mass[neighborIndex] * (vi - body.vel[neighborIndex]).dot(m_data.gradW(xi - body.pos[neighborIndex]));
                );

        //////////////////////////////////////////////////////////////////////////
        // Dynamic Bodies
        //////////////////////////////////////////////////////////////////////////
        ITER_NEIGHBORS_SOLIDS(
                    index,
                    densityAdv += body.mass[neighborIndex] * (vi - body.vel[neighborIndex]).dot(m_data.gradW(xi - body.pos[neighborIndex]));
                );

        // only correct positive divergence
        particleSet->densityAdv[index] = MAX_MACRO_CUDA(densityAdv, 0.0);
    }
}


template <bool warm_start> __device__ void divergenceSolveParticle(SPH::DFSPHCData& m_data, SPH::UnifiedParticleSet* particleSet, const unsigned int i) {
    Vector3d v_i = Vector3d(0, 0, 0);
    //////////////////////////////////////////////////////////////////////////
    // Evaluate rhs
    //////////////////////////////////////////////////////////////////////////
    const RealCuda ki = (warm_start) ? particleSet->kappaV[i] : (particleSet->densityAdv[i])*particleSet->factor[i];

#ifdef USE_WARMSTART_V
    if (!warm_start) { particleSet->kappaV[i] += ki; }
#endif

    const Vector3d &xi = particleSet->pos[i];


    //////////////////////////////////////////////////////////////////////////
    // Fluid
    //////////////////////////////////////////////////////////////////////////
    ITER_NEIGHBORS_INIT(i);

    ITER_NEIGHBORS_FLUID(
                i,
                const RealCuda kSum = (ki + ((warm_start) ? body.kappaV[neighborIndex] : (body.densityAdv[neighborIndex])*body.factor[neighborIndex]));
            if (fabs(kSum) > m_eps)
    {
        // ki, kj already contain inverse density
        v_i += kSum *  body.mass[neighborIndex] * m_data.gradW(xi - body.pos[neighborIndex]);
    }
    );


    if (fabs(ki) > m_eps)
    {
        //////////////////////////////////////////////////////////////////////////
        // Boundary
        //////////////////////////////////////////////////////////////////////////
        ITER_NEIGHBORS_BOUNDARIES(
                    i,
                    const Vector3d delta = ki * body.mass[neighborIndex] * m_data.gradW(xi - body.pos[neighborIndex]);
                v_i += delta;// ki already contains inverse density
        );


        //////////////////////////////////////////////////////////////////////////
        // Dynamic bodies
        //////////////////////////////////////////////////////////////////////////

        ITER_NEIGHBORS_SOLIDS(
                    i,
                    Vector3d delta = ki * body.mass[neighborIndex] * m_data.gradW(xi - body.pos[neighborIndex]);
                v_i += delta;// ki already contains inverse density

        //we apply the force to the body particle (no invH since it has been fatorized at the end)
        delta *= -particleSet->mass[i];
        atomicAdd(&(body.F[neighborIndex].x), delta.x);
        atomicAdd(&(body.F[neighborIndex].y), delta.y);
        atomicAdd(&(body.F[neighborIndex].z), delta.z);
        );
    }

    particleSet->vel[i] += v_i*m_data.h;
}

__device__ void computeDensityAdv(SPH::DFSPHCData& m_data, SPH::UnifiedParticleSet* particleSet, const unsigned int index) {
    const Vector3d xi = particleSet->pos[index];
    const Vector3d vi = particleSet->vel[index];
    RealCuda delta = 0;


    //////////////////////////////////////////////////////////////////////////
    // Fluid
    //////////////////////////////////////////////////////////////////////////
    ITER_NEIGHBORS_INIT(index);

    ITER_NEIGHBORS_FLUID(
                index,
                delta += body.mass[neighborIndex] * (vi - body.vel[neighborIndex]).dot(m_data.gradW(xi - body.pos[neighborIndex]));
            );

    //////////////////////////////////////////////////////////////////////////
    // Boundary
    //////////////////////////////////////////////////////////////////////////
    ITER_NEIGHBORS_BOUNDARIES(
                index,
                delta += body.mass[neighborIndex] * (vi - body.vel[neighborIndex]).dot(m_data.gradW(xi - body.pos[neighborIndex]));
            );

    //////////////////////////////////////////////////////////////////////////
    // Dynamic bodies
    //////////////////////////////////////////////////////////////////////////
    ITER_NEIGHBORS_SOLIDS(
                index,
                delta += body.mass[neighborIndex] * (vi - body.vel[neighborIndex]).dot(m_data.gradW(xi - body.pos[neighborIndex]));
            )

            particleSet->densityAdv[index] = MAX_MACRO_CUDA(particleSet->density[index] + m_data.h_future*delta - m_data.density0, 0.0);


#ifdef USE_WARMSTART
    particleSet->kappa[index] += (particleSet->densityAdv[index])*particleSet->factor[index];

#endif
}

__device__ void computeDensityAdv(const unsigned int index, Vector3d* posFluid, Vector3d* velFluid, int* neighbourgs, int * numberOfNeighbourgs,
                                  RealCuda* mass, SPH::PrecomputedCubicKernelPerso m_kernel_precomp, RealCuda* boundaryPsi, Vector3d* posBoundary, Vector3d* velBoundary,
                                  SPH::UnifiedParticleSet* vector_dynamic_bodies_data_cuda, RealCuda* densityAdv, RealCuda* density, RealCuda h_future, RealCuda density0) {
    const Vector3d xi = posFluid[index];
    const Vector3d vi = velFluid[index];
    RealCuda delta = 0;

    //////////////////////////////////////////////////////////////////////////
    // Fluid
    //////////////////////////////////////////////////////////////////////////
    int* neighbors_ptr = getNeighboursPtr(neighbourgs, index);
    int* end_ptr = neighbors_ptr + getNumberOfNeighbourgs(numberOfNeighbourgs, index);
    while (neighbors_ptr != end_ptr)
    {
        const unsigned int neighborIndex = *neighbors_ptr++;
        delta += mass[neighborIndex] * (vi - velFluid[neighborIndex]).dot(m_kernel_precomp.gradW(xi - posFluid[neighborIndex]));
    }

    //////////////////////////////////////////////////////////////////////////
    // Boundary
    //////////////////////////////////////////////////////////////////////////
    end_ptr += getNumberOfNeighbourgs(numberOfNeighbourgs, index, 1);
    while (neighbors_ptr != end_ptr)
    {
        const unsigned int neighborIndex = *neighbors_ptr++;
        delta += boundaryPsi[neighborIndex] * (vi - velBoundary[neighborIndex]).dot(m_kernel_precomp.gradW(xi - posBoundary[neighborIndex]));
    }

    //////////////////////////////////////////////////////////////////////////
    // Dynamic bodies
    //////////////////////////////////////////////////////////////////////////
    end_ptr += getNumberOfNeighbourgs(numberOfNeighbourgs, index, 2);
    while (neighbors_ptr != end_ptr)
    {
        READ_DYNAMIC_BODIES_PARTICLES_INDEX(neighbors_ptr, bodyIndex, neighborIndex);
        SPH::UnifiedParticleSet& body = vector_dynamic_bodies_data_cuda[bodyIndex];
        delta += body.mass[neighborIndex] * (vi - body.vel[neighborIndex]).dot(m_kernel_precomp.gradW(xi - body.pos[neighborIndex]));
    }




    densityAdv[index] = MAX_MACRO_CUDA(density[index] + h_future*delta - density0, 0.0);
}

template <bool warm_start> __device__ void pressureSolveParticle(SPH::DFSPHCData& m_data, SPH::UnifiedParticleSet* particleSet, const unsigned int i) {
    //////////////////////////////////////////////////////////////////////////
    // Evaluate rhs
    //////////////////////////////////////////////////////////////////////////
    const RealCuda ki = (warm_start) ? particleSet->kappa[i] : (particleSet->densityAdv[i])*particleSet->factor[i];

#ifdef USE_WARMSTART
    //if (!warm_start) { particleSet->kappa[i] += ki; } //moved to the evaluation
#endif


    Vector3d v_i = Vector3d(0, 0, 0);
    const Vector3d &xi = particleSet->pos[i];

    //////////////////////////////////////////////////////////////////////////
    // Fluid
    //////////////////////////////////////////////////////////////////////////
    ITER_NEIGHBORS_INIT(i);

    ITER_NEIGHBORS_FLUID(
                i,
                const RealCuda kSum = (ki + ((warm_start) ? body.kappa[neighborIndex] : (body.densityAdv[neighborIndex])*body.factor[neighborIndex]));
            if (fabs(kSum) > m_eps)
    {
        // ki, kj already contain inverse density
        v_i += kSum * body.mass[neighborIndex] * m_data.gradW(xi - body.pos[neighborIndex]);
    }
    );

    if (fabs(ki) > m_eps)
    {
        //////////////////////////////////////////////////////////////////////////
        // Boundary
        //////////////////////////////////////////////////////////////////////////
        //#define PRESSURE_COMPUTATION_BOUNDARIES_FULL
#ifndef PRESSURE_COMPUTATION_BOUNDARIES_FULL
        ITER_NEIGHBORS_BOUNDARIES(
                    i,
                    v_i += ki * body.mass[neighborIndex] * m_data.gradW(xi - body.pos[neighborIndex]);
                );
#else
        ITER_NEIGHBORS_BOUNDARIES(
                    i,
                    const RealCuda kSum = (ki + ((warm_start) ? body.kappa[neighborIndex] : (body.densityAdv[neighborIndex])*body.factor[neighborIndex]));
                if (fabs(kSum) > m_eps)
        {
            // ki, kj already contain inverse density
            v_i += kSum * body.mass[neighborIndex] * m_data.gradW(xi - body.pos[neighborIndex]);
        }
        );
#endif


        //////////////////////////////////////////////////////////////////////////
        // Dynamic bodies
        //////////////////////////////////////////////////////////////////////////
        ITER_NEIGHBORS_SOLIDS(
                    i,
                    Vector3d delta = ki * body.mass[neighborIndex] * m_data.gradW(xi - body.pos[neighborIndex]);
                v_i += delta;// ki already contains inverse density

        //we apply the force to the body particle (no invH since it has been fatorized at the end)
        delta *= -particleSet->mass[i];
        atomicAdd(&(body.F[neighborIndex].x), delta.x);
        atomicAdd(&(body.F[neighborIndex].y), delta.y);
        atomicAdd(&(body.F[neighborIndex].z), delta.z);
        );
    }

    // Directly update velocities instead of storing pressure accelerations
    particleSet->vel[i] += v_i*m_data.h_future;
}

template <bool ignore_when_no_fluid_near>
__global__ void DFSPH_divergence_warmstart_init_kernel(SPH::DFSPHCData m_data, SPH::UnifiedParticleSet* particleSet) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= particleSet->numParticles) { return; }

    if (ignore_when_no_fluid_near) {
        if (particleSet->getNumberOfNeighbourgs(i) == 0) {
            return;
        }
    }

    particleSet->kappaV[i] = MAX_MACRO_CUDA(particleSet->kappaV[i] * m_data.h_ratio_to_past / 2, -0.5);
    //computeDensityChange(m_data, i);


    //I can actually make the factor and desity computation here
    {
        //////////////////////////////////////////////////////////////////////////
        // Compute gradient dp_i/dx_j * (1/k)  and dp_j/dx_j * (1/k)
        //////////////////////////////////////////////////////////////////////////
        const Vector3d &xi = particleSet->pos[i];
        const Vector3d &vi = particleSet->vel[i];
        RealCuda sum_grad_p_k = 0;
        Vector3d grad_p_i;
        grad_p_i.setZero();

        RealCuda density = particleSet->mass[i] * m_data.W_zero;
        RealCuda densityAdv = 0;

        //////////////////////////////////////////////////////////////////////////
        // Fluid
        //////////////////////////////////////////////////////////////////////////
        ITER_NEIGHBORS_INIT(i);

        ITER_NEIGHBORS_FLUID(
                    i,
                    const Vector3d &xj = body.pos[neighborIndex];
                density += body.mass[neighborIndex] * m_data.W(xi - xj);
        const Vector3d grad_p_j = body.mass[neighborIndex] * m_data.gradW(xi - xj);
        sum_grad_p_k += grad_p_j.squaredNorm();
        grad_p_i += grad_p_j;
        densityAdv += (vi - body.vel[neighborIndex]).dot(grad_p_j);
        );


        //////////////////////////////////////////////////////////////////////////
        // Boundary
        //////////////////////////////////////////////////////////////////////////
        ITER_NEIGHBORS_BOUNDARIES(
                    i,
                    const Vector3d &xj = body.pos[neighborIndex];
                density += body.mass[neighborIndex] * m_data.W(xi - xj);
        const Vector3d grad_p_j = body.mass[neighborIndex] * m_data.gradW(xi - xj);
        sum_grad_p_k += grad_p_j.squaredNorm();
        grad_p_i += grad_p_j;
        densityAdv += (vi - body.vel[neighborIndex]).dot(grad_p_j);
        );

        //////////////////////////////////////////////////////////////////////////
        // Dynamic bodies
        //////////////////////////////////////////////////////////////////////////
        //*
        ITER_NEIGHBORS_SOLIDS(
                    i,
                    const Vector3d &xj = body.pos[neighborIndex];
                density += body.mass[neighborIndex] * m_data.W(xi - xj);
        const Vector3d grad_p_j = body.mass[neighborIndex] * m_data.gradW(xi - xj);
        sum_grad_p_k += grad_p_j.squaredNorm();
        grad_p_i += grad_p_j;
        densityAdv += (vi - body.vel[neighborIndex]).dot(grad_p_j);
        );
        //*/


        sum_grad_p_k += grad_p_i.squaredNorm();

        //////////////////////////////////////////////////////////////////////////
        // Compute pressure stiffness denominator
        //////////////////////////////////////////////////////////////////////////
        particleSet->factor[i] = (-m_data.invH / (MAX_MACRO_CUDA(sum_grad_p_k, m_eps)));
        particleSet->density[i] = density;

        //end the density adv computation
        unsigned int numNeighbors = particleSet->getNumberOfNeighbourgs(i);
        // in case of particle deficiency do not perform a divergence solve
        if (numNeighbors < 20) {
            for (unsigned int pid = 1; pid < 3; pid++)
            {
                numNeighbors += particleSet->getNumberOfNeighbourgs(i, pid);
            }
        }
        if (numNeighbors < 20) {
            particleSet->densityAdv[i] = 0;
        }
        else {
            particleSet->densityAdv[i] = MAX_MACRO_CUDA(densityAdv, 0.0);

        }

    }

}

void cuda_divergence_warmstart_init(SPH::DFSPHCData& data) {
    {//fluid
        int numBlocks = (data.fluid_data[0].numParticles + BLOCKSIZE - 1) / BLOCKSIZE;
        DFSPH_divergence_warmstart_init_kernel<false> << <numBlocks, BLOCKSIZE >> > (data, data.fluid_data[0].gpu_ptr);
    }

    if (data.boundaries_data[0].has_factor_computation){//boundaries (technically computing the density adv is useless here but nvm)
        int numBlocks = (data.boundaries_data[0].numParticles + BLOCKSIZE - 1) / BLOCKSIZE;
        DFSPH_divergence_warmstart_init_kernel<true> << <numBlocks, BLOCKSIZE >> > (data, data.boundaries_data[0].gpu_ptr);
    }


    hipError_t cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "cuda_divergence_warmstart_init failed: %d\n", (int)cudaStatus);
        exit(1598);
    }
}

template<bool warmstart> __global__ void DFSPH_divergence_compute_kernel(SPH::DFSPHCData m_data, SPH::UnifiedParticleSet* particleSet) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= particleSet->numParticles) { return; }

    if (warmstart) {
        if (particleSet->densityAdv[i] > 0.0) {
            divergenceSolveParticle<warmstart>(m_data, particleSet, i);
        }
    }
    else {
        divergenceSolveParticle<warmstart>(m_data, particleSet, i);
    }

}

template<bool warmstart> void cuda_divergence_compute(SPH::DFSPHCData& data) {
    int numBlocks = (data.fluid_data[0].numParticles + BLOCKSIZE - 1) / BLOCKSIZE;
    DFSPH_divergence_compute_kernel<warmstart> << <numBlocks, BLOCKSIZE >> > (data, data.fluid_data[0].gpu_ptr);

    hipError_t cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "cuda_divergence_compute failed: %d\n", (int)cudaStatus);
        exit(1598);
    }
}
template void cuda_divergence_compute<true>(SPH::DFSPHCData& data);
template void cuda_divergence_compute<false>(SPH::DFSPHCData& data);

__global__ void DFSPH_divergence_init_kernel(SPH::DFSPHCData m_data, SPH::UnifiedParticleSet* particleSet) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= particleSet->numParticles) { return; }

    {
#ifdef USE_WARMSTART_V
        particleSet->kappaV[i] = 0;
#endif

        ///TODO when doing this kernel I can actually fuse the code for all those computation to limit the number
        ///of time I read the particles positions
        computeDensityChange(m_data, particleSet, i);

#ifndef USE_WARMSTART_V
        //I can actually make the factor and desity computation here
        {
            //////////////////////////////////////////////////////////////////////////
            // Compute gradient dp_i/dx_j * (1/k)  and dp_j/dx_j * (1/k)
            //////////////////////////////////////////////////////////////////////////
            const Vector3d &xi = particleSet->pos[i];
            RealCuda sum_grad_p_k = 0;
            Vector3d grad_p_i;
            grad_p_i.setZero();

            RealCuda density = particleSet->mass[i] * m_data.W_zero;

            //////////////////////////////////////////////////////////////////////////
            // Fluid
            //////////////////////////////////////////////////////////////////////////
            ITER_NEIGHBORS_INIT(i);

            ITER_NEIGHBORS_FLUID(
                        i,
                        const Vector3d &xj = body.pos[neighborIndex];
                    density += body.mass[neighborIndex] * m_data.W(xi - xj);
            const Vector3d grad_p_j = body.mass[neighborIndex] * m_data.gradW(xi - xj);
            sum_grad_p_k += grad_p_j.squaredNorm();
            grad_p_i += grad_p_j;
            );

            //////////////////////////////////////////////////////////////////////////
            // Boundary
            //////////////////////////////////////////////////////////////////////////
            ITER_NEIGHBORS_BOUNDARIES(
                        i,
                        const Vector3d &xj = body.pos[neighborIndex];
                    density += body.mass[neighborIndex] * m_data.W(xi - xj);
            const Vector3d grad_p_j = body.mass[neighborIndex] * m_data.gradW(xi - xj);
            sum_grad_p_k += grad_p_j.squaredNorm();
            grad_p_i += grad_p_j;
            );

            //////////////////////////////////////////////////////////////////////////
            // Dynamic bodies
            //////////////////////////////////////////////////////////////////////////
            //*
            ITER_NEIGHBORS_SOLIDS(
                        i,
                        const Vector3d &xj = body.pos[neighborIndex];
                    density += body.mass[neighborIndex] * m_data.W(xi - xj);
            const Vector3d grad_p_j = body.mass[neighborIndex] * m_data.gradW(xi - xj);
            sum_grad_p_k += grad_p_j.squaredNorm();
            grad_p_i += grad_p_j;
            );
            //*/


            sum_grad_p_k += grad_p_i.squaredNorm();

            //////////////////////////////////////////////////////////////////////////
            // Compute pressure stiffness denominator
            //////////////////////////////////////////////////////////////////////////
            particleSet->factor[i] = (-m_data.invH / (MAX_MACRO_CUDA(sum_grad_p_k, m_eps)));
            particleSet->density[i] = density;

        }
#endif


    }

}

void cuda_divergence_init(SPH::DFSPHCData& data) {
    int numBlocks = (data.fluid_data[0].numParticles + BLOCKSIZE - 1) / BLOCKSIZE;
    DFSPH_divergence_init_kernel << <numBlocks, BLOCKSIZE >> > (data, data.fluid_data[0].gpu_ptr);

    hipError_t cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "cuda_divergence_init failed: %d\n", (int)cudaStatus);
        exit(1598);
    }
}

__global__ void DFSPH_divergence_loop_end_kernel(SPH::DFSPHCData m_data, SPH::UnifiedParticleSet* particleSet, RealCuda* avg_density_err) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= particleSet->numParticles) { return; }

    computeDensityChange(m_data, particleSet, i);
    //atomicAdd(avg_density_err, m_data.densityAdv[i]);
}

RealCuda cuda_divergence_loop_end(SPH::DFSPHCData& data) {
    int numBlocks = (data.fluid_data[0].numParticles + BLOCKSIZE - 1) / BLOCKSIZE;
    static RealCuda* avg_density_err = NULL;
    if (avg_density_err == NULL) {
        hipMalloc(&(avg_density_err), sizeof(RealCuda));
    }

    DFSPH_divergence_loop_end_kernel << <numBlocks, BLOCKSIZE >> > (data, data.fluid_data[0].gpu_ptr, avg_density_err);

    hipError_t cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "cuda_divergence_loop_end failed: %d\n", (int)cudaStatus);
        exit(1598);
    }


    // Run sum-reduction
    hipcub::DeviceReduce::Sum(data.fluid_data->d_temp_storage, data.fluid_data->temp_storage_bytes, data.fluid_data->densityAdv, avg_density_err, data.fluid_data[0].numParticles);
    gpuErrchk(hipDeviceSynchronize());


    RealCuda result = 0;
    gpuErrchk(hipMemcpy(&result, avg_density_err, sizeof(RealCuda), hipMemcpyDeviceToHost));

    return result;
}

__global__ void DFSPH_viscosityXSPH_kernel(SPH::DFSPHCData m_data, SPH::UnifiedParticleSet* particleSet) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= particleSet->numParticles) { return; }

    //I set the gravitation directly here to lover the number of kernels
    Vector3d ai = Vector3d(0, 0, 0);
    const Vector3d &xi = particleSet->pos[i];
    const Vector3d &vi = particleSet->vel[i];

    //////////////////////////////////////////////////////////////////////////
    // Fluid
    //////////////////////////////////////////////////////////////////////////
    ITER_NEIGHBORS_INIT(i);

    ITER_NEIGHBORS_FLUID(
                i,
                ai -= m_data.invH * m_data.viscosity * (body.mass[neighborIndex] / body.density[neighborIndex]) *
            (vi - body.vel[neighborIndex]) * m_data.W(xi - body.pos[neighborIndex]);
            )

            particleSet->acc[i] = m_data.gravitation + ai;
}

void cuda_viscosityXSPH(SPH::DFSPHCData& data) {
    int numBlocks = (data.fluid_data[0].numParticles + BLOCKSIZE - 1) / BLOCKSIZE;
    DFSPH_viscosityXSPH_kernel << <numBlocks, BLOCKSIZE >> > (data, data.fluid_data[0].gpu_ptr);

    hipError_t cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "cuda_viscosityXSPH failed: %d\n", (int)cudaStatus);
        exit(1598);
    }
}

__global__ void DFSPH_CFL_kernel(SPH::DFSPHCData m_data, SPH::UnifiedParticleSet particleSet, RealCuda* maxVel) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= m_data.fluid_data[0].numParticles) { return; }

    for (unsigned int i = 0; i < m_data.fluid_data[0].numParticles; i++)
    {
        const RealCuda velMag = (particleSet.vel[i] + particleSet.acc[i] * m_data.h).squaredNorm();
        if (velMag > *maxVel)
            *maxVel = velMag;
    }
}

__global__ void DFSPH_CFLVelSquaredNorm_kernel(SPH::DFSPHCData m_data, SPH::UnifiedParticleSet* particleSet, RealCuda* sqaredNorm) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= particleSet->numParticles) { return; }

    sqaredNorm[i] = (particleSet->vel[i] + particleSet->acc[i] * m_data.h).squaredNorm();
}

__global__ void DFSPH_CFLAdvanced_kernel(SPH::DFSPHCData m_data, RealCuda *max, int *mutex, unsigned int n)
{
    unsigned int index = threadIdx.x + blockIdx.x*blockDim.x;
    unsigned int stride = gridDim.x*blockDim.x;
    unsigned int offset = 0;

    __shared__ RealCuda cache[256];


    RealCuda temp = 0;
    while (index + offset < n) {
        int i = index + offset;
        const RealCuda velMag = (m_data.fluid_data_cuda->vel[i] + m_data.fluid_data_cuda->acc[i] * m_data.h).squaredNorm();
        temp = fmaxf(temp, velMag);

        offset += stride;
    }

    cache[threadIdx.x] = temp;

    __syncthreads();


    // reduction
    unsigned int i = blockDim.x / 2;
    while (i != 0) {
        if (threadIdx.x < i) {
            cache[threadIdx.x] = MAX_MACRO_CUDA(cache[threadIdx.x], cache[threadIdx.x + i]);
        }

        __syncthreads();
        i /= 2;
    }

    if (threadIdx.x == 0) {
        while (atomicCAS(mutex, 0, 1) != 0);  //lock
        *max = MAX_MACRO_CUDA(*max, cache[0]);
        atomicExch(mutex, 0);  //unlock
    }
}

void cuda_CFL(SPH::DFSPHCData& m_data, const RealCuda minTimeStepSize, RealCuda m_cflFactor, RealCuda m_cflMaxTimeStepSize) {

    //we compute the square norm

    std::chrono::steady_clock::time_point t0 = std::chrono::steady_clock::now();

    RealCuda* out_buff;
    hipMalloc(&(out_buff), sizeof(RealCuda));

    if (true) {

        //cub version
        static RealCuda* temp_buff = NULL;
        if (temp_buff == NULL) {
            hipMallocManaged(&(temp_buff), m_data.fluid_data[0].numParticles * sizeof(RealCuda));
        }
        int numBlocks = (m_data.fluid_data[0].numParticles + BLOCKSIZE - 1) / BLOCKSIZE;
        DFSPH_CFLVelSquaredNorm_kernel << <numBlocks, BLOCKSIZE >> > (m_data, m_data.fluid_data[0].gpu_ptr, temp_buff);

        hipError_t cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "cuda_cfl squared norm failed: %d\n", (int)cudaStatus);
            exit(1598);
        }

        // Determine temporary device storage requirements
        static void     *d_temp_storage = NULL;
        static size_t   temp_storage_bytes = 0;
        if (d_temp_storage == NULL) {
            hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, temp_buff, out_buff, m_data.fluid_data[0].numParticles);
            // Allocate temporary storage
            hipMalloc(&d_temp_storage, temp_storage_bytes);
        }
        // Run max-reduction
        hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, temp_buff, out_buff, m_data.fluid_data[0].numParticles);

    }
    else {
        //manual
        int *d_mutex;
        hipMalloc((void**)&d_mutex, sizeof(int));
        hipMemset(d_mutex, 0, sizeof(float));

        int numBlocks = (m_data.fluid_data[0].numParticles + BLOCKSIZE - 1) / BLOCKSIZE;
        DFSPH_CFLAdvanced_kernel << < numBlocks, BLOCKSIZE >> > (m_data, out_buff, d_mutex, m_data.fluid_data[0].numParticles);

        hipError_t cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "cuda_cfl failed: %d\n", (int)cudaStatus);
            exit(1598);
        }
        hipFree(d_mutex);
    }
    RealCuda maxVel;
    hipMemcpy(&maxVel, out_buff, sizeof(RealCuda), hipMemcpyDeviceToHost);
    hipFree(out_buff);

    std::chrono::steady_clock::time_point t1 = std::chrono::steady_clock::now();

    RealCuda h = m_data.h;

    // Approximate max. time step size
    h = m_cflFactor * .4 * (2.0*m_data.particleRadius / (sqrt(maxVel)));

    h = min(h, m_cflMaxTimeStepSize);
    h = max(h, minTimeStepSize);

    m_data.updateTimeStep(h);//*/


    std::chrono::steady_clock::time_point t2 = std::chrono::steady_clock::now();



    float time_search = std::chrono::duration_cast<std::chrono::nanoseconds> (t1 - t0).count() / 1000000.0f;
    float time_comp = std::chrono::duration_cast<std::chrono::nanoseconds> (t2 - t1).count() / 1000000.0f;

    printf("Time to do cfl (search,comp): %f    %f\n", time_search, time_comp);
}

__global__ void DFSPH_update_vel_kernel(SPH::DFSPHCData m_data, SPH::UnifiedParticleSet* particleSet) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= particleSet->numParticles) { return; }

    particleSet->vel[i] += m_data.h * particleSet->acc[i];

#ifdef USE_WARMSTART	
    //done here to have one less kernel
    particleSet->kappa[i] = MAX_MACRO_CUDA(particleSet->kappa[i] * m_data.h_ratio_to_past2, -0.5);
#endif
}




void cuda_update_vel(SPH::DFSPHCData& data) {
    int numBlocks = (data.fluid_data[0].numParticles + BLOCKSIZE - 1) / BLOCKSIZE;
    DFSPH_update_vel_kernel << <numBlocks, BLOCKSIZE >> > (data, data.fluid_data[0].gpu_ptr);

    hipError_t cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "cuda_update_vel failed: %d\n", (int)cudaStatus);
        exit(1598);
    }


}

template<bool warmstart> __global__ void DFSPH_pressure_compute_kernel(SPH::DFSPHCData m_data, SPH::UnifiedParticleSet* particleSet) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= particleSet->numParticles) { return; }

    pressureSolveParticle<warmstart>(m_data, particleSet, i);

}

template<bool warmstart> void cuda_pressure_compute(SPH::DFSPHCData& data) {
    int numBlocks = (data.fluid_data[0].numParticles + BLOCKSIZE - 1) / BLOCKSIZE;
    DFSPH_pressure_compute_kernel<warmstart> << <numBlocks, BLOCKSIZE >> > (data, data.fluid_data[0].gpu_ptr);

    hipError_t cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "cuda_pressure_compute failed: %d\n", (int)cudaStatus);
        exit(1598);
    }
}
template void cuda_pressure_compute<true>(SPH::DFSPHCData& data);
template void cuda_pressure_compute<false>(SPH::DFSPHCData& data);


template <bool ignore_when_no_fluid_near>
__global__ void DFSPH_pressure_init_kernel(SPH::DFSPHCData m_data, SPH::UnifiedParticleSet* particleSet) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= particleSet->numParticles) { return; }

#ifdef USE_WARMSTART
    particleSet->kappa[i] = 0;
#endif

    if (ignore_when_no_fluid_near) {
        if (particleSet->getNumberOfNeighbourgs(i) == 0) {
            return;
        }
    }

    particleSet->factor[i] *= m_data.invH_future;

    computeDensityAdv(m_data, particleSet, i);


}

void cuda_pressure_init(SPH::DFSPHCData& data) {
    {//fluid
        int numBlocks = (data.fluid_data[0].numParticles + BLOCKSIZE - 1) / BLOCKSIZE;
        DFSPH_pressure_init_kernel<false> << <numBlocks, BLOCKSIZE >> > (data, data.fluid_data[0].gpu_ptr);
    }
    if (data.boundaries_data[0].has_factor_computation) {//boundaries
        int numBlocks = (data.boundaries_data[0].numParticles + BLOCKSIZE - 1) / BLOCKSIZE;
        DFSPH_pressure_init_kernel<true> << <numBlocks, BLOCKSIZE >> > (data, data.boundaries_data[0].gpu_ptr);
    }

    hipError_t cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "cuda_pressure_init failed: %d\n", (int)cudaStatus);
        exit(1598);
    }
}

template <bool ignore_when_no_fluid_near>
__global__ void DFSPH_pressure_loop_end_kernel(SPH::DFSPHCData m_data, SPH::UnifiedParticleSet* particleSet, RealCuda* avg_density_err) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= particleSet->numParticles) { return; }

    if (ignore_when_no_fluid_near) {
        if (particleSet->getNumberOfNeighbourgs(i) == 0) {
            return;
        }
    }

    computeDensityAdv(m_data, particleSet, i);
    //atomicAdd(avg_density_err, m_data.densityAdv[i]);
}
/*
__global__ void DFSPH_pressure_loop_end_kernel(int numFluidParticles, Vector3d* posFluid, Vector3d* velFluid, int* neighbourgs, int * numberOfNeighbourgs,
    RealCuda* mass, SPH::PrecomputedCubicKernelPerso m_kernel_precomp, RealCuda* boundaryPsi, Vector3d* posBoundary, Vector3d* velBoundary,
    SPH::UnifiedParticleSet* vector_dynamic_bodies_data_cuda, RealCuda* densityAdv, RealCuda* density, RealCuda h_future, RealCuda density0) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= numFluidParticles) { return; }

    computeDensityAdv(i, posFluid, velFluid, neighbourgs, numberOfNeighbourgs,
        mass, m_kernel_precomp, boundaryPsi, posBoundary, velBoundary,
        vector_dynamic_bodies_data_cuda, densityAdv, density, h_future, density0);
}//*/
//*/
RealCuda cuda_pressure_loop_end(SPH::DFSPHCData& data) {

    std::chrono::steady_clock::time_point p0 = std::chrono::steady_clock::now();

    static RealCuda* avg_density_err = NULL;
    if (avg_density_err == NULL) {
        hipMalloc(&(avg_density_err), sizeof(RealCuda));
    }
    {
        int numBlocks = (data.fluid_data[0].numParticles + BLOCKSIZE - 1) / BLOCKSIZE;
        DFSPH_pressure_loop_end_kernel<false> << <numBlocks, BLOCKSIZE >> > (data, data.fluid_data[0].gpu_ptr, avg_density_err);
    }
    if (data.boundaries_data[0].has_factor_computation) {//boundaries
        int numBlocks = (data.boundaries_data[0].numParticles + BLOCKSIZE - 1) / BLOCKSIZE;
        DFSPH_pressure_loop_end_kernel<true> << <numBlocks, BLOCKSIZE >> > (data, data.boundaries_data[0].gpu_ptr, avg_density_err);
    }

    /*
    ///LOL the detailed implementation is slower so no need to even think about developping data
    DFSPH_pressure_loop_end_kernel << <numBlocks, BLOCKSIZE >> > (data.numFluidParticles, data.posFluid, data.velFluid,
    data.neighbourgs, data.numberOfNeighbourgs,
    data.mass, data.m_kernel_precomp, data.boundaryPsi, data.posBoundary, data.velBoundary,
    data.vector_dynamic_bodies_data_cuda, data.densityAdv, data.density, data.h_future, data.density0);
    //*/

    hipError_t cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "cuda_pressure_loop_end failed: %d\n", (int)cudaStatus);
        exit(1598);
    }

    std::chrono::steady_clock::time_point p1 = std::chrono::steady_clock::now();

    // Run sum-reduction
    hipcub::DeviceReduce::Sum(data.fluid_data->d_temp_storage, data.fluid_data->temp_storage_bytes, data.fluid_data->densityAdv, avg_density_err, data.fluid_data[0].numParticles);


    RealCuda result = 0;
    gpuErrchk(hipMemcpy(&result, avg_density_err, sizeof(RealCuda), hipMemcpyDeviceToHost));


    std::chrono::steady_clock::time_point p2 = std::chrono::steady_clock::now();
    float time1 = std::chrono::duration_cast<std::chrono::nanoseconds> (p1 - p0).count() / 1000000.0f;
    float time2 = std::chrono::duration_cast<std::chrono::nanoseconds> (p2 - p1).count() / 1000000.0f;

    //std::cout << "pressure loop end details: " << time1 << "  " << time2 << std::endl;

    return result;
}

__global__ void DFSPH_update_pos_kernel(SPH::DFSPHCData data, SPH::UnifiedParticleSet* particleSet) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= particleSet->numParticles) { return; }

    if (data.damp_borders) {
        /*
        RealCuda max_vel_sq = (data.particleRadius / 2.0f) / data.h;
        max_vel_sq *= max_vel_sq;
        RealCuda cur_vel_sq = particleSet->vel[i].squaredNorm();
        if (cur_vel_sq> max_vel_sq)
        {
            particleSet->vel[i] *= max_vel_sq / cur_vel_sq;
        }//*/

        RealCuda affected_distance_sq= data.particleRadius*6;
        affected_distance_sq *= affected_distance_sq;

        for (int k = 0; k < data.damp_planes_count; ++k) {
            Vector3d plane = data.damp_planes[k];
            if ((particleSet->pos[i] * plane.abs() / plane.norm() - plane).squaredNorm() < affected_distance_sq) {
                RealCuda max_vel_sq = (data.particleRadius / 25.0f) / data.h;
                max_vel_sq *= max_vel_sq;
                RealCuda cur_vel_sq = particleSet->vel[i].squaredNorm();
                if (cur_vel_sq> max_vel_sq)
                {
                    particleSet->vel[i] *= max_vel_sq / cur_vel_sq;
                }
                //if we triggered once no need to check for the other planes
                break;
            }
        }
    }


    particleSet->pos[i] += data.h * particleSet->vel[i];
}



void cuda_update_pos(SPH::DFSPHCData& data) {
    if (data.damp_borders) {
        for (int k = 0; k < data.damp_planes_count; ++k) {
            Vector3d plane = data.damp_planes[k];
            std::cout << "damping plane: " << plane.x << "  " << plane.y << "  " << plane.z << std::endl;
        }
    }

    int numBlocks = (data.fluid_data[0].numParticles + BLOCKSIZE - 1) / BLOCKSIZE;
    DFSPH_update_pos_kernel << <numBlocks, BLOCKSIZE >> > (data, data.fluid_data[0].gpu_ptr);

    data.damp_borders_steps_count--;
    if (data.damp_borders_steps_count == 0) {
        data.damp_borders = false;
    }

    hipError_t cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "cuda_update_pos failed: %d\n", (int)cudaStatus);
        exit(1598);
    }
}


int cuda_divergenceSolve(SPH::DFSPHCData& m_data, const unsigned int maxIter, const RealCuda maxError) {
    //////////////////////////////////////////////////////////////////////////
    // Init parameters
    //////////////////////////////////////////////////////////////////////////

    const RealCuda h = m_data.h;
    const int numParticles = m_data.fluid_data[0].numParticles;
    const RealCuda density0 = m_data.density0;

    std::chrono::steady_clock::time_point start = std::chrono::steady_clock::now();

#ifdef USE_WARMSTART_V
    cuda_divergence_warmstart_init(m_data);

    std::chrono::steady_clock::time_point m0 = std::chrono::steady_clock::now();
    cuda_divergence_compute<true>(m_data);
#endif

    std::chrono::steady_clock::time_point m1 = std::chrono::steady_clock::now();
    //////////////////////////////////////////////////////////////////////////
    // Compute velocity of density change
    //////////////////////////////////////////////////////////////////////////
    cuda_divergence_init(m_data);

    std::chrono::steady_clock::time_point m2 = std::chrono::steady_clock::now();

    unsigned int m_iterationsV = 0;

    //////////////////////////////////////////////////////////////////////////
    // Start solver
    //////////////////////////////////////////////////////////////////////////

    // Maximal allowed density fluctuation
    // use maximal density error divided by time step size
    const RealCuda eta = maxError * 0.01 * density0 / h;  // maxError is given in percent

    float time_3_1 = 0;
    float time_3_2 = 0;
    RealCuda avg_density_err = 0.0;
    while (((avg_density_err > eta) || (m_iterationsV < 1)) && (m_iterationsV < maxIter))
    {

        //////////////////////////////////////////////////////////////////////////
        // Perform Jacobi iteration over all blocks
        //////////////////////////////////////////////////////////////////////////
        std::chrono::steady_clock::time_point p0 = std::chrono::steady_clock::now();
        cuda_divergence_compute<false>(m_data);
        std::chrono::steady_clock::time_point p1 = std::chrono::steady_clock::now();

        avg_density_err = cuda_divergence_loop_end(m_data);
        std::chrono::steady_clock::time_point p2 = std::chrono::steady_clock::now();

        avg_density_err /= numParticles;
        m_iterationsV++;

        time_3_1 += std::chrono::duration_cast<std::chrono::nanoseconds> (p1 - p0).count() / 1000000.0f;
        time_3_2 += std::chrono::duration_cast<std::chrono::nanoseconds> (p2 - p1).count() / 1000000.0f;
    }

    /*
    std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();

    float time_0 = std::chrono::duration_cast<std::chrono::nanoseconds> (m0 - start).count() / 1000000.0f;
    float time_1 = std::chrono::duration_cast<std::chrono::nanoseconds> (m1 - m0).count() / 1000000.0f;
    float time_2 = std::chrono::duration_cast<std::chrono::nanoseconds> (m2 - m1).count() / 1000000.0f;
    float time_3 = std::chrono::duration_cast<std::chrono::nanoseconds> (end - m2).count() / 1000000.0f;

    std::cout << "detail pressure solve (iter total (varible_comp warm_comp init actual_comp (t1 t2))): " << m_iterationsV << "  " << time_0+ time_1 + time_2 + time_3 <<
        "  (" << time_0 << "  " << time_1 << "  " << time_2 << "  " << time_3 << "(" << time_3_1 << " " << time_3_2 << ") )" << std::endl;

    //*/
    return m_iterationsV;
}



int cuda_pressureSolve(SPH::DFSPHCData& m_data, const unsigned int m_maxIterations, const RealCuda m_maxError) {
    const RealCuda density0 = m_data.density0;
    const int numParticles = (int)m_data.fluid_data[0].numParticles;
    RealCuda avg_density_err = 0.0;


    std::chrono::steady_clock::time_point start = std::chrono::steady_clock::now();


#ifdef USE_WARMSTART		
    cuda_pressure_compute<true>(m_data);
#endif


    std::chrono::steady_clock::time_point m1 = std::chrono::steady_clock::now();

    //////////////////////////////////////////////////////////////////////////
    // Compute rho_adv
    //////////////////////////////////////////////////////////////////////////
    cuda_pressure_init(m_data);


    std::chrono::steady_clock::time_point m2 = std::chrono::steady_clock::now();


    unsigned int m_iterations = 0;

    //////////////////////////////////////////////////////////////////////////
    // Start solver
    //////////////////////////////////////////////////////////////////////////

    // Maximal allowed density fluctuation
    const RealCuda eta = m_maxError * 0.01 * density0;  // maxError is given in percent

    float time_3_1 = 0;
    float time_3_2 = 0;
    while (((avg_density_err > eta) || (m_iterations < 2)) && (m_iterations < m_maxIterations))
    {
        std::chrono::steady_clock::time_point p0 = std::chrono::steady_clock::now();
        cuda_pressure_compute<false>(m_data);
        std::chrono::steady_clock::time_point p1 = std::chrono::steady_clock::now();
        avg_density_err = cuda_pressure_loop_end(m_data);
        std::chrono::steady_clock::time_point p2 = std::chrono::steady_clock::now();
        avg_density_err /= numParticles;

        m_iterations++;

        time_3_1 += std::chrono::duration_cast<std::chrono::nanoseconds> (p1 - p0).count() / 1000000.0f;
        time_3_2 += std::chrono::duration_cast<std::chrono::nanoseconds> (p2 - p1).count() / 1000000.0f;
    }
    /*
    std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();

    float time_1 = std::chrono::duration_cast<std::chrono::nanoseconds> (m1 - start).count() / 1000000.0f;
    float time_2 = std::chrono::duration_cast<std::chrono::nanoseconds> (m2 - m1).count() / 1000000.0f;
    float time_3 = std::chrono::duration_cast<std::chrono::nanoseconds> (end - m2).count() / 1000000.0f;

    std::cout << "detail pressure solve (iter total (warm init actual_comp (t1 t2))): " <<m_iterations <<"  "<< time_1 + time_2 +time_3 <<
    "  (" << time_1 << "  " << time_2<< "  "<< time_3 <<"("<< time_3_1<<" "<< time_3_2<<") )" << std::endl;

    //*/

    return m_iterations;

}


template<unsigned int grid_size, bool z_curve>
__global__ void DFSPH_computeGridIdx_kernel(Vector3d* in, unsigned int* out, RealCuda kernel_radius, unsigned int num_particles,
                                            Vector3i gridOffset) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= num_particles) { return; }

    if (z_curve) {

    }
    else {
        //the offset is used to be able to use a small grid bu placing the simulation correctly inside it
        Vector3d pos = (in[i] / kernel_radius) + gridOffset;
        pos.toFloor();
        out[i] = COMPUTE_CELL_INDEX(pos.x, pos.y, pos.z);
    }
}

__global__ void DFSPH_Histogram_kernel(unsigned int* in, unsigned int* out, unsigned int num_particles) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= num_particles) { return; }

    atomicAdd(&(out[in[i]]), 1);

}

__global__ void DFSPH_setBufferValueToItself_kernel(unsigned int* buff, unsigned int buff_size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= buff_size) { return; }

    buff[i] = i;
}

__global__ void DFSPH_setVector3dBufferToZero_kernel(Vector3d* buff, unsigned int buff_size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= buff_size) { return; }

    buff[i] = Vector3d(0, 0, 0);
}

__global__ void DFSPH_neighborsSearch_kernel(SPH::DFSPHCData data, SPH::UnifiedParticleSet* particleSet) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= particleSet->numParticles) { return; }


    RealCuda radius_sq = data.m_kernel_precomp.getRadius();
    Vector3d pos = particleSet->pos[i];
    Vector3d pos_cell = (pos / radius_sq) + data.gridOffset; //on that line the radius is not yet squared
    pos_cell.toFloor();
    int x = pos_cell.x;
    int y = pos_cell.y;
    int z = pos_cell.z;
    radius_sq *= radius_sq;

    unsigned int nb_neighbors_fluid = 0;
    unsigned int nb_neighbors_boundary = 0;
    unsigned int nb_neighbors_dynamic_objects = 0;
    int* cur_neighbor_ptr = particleSet->neighbourgs + i*MAX_NEIGHBOURS;
    //int neighbors_fluid[MAX_NEIGHBOURS];//doing it with local buffer was not faster
    //int neighbors_boundary[MAX_NEIGHBOURS];

#ifdef USE_COMPLETE
    ///this version uses the morton indexes
#define ITER_CELLS_FOR_BODY(input_body,code){\
    const SPH::UnifiedParticleSet& body = input_body;\
    for (int k = -1; k < 2; ++k) {\
    for (int m = -1; m < 2; ++m) {\
    for (int n = -1; n < 2; ++n) {\
    unsigned int cur_cell_id = COMPUTE_CELL_INDEX(x + n, y + k, z + m);\
    unsigned int end = body.neighborsDataSet->cell_start_end[cur_cell_id + 1];\
    for (unsigned int cur_particle = body.neighborsDataSet->cell_start_end[cur_cell_id]; cur_particle < end; ++cur_particle) {\
    unsigned int j = body.neighborsDataSet->p_id_sorted[cur_particle];\
    if ((pos - body.pos[j]).squaredNorm() < radius_sq) {\
    code\
}\
}\
}\
}\
}\
}
#else
    ///this version uses  standart indexes

    //since this version use the std index to be able to iterate on 3 successive cells
    //I can do the -1 at the start on x.
    //one thing: it x=0 then we can only iterate 2 cells at a time
    unsigned int successive_cells_count = (x > 0) ? 3 : 2;
    x = (x > 0) ? x - 1 : x;

#define ITER_CELLS_FOR_BODY(neighborsDataSet_i,pos_body_particles_i,code){\
    SPH::NeighborsSearchDataSet* neighborsDataSet= neighborsDataSet_i;\
    Vector3d* pos_body_particles=pos_body_particles_i;\
    for (int k = -1; k < 2; ++k) {\
    for (int m = -1; m < 2; ++m) {\
    unsigned int cur_cell_id = COMPUTE_CELL_INDEX(x, y + k, z + m);\
    unsigned int end = neighborsDataSet->cell_start_end[cur_cell_id + successive_cells_count];\
    for (unsigned int cur_particle = neighborsDataSet->cell_start_end[cur_cell_id]; cur_particle < end; ++cur_particle) {\
    unsigned int j = neighborsDataSet->p_id_sorted[cur_particle];\
    if ((pos - pos_body_particles[j]).squaredNorm() < radius_sq) {\
    code\
}\
}\
}\
}\
}
#endif


    if (data.is_fluid_aggregated){
        int neighbors_solids[MAX_NEIGHBOURS];

        //dynamic bodies
        if (data.vector_dynamic_bodies_data_cuda != NULL) {

#ifdef GROUP_DYNAMIC_BODIES_NEIGHBORS_SEARCH
            ITER_CELLS_FOR_BODY(data.neighborsDataSetGroupedDynamicBodies_cuda, data.posBufferGroupedDynamicBodies,
                                if(j<data.fluid_data_cuda->numParticles){
                                    if (i != j) { *cur_neighbor_ptr++ = j;	nb_neighbors_fluid++; }
                                }else{int body_id=0; int count_particles_previous_bodies=data.fluid_data_cuda->numParticles;
                                      while((count_particles_previous_bodies+data.vector_dynamic_bodies_data_cuda[body_id].numParticles)<j ){
                                          count_particles_previous_bodies+=data.vector_dynamic_bodies_data_cuda[body_id].numParticles;
                                          body_id++;
                                      }
                                      //*cur_neighbor_ptr++ = WRITTE_DYNAMIC_BODIES_PARTICLES_INDEX(body_id, j-count_particles_previous_bodies);
                                      neighbors_solids[nb_neighbors_dynamic_objects]=WRITTE_DYNAMIC_BODIES_PARTICLES_INDEX(body_id, j-count_particles_previous_bodies);
                                      nb_neighbors_dynamic_objects++;} )
        #else
            for (int id_body = 0; id_body < data.numDynamicBodies; ++id_body) {
                ITER_CELLS_FOR_BODY(data.vector_dynamic_bodies_data_cuda[id_body].neighborsDataSet, data.vector_dynamic_bodies_data_cuda[id_body].pos,
                                    *cur_neighbor_ptr++ = WRITTE_DYNAMIC_BODIES_PARTICLES_INDEX(id_body, j); nb_neighbors_dynamic_objects++; )
            }
#endif

        }else{
            //fluid
            ITER_CELLS_FOR_BODY(data.fluid_data_cuda[0].neighborsDataSet, data.fluid_data_cuda[0].pos,
                    if (i != j) { *cur_neighbor_ptr++ = j;	nb_neighbors_fluid++; });
        }

        //boundaries
        ITER_CELLS_FOR_BODY(data.boundaries_data_cuda[0].neighborsDataSet, data.boundaries_data_cuda[0].pos,
                *cur_neighbor_ptr++ = j; nb_neighbors_boundary++; );


        //copy the dynamic bodies at the end
        for (int j=0;j<nb_neighbors_dynamic_objects;++j){
            *cur_neighbor_ptr++=neighbors_solids[j];
        }


    }else{
        //uses the standart version
        //fluid
        ITER_CELLS_FOR_BODY(data.fluid_data_cuda[0].neighborsDataSet, data.fluid_data_cuda[0].pos,
                if (i != j) { *cur_neighbor_ptr++ = j;	nb_neighbors_fluid++; });

        //boundaries
        ITER_CELLS_FOR_BODY(data.boundaries_data_cuda[0].neighborsDataSet, data.boundaries_data_cuda[0].pos,
                *cur_neighbor_ptr++ = j; nb_neighbors_boundary++; );


        if (data.vector_dynamic_bodies_data_cuda != NULL) {

#ifdef GROUP_DYNAMIC_BODIES_NEIGHBORS_SEARCH
            ITER_CELLS_FOR_BODY(data.neighborsDataSetGroupedDynamicBodies_cuda, data.posBufferGroupedDynamicBodies,
            {int body_id=0; int count_particles_previous_bodies=0;
             while((count_particles_previous_bodies+data.vector_dynamic_bodies_data_cuda[body_id].numParticles)<j ){
                 count_particles_previous_bodies+=data.vector_dynamic_bodies_data_cuda[body_id].numParticles;
                 body_id++;
             }
             *cur_neighbor_ptr++ = WRITTE_DYNAMIC_BODIES_PARTICLES_INDEX(body_id, j-count_particles_previous_bodies);
             nb_neighbors_dynamic_objects++;} )
        #else
            for (int id_body = 0; id_body < data.numDynamicBodies; ++id_body) {
                ITER_CELLS_FOR_BODY(data.vector_dynamic_bodies_data_cuda[id_body].neighborsDataSet, data.vector_dynamic_bodies_data_cuda[id_body].pos,
                                    *cur_neighbor_ptr++ = WRITTE_DYNAMIC_BODIES_PARTICLES_INDEX(id_body, j); nb_neighbors_dynamic_objects++; )
            }
#endif

        }
    }



    particleSet->numberOfNeighbourgs[3 * i] = nb_neighbors_fluid;
    particleSet->numberOfNeighbourgs[3 * i + 1] = nb_neighbors_boundary;
    particleSet->numberOfNeighbourgs[3 * i + 2] = nb_neighbors_dynamic_objects;

    //memcpy((neighbors_buff + i*MAX_NEIGHBOURS*2), neighbors_fluid, sizeof(int)*nb_neighbors_fluid);
    //memcpy((neighbors_buff + i*MAX_NEIGHBOURS * 2 + MAX_NEIGHBOURS), neighbors_boundary, sizeof(int)*nb_neighbors_boundary);


}

__global__ void DFSPH_neighborsSearchBasic_kernel(unsigned int numFluidParticles, RealCuda radius,
                                                  SPH::UnifiedParticleSet* fluid_data,
                                                  SPH::UnifiedParticleSet* boundaries_data,
                                                  SPH::UnifiedParticleSet* vect_dynamic_bodies, int nb_dynamic_bodies) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= numFluidParticles) { return; }


    RealCuda radius_sq = radius;
    Vector3d pos = fluid_data->pos[i];
    radius_sq *= radius_sq;

    unsigned int nb_neighbors_fluid = 0;
    unsigned int nb_neighbors_boundary = 0;
    unsigned int nb_neighbors_dynamic_objects = 0;
    int* cur_neighbor_ptr = fluid_data->neighbourgs + i*MAX_NEIGHBOURS;

    for (int k = 0; k < fluid_data->numParticles; ++k) {
        if (i != k) {
            if ((fluid_data->pos[k] - pos).squaredNorm() < radius_sq) {
                *cur_neighbor_ptr++ = k;	nb_neighbors_fluid++;
            }
        }
    }

    /*
    for (int k = 0; k < boundaries_data->numParticles; ++k) {
        if ((boundaries_data->pos[k] - pos).squaredNorm() < radius_sq) {
            *cur_neighbor_ptr++ = k; nb_neighbors_boundary++;
        }
    }
    //*/

    /*
    for (int id_body = 0; id_body < nb_dynamic_bodies; ++id_body) {
        for (int k = 0; k < vect_dynamic_bodies[id_body].numParticles; ++k) {
            if ((vect_dynamic_bodies[id_body].pos[k] - pos).squaredNorm() < radius_sq) {
                *cur_neighbor_ptr++ = WRITTE_DYNAMIC_BODIES_PARTICLES_INDEX(id_body, k); nb_neighbors_dynamic_objects++;
            }
        }
    }
    //*/


    fluid_data->numberOfNeighbourgs[3 * i] = nb_neighbors_fluid;
    fluid_data->numberOfNeighbourgs[3 * i + 1] = nb_neighbors_boundary;
    fluid_data->numberOfNeighbourgs[3 * i + 2] = nb_neighbors_dynamic_objects;

}

void cuda_neighborsSearchInternal_sortParticlesId(Vector3d* pos, RealCuda kernel_radius, Vector3i gridOffset, int numParticles,
                                                  void **d_temp_storage_pair_sort, size_t   &temp_storage_bytes_pair_sort,
                                                  unsigned int* cell_id, unsigned int* cell_id_sorted,
                                                  unsigned int* p_id, unsigned int* p_id_sorted) {
    hipError_t cudaStatus;


    /*
    //some test for the definition domain (it is just for debugging purposes)
    //check for negatives values
    for (int i = 0; i < numParticles; ++i) {
    Vector3d temp = (pos[i] / kernel_radius) + 2;
    if (temp.x <= 0 || temp.y <= 0 || temp.z <= 0 ) {
    fprintf(stderr, "negative coordinates: %d\n", (int)i);
    exit(1598);
    }
    }


    //find the bounding box of the particles
    Vector3d min = pos[0];
    Vector3d max = pos[0];
    for (int i = 0; i < numParticles; ++i) {

    if (pos[i].x < min.x) { min.x = pos[i].x; }
    if (pos[i].y < min.y) { min.y = pos[i].y; }
    if (pos[i].z < min.z) { min.z = pos[i].z; }

    if (pos[i].x > max.x) { max.x = pos[i].x; }
    if (pos[i].y > max.y) { max.y = pos[i].y; }
    if (pos[i].z > max.z) { max.z = pos[i].z; }

    }
    fprintf(stderr, "min: %f // %f // %f\n", min.x, min.y, min.z);
    fprintf(stderr, "max: %f // %f // %f\n", max.x, max.y, max.z);
    fprintf(stderr, "description: %f\n", CELL_ROW_LENGTH*kernel_radius);
    exit(1598);
    //*/
    int numBlocks = (numParticles + BLOCKSIZE - 1) / BLOCKSIZE;


    //compute the idx of the cell for each particles
    DFSPH_computeGridIdx_kernel<CELL_ROW_LENGTH, false> << <numBlocks, BLOCKSIZE >> > (pos, cell_id,
                                                                                       kernel_radius, numParticles, gridOffset);

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "idxs failed: %d\n", (int)cudaStatus);
        exit(1598);
    }


    //do the actual sort
    // Run sorting operation
    hipcub::DeviceRadixSort::SortPairs(*d_temp_storage_pair_sort, temp_storage_bytes_pair_sort,
                                    cell_id, cell_id_sorted, p_id, p_id_sorted, numParticles);
    //*/


    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "sort failed: %d\n", (int)cudaStatus);
        exit(1598);
    }

}

void cuda_neighborsSearchInternal_computeCellStartEnd(int numParticles, unsigned int* cell_id_sorted,
                                                      unsigned int* hist, void **d_temp_storage_cumul_hist, size_t   &temp_storage_bytes_cumul_hist, unsigned int* cell_start_end) {
    hipError_t cudaStatus;
    int numBlocks = (numParticles + BLOCKSIZE - 1) / BLOCKSIZE;


    //Now we need to determine the start and end of each cell
    //init the histogram values. Maybe doing it wiith thrust fill is faster.
    //the doc is not realy clear
    hipMemset(hist, 0, (CELL_COUNT + 1) * sizeof(unsigned int));

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "histogram value reset failed: %d\n", (int)cudaStatus);
        exit(1598);
    }

    //compute the actual histogram (done here with atomic adds)
    DFSPH_Histogram_kernel << <numBlocks, BLOCKSIZE >> > (cell_id_sorted, hist, numParticles);

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        std::cerr << "histogram failed: " << (int)cudaStatus << std::endl;
        exit(1598);
    }

    //transformour histogram to a cumulative histogram to have  the start and end of each cell
    //note: the exlusive sum make so that each cell will contains it's start value
    // Run exclusive prefix sum
    hipcub::DeviceScan::ExclusiveSum(*d_temp_storage_cumul_hist, temp_storage_bytes_cumul_hist, hist, cell_start_end, (CELL_COUNT + 1));

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "cumulative histogram failed: %d\n", (int)cudaStatus);
        exit(1598);
    }
}



//this is the bases for all kernels based function
template<typename T>
__global__ void DFSPH_sortFromIndex_kernel(T* in, T* out, unsigned int* index, unsigned int nbElements) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= nbElements) { return; }

    out[i] = in[index[i]];
}




#include <sstream>
void cuda_sortData(SPH::UnifiedParticleSet& particleSet, unsigned int * sort_id) {
    //*
    unsigned int numParticles = particleSet.neighborsDataSet->numParticles;
    int numBlocks = (numParticles + BLOCKSIZE - 1) / BLOCKSIZE;
    unsigned int *p_id_sorted = sort_id;

    Vector3d* intermediate_buffer_v3d = particleSet.neighborsDataSet->intermediate_buffer_v3d;
    RealCuda* intermediate_buffer_real = particleSet.neighborsDataSet->intermediate_buffer_real;

    //pos
    DFSPH_sortFromIndex_kernel<Vector3d> << <numBlocks, BLOCKSIZE >> > (particleSet.pos, intermediate_buffer_v3d, p_id_sorted, numParticles);
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipMemcpy(particleSet.pos, intermediate_buffer_v3d, numParticles * sizeof(Vector3d), hipMemcpyDeviceToDevice));

    //vel
    DFSPH_sortFromIndex_kernel<Vector3d> << <numBlocks, BLOCKSIZE >> > (particleSet.vel, intermediate_buffer_v3d, p_id_sorted, numParticles);
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipMemcpy(particleSet.vel, intermediate_buffer_v3d, numParticles * sizeof(Vector3d), hipMemcpyDeviceToDevice));

    //mass
    DFSPH_sortFromIndex_kernel<RealCuda> << <numBlocks, BLOCKSIZE >> > (particleSet.mass, intermediate_buffer_real, p_id_sorted, numParticles);
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipMemcpy(particleSet.mass, intermediate_buffer_real, numParticles * sizeof(RealCuda), hipMemcpyDeviceToDevice));

    if (particleSet.velocity_impacted_by_fluid_solver) {
        //kappa
        DFSPH_sortFromIndex_kernel<RealCuda> << <numBlocks, BLOCKSIZE >> > (particleSet.kappa, intermediate_buffer_real, p_id_sorted, numParticles);
        gpuErrchk(hipDeviceSynchronize());
        gpuErrchk(hipMemcpy(particleSet.kappa, intermediate_buffer_real, numParticles * sizeof(RealCuda), hipMemcpyDeviceToDevice));

        //kappav
        DFSPH_sortFromIndex_kernel<RealCuda> << <numBlocks, BLOCKSIZE >> > (particleSet.kappaV, intermediate_buffer_real, p_id_sorted, numParticles);
        gpuErrchk(hipDeviceSynchronize());
        gpuErrchk(hipMemcpy(particleSet.kappaV, intermediate_buffer_real, numParticles * sizeof(RealCuda), hipMemcpyDeviceToDevice));
    }



    //now that everything is sorted we can set each particle index to itself
    gpuErrchk(hipMemcpy(p_id_sorted, particleSet.neighborsDataSet->p_id, numParticles * sizeof(unsigned int), hipMemcpyDeviceToDevice));

}




#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>


__global__ void generateShuffleIndex_kernel(unsigned int *shuffle_index, unsigned int nbElements, hiprandState *state) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= 1) { return; }

    for (int j = 0; j < nbElements; ++j) {
        shuffle_index[j] = j;
    }



    hiprandState localState = *state;
    for (int j = 0; j < nbElements; ++j) {
        float x = hiprand_uniform(&localState);
        x *= nbElements;
        unsigned int idx = x;
        if (x < nbElements) {
            unsigned int temp = shuffle_index[idx];
            shuffle_index[idx] = shuffle_index[i];
            shuffle_index[i] = temp;
        }
    }
    *state = localState;
}

template<class T>
__global__ void fillRandom_kernel(unsigned int *buff, unsigned int nbElements, T min, T max, hiprandState *state) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= 1) { return; }

    hiprandState localState = *state;
    for (int j = 0; j < nbElements; ++j) {
        T x= hiprand(&localState);
        x *= (max-min);
        x += min;
        buff[i] = x;
    }
    *state = localState;
}

//*
__global__ void initCurand_kernel(hiprandState *state) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= 1) { return; }

    hiprand_init(1234, 0, 0, state);
}
//*/

void cuda_shuffleData(SPH::UnifiedParticleSet& particleSet) {
    unsigned int numParticles = particleSet.numParticles;
    int numBlocks = (numParticles + BLOCKSIZE - 1) / BLOCKSIZE;

    //create a random sorting index
    static unsigned int* shuffle_index = NULL;
    static hiprandState *state;
    if (shuffle_index == NULL) {
        hipMallocManaged(&(shuffle_index), particleSet.numParticlesMax * sizeof(unsigned int));
        hipMalloc(&(state), sizeof(hiprandState));
        initCurand_kernel << <1, 1 >> > (state);

        gpuErrchk(hipDeviceSynchronize());
    }


    generateShuffleIndex_kernel << <1, 1 >> > (shuffle_index, numParticles, state);
    gpuErrchk(hipDeviceSynchronize());


    unsigned int *p_id_sorted = shuffle_index;

    Vector3d* intermediate_buffer_v3d = particleSet.neighborsDataSet->intermediate_buffer_v3d;
    RealCuda* intermediate_buffer_real = particleSet.neighborsDataSet->intermediate_buffer_real;

    //pos
    DFSPH_sortFromIndex_kernel<Vector3d> << <numBlocks, BLOCKSIZE >> > (particleSet.pos, intermediate_buffer_v3d, p_id_sorted, numParticles);
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipMemcpy(particleSet.pos, intermediate_buffer_v3d, numParticles * sizeof(Vector3d), hipMemcpyDeviceToDevice));

    //vel
    DFSPH_sortFromIndex_kernel<Vector3d> << <numBlocks, BLOCKSIZE >> > (particleSet.vel, intermediate_buffer_v3d, p_id_sorted, numParticles);
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipMemcpy(particleSet.vel, intermediate_buffer_v3d, numParticles * sizeof(Vector3d), hipMemcpyDeviceToDevice));

    //mass
    DFSPH_sortFromIndex_kernel<RealCuda> << <numBlocks, BLOCKSIZE >> > (particleSet.mass, intermediate_buffer_real, p_id_sorted, numParticles);
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipMemcpy(particleSet.mass, intermediate_buffer_real, numParticles * sizeof(RealCuda), hipMemcpyDeviceToDevice));

    if (particleSet.velocity_impacted_by_fluid_solver) {
        //kappa
        DFSPH_sortFromIndex_kernel<RealCuda> << <numBlocks, BLOCKSIZE >> > (particleSet.kappa, intermediate_buffer_real, p_id_sorted, numParticles);
        gpuErrchk(hipDeviceSynchronize());
        gpuErrchk(hipMemcpy(particleSet.kappa, intermediate_buffer_real, numParticles * sizeof(RealCuda), hipMemcpyDeviceToDevice));

        //kappav
        DFSPH_sortFromIndex_kernel<RealCuda> << <numBlocks, BLOCKSIZE >> > (particleSet.kappaV, intermediate_buffer_real, p_id_sorted, numParticles);
        gpuErrchk(hipDeviceSynchronize());
        gpuErrchk(hipMemcpy(particleSet.kappaV, intermediate_buffer_real, numParticles * sizeof(RealCuda), hipMemcpyDeviceToDevice));
    }



}


//this is the bases for all kernels based function
//I also use that kernel to reset the force

__global__ void DFSPH_updateDynamicObjectParticles_kernel(int numParticles, Vector3d* pos, Vector3d* vel, Vector3d* pos0,
                                                          Vector3d position, Vector3d velocity, Quaternion q, Vector3d angular_vel, Vector3d* F) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= numParticles) { return; }

    //reset the force
    F[i] = Vector3d(0, 0, 0);

    //update location and velocity
    pos[i] = q.rotate(pos0[i]) + position;
    vel[i] = angular_vel.cross(pos[i] - position) + velocity;

}

void update_dynamicObject_UnifiedParticleSet_cuda(SPH::UnifiedParticleSet& particle_set) {
    if (particle_set.is_dynamic_object) {
        int numBlocks = (particle_set.numParticles + BLOCKSIZE - 1) / BLOCKSIZE;


        //update the particle location and velocity
        DFSPH_updateDynamicObjectParticles_kernel << <numBlocks, BLOCKSIZE >> > (particle_set.numParticles,
                                                                                 particle_set.pos, particle_set.vel, particle_set.pos0,
                                                                                 particle_set.rigidBody_cpu->position, particle_set.rigidBody_cpu->velocity,
                                                                                 particle_set.rigidBody_cpu->q, particle_set.rigidBody_cpu->angular_vel,
                                                                                 particle_set.F);

        //also we can use that time to reset the force buffer
        //directly done in the other kernel
        //DFSPH_setVector3dBufferToZero_kernel << <numBlocks, BLOCKSIZE >> > (container.F, container.numParticles);

        hipError_t cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "update_dynamicObject_UnifiedParticleSet_cuda failed: %d\n", (int)cudaStatus);
            exit(1369);
        }
    }
}





__global__ void apply_delta_to_buffer_kernel(Vector3d* buffer, Vector3d delta, const unsigned int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= size) { return; }

    buffer[i] += delta;
}





__global__ void remove_particle_layer_kernel(SPH::UnifiedParticleSet* particleSet, Vector3d movement, Vector3d* min, Vector3d *max,
                                             RealCuda kernel_radius, Vector3i gridOffset,
                                             int* count_moved_particles, int* count_possible_particles) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= particleSet->numParticles) { return; }

    Vector3d source_id = *min;
    Vector3d target_id = *max;

    if (movement.abs() != movement) {
        source_id = *max;
        target_id = *min;
    }


    Vector3d motion_axis = (movement / movement.norm()).abs();

    //compute the source and target cell row, we only keep the component in the direction of the motion
    source_id = (source_id / kernel_radius) + gridOffset ;
    source_id.toFloor();
    source_id *= motion_axis;

    target_id = (target_id / kernel_radius) + gridOffset;
    target_id.toFloor();
    target_id *= motion_axis;

    //compute the elll row for the particle and only keep the  component in the direction of the motion
    Vector3d pos = (particleSet->pos[i] / kernel_radius) + gridOffset;
    pos.toFloor();
    pos *= motion_axis;

    //I'll tag the particles that need to be moved with 25000000
    particleSet->neighborsDataSet->cell_id[i] = 0;

    if (pos == (source_id+movement)) {
        //I'll also move the paticles away
        particleSet->pos[i].y += 2.0f;
        particleSet->neighborsDataSet->cell_id[i] = 25000000;
        atomicAdd(count_moved_particles, 1);

    }else if (pos == (target_id - movement)) {
        int id = atomicAdd(count_possible_particles, 1);
        particleSet->neighborsDataSet->p_id_sorted[id] = i;
    }else if (pos == target_id || pos == source_id) {
        //move the particles that are on the border
        particleSet->pos[i] += movement*kernel_radius;
    }

}

__global__ void adapt_inserted_particles_position_kernel(SPH::UnifiedParticleSet* particleSet, int* count_moved_particles, int* count_possible_particles, 
                                                         Vector3d mov_pos, Vector3d plane_for_remaining) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= particleSet->numParticles) { return; }

    if (particleSet->neighborsDataSet->cell_id[i] == 25000000) {
        int id = atomicAdd(count_moved_particles, 1);

        if (id < (*count_possible_particles)) {
            int ref_particle_id = particleSet->neighborsDataSet->p_id_sorted[id];
            particleSet->pos[i] = particleSet->pos[ref_particle_id] + mov_pos;
            particleSet->vel[i] = particleSet->vel[ref_particle_id];
            particleSet->kappa[i] = particleSet->kappa[ref_particle_id];
            particleSet->kappaV[i] = particleSet->kappaV[ref_particle_id];

            particleSet->neighborsDataSet->cell_id[i] = 0;
        }
        else {
            particleSet->pos[i].z+=1;// = plane_for_remaining;

        }
    }

}

__global__ void find_column_max_height_kernel(SPH::UnifiedParticleSet* particleSet, RealCuda* column_max_height) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= CELL_ROW_LENGTH*CELL_ROW_LENGTH) { return; }

    int z = i / CELL_ROW_LENGTH;
    int x = i - z*CELL_ROW_LENGTH;

    RealCuda max_height = -2;

    for (int y = CELL_ROW_LENGTH - 1; y >= 0; --y) {
        int cell_id=COMPUTE_CELL_INDEX(x, y, z);
        if (particleSet->neighborsDataSet->cell_start_end[cell_id + 1] != particleSet->neighborsDataSet->cell_start_end[cell_id]) {
            unsigned int end = particleSet->neighborsDataSet->cell_start_end[cell_id + 1];
            for (unsigned int cur_particle = particleSet->neighborsDataSet->cell_start_end[cell_id]; cur_particle < end; ++cur_particle) {
                unsigned int j = particleSet->neighborsDataSet->p_id_sorted[cur_particle];
                if (particleSet->pos[j].y > max_height) {
                    max_height = particleSet->pos[j].y;
                }
            }
            break;
        }
    }

    column_max_height[i] = max_height;

}

__global__ void translate_borderline_particles_kernel(SPH::DFSPHCData data, SPH::UnifiedParticleSet* particleSet, RealCuda* column_max_height,
                                                      int* moved_particles_min_plane, int* moved_particles_max_plane, Vector3d movement) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= particleSet->numParticles) { return; }

    RealCuda affected_distance_sq = data.particleRadius*1.5;
    affected_distance_sq *= affected_distance_sq;

    RealCuda precise_affected_distance_sq = data.particleRadius*2;
    precise_affected_distance_sq *= precise_affected_distance_sq;



    //compute tsome constants
    Vector3d min=*data.bmin;
    Vector3d max=*data.bmax;
#define max_row 10
    RealCuda p_distance = data.particleRadius * 2;
    Vector3d plane_unit = movement.abs() / movement.norm();
    bool positive_motion = plane_unit.dot(movement)>0;
    Vector3d plane_unit_perp = (Vector3d(1, 0, 1) - plane_unit);
    //I need to know the width I have
    Vector3d width = (max) - (min);
    //and only kee the component oriented perpendicular with the plane
    width = width * plane_unit_perp;
    int max_count_width = width.norm() / p_distance;
    //idk why but with that computation it's missing one particle so I'll add it
    max_count_width ++;


    //just basic one that move the particle above for testing putposes
    /*
    for (int k = 0; k < 2; ++k) {
        Vector3d plane = data.damp_planes[k];
        if ((particleSet->pos[i] * plane_unit - plane).squaredNorm() < affected_distance_sq) {
            particleSet->pos[i].y += 2.0f;
            break;
        }
    }
    return;
    //*/

    //so I know I onlyhave 2 damp planes the first one being the one near the min
    for (int k = 0; k < 2; ++k) {

        Vector3d plane = data.damp_planes[k];
        if ((particleSet->pos[i] * plane_unit - plane).squaredNorm() < affected_distance_sq) {
            //let's try to estimate the density to see if there are actual surpression
            bool distance_too_short=false;
            if (k==0){
                //we can do a simple distance check in essence

                Vector3d cur_particle_pos=particleSet->pos[i];


                Vector3i cell_pos=(particleSet->pos[i]/data.getKernelRadius()).toFloor()+data.gridOffset;
                cell_pos+=Vector3i(0,-1,0);
                //ok since I want to explore the bottom cell firts I need to move in the plane
                cell_pos-=plane_unit_perp;

                //potential offset
                Vector3d particle_offset=Vector3d(0,0,0);
//*
                if (positive_motion){
                    //for positive motion the lower plane is on the source
                    if (plane_unit.dot(cur_particle_pos) <= plane_unit.dot(data.damp_planes[0])){
                        continue;
                        cell_pos += plane_unit*1;//since the particle lower than that have already been moved in the direction once
                    }else{
                        cell_pos -= plane_unit*2;
                    }
                }else{
                    //if the motion is negative then the lower plane is the target
                    if (plane_unit.dot(cur_particle_pos) <= plane_unit.dot(data.damp_planes[0])){
                        //the cell that need to be explored are on row away from us
                        cell_pos+=plane_unit;
                    }else{
                        //we need to move the particle we are checking toward on rows in the direction of the movement
                        particle_offset=plane_unit*data.getKernelRadius()*-1;
                    }
                }
//*/

                //I only need to check if the other side of the jonction border is too close, no need to check the same side since
                //it was part of a fluid at rest
                for (int k=0;k<3;++k){//that's y
                    for (int l=0;l<3;++l){//that's the coordinate in the plane

                        Vector3i cur_cell_pos=cell_pos+plane_unit_perp*l;
                        int cur_cell_id=COMPUTE_CELL_INDEX(cur_cell_pos.x,cur_cell_pos.y+k,cur_cell_pos.z);
                        UnifiedParticleSet* body=data.fluid_data_cuda;
                        NeighborsSearchDataSet* neighborsDataSet=body->neighborsDataSet;
                        unsigned int end = neighborsDataSet->cell_start_end[cur_cell_id+1];
                        for (unsigned int cur_particle = neighborsDataSet->cell_start_end[cur_cell_id]; cur_particle < end; ++cur_particle) {
                            unsigned int j = neighborsDataSet->p_id_sorted[cur_particle];
                            if ((cur_particle_pos - (body->pos[j]+particle_offset)).squaredNorm() < precise_affected_distance_sq) {
                                distance_too_short=true;
                                break;
                            }
                        }
                    }
                    if (distance_too_short){break;}
                }

            }else{
                Vector3d cur_particle_pos=particleSet->pos[i];


                Vector3i cell_pos=(particleSet->pos[i]/data.getKernelRadius()).toFloor()+data.gridOffset;
                cell_pos+=Vector3i(0,-1,0);
                //ok since I want to explore the bottom cell firts I need to move in the plane
                cell_pos-=plane_unit_perp;

                //on the target side the cell of the right side are a copy of the left side !
                // so we have to check the row agaisnt itself
                //but we will have to translate the particles depending on the side we are on
                Vector3d particle_offset=Vector3d(0,0,0);


                if (positive_motion){
                    if (plane_unit.dot(cur_particle_pos) > plane_unit.dot(data.damp_planes[1])) {
                        //the cell that need to be explored are on row away from us
                        cell_pos-=plane_unit;
                    }else{
                        continue;
                        //we need to move the particle we are checking toward on rows in the direction of the movement
                        particle_offset=plane_unit*data.getKernelRadius();
                    }
                }else{
                    if (plane_unit.dot(cur_particle_pos) > plane_unit.dot(data.damp_planes[1])) {
                        cell_pos -= plane_unit*1;//since the particle lower than that have already been moved in the direction once
                    }else{
                        cell_pos += plane_unit*2;
                    }

                }


                //I only need to check if the other side of the jonction border is too close, no need to check the same side since
                //it was part of a fluid at rest
                for (int k=0;k<3;++k){//that's y
                    for (int l=0;l<3;++l){//that's the coordinate in the plane

                        Vector3i cur_cell_pos=cell_pos+plane_unit_perp*l;
                        int cur_cell_id=COMPUTE_CELL_INDEX(cur_cell_pos.x,cur_cell_pos.y+k,cur_cell_pos.z);
                        UnifiedParticleSet* body=data.fluid_data_cuda;
                        NeighborsSearchDataSet* neighborsDataSet=body->neighborsDataSet;
                        unsigned int end = neighborsDataSet->cell_start_end[cur_cell_id + 1];
                        for (unsigned int cur_particle = neighborsDataSet->cell_start_end[cur_cell_id]; cur_particle < end; ++cur_particle) {
                            unsigned int j = neighborsDataSet->p_id_sorted[cur_particle];
                            if ((cur_particle_pos - (body->pos[j]+particle_offset)).squaredNorm() < precise_affected_distance_sq) {
                                distance_too_short=true;
                                break;
                            }
                        }
                        if (distance_too_short){break;}
                    }
                }

            }

            if (!distance_too_short){
                //that mean this particle is not too close for another and there is no need to handle it
                continue;
            }else{
                //for testing purposes
                //particleSet->pos[i].y+=2.0f;
                //return;
            }



            bool near_min=true;
            //get a unique id to compute the position
            //int id = atomicAdd((k==0)? moved_particles_min_plane : moved_particles_max_plane, 1);
            int id = atomicAdd(moved_particles_max_plane, 1);
            /*
            if ((id%3)!=0){
                id/=3;
                near_min=false;
            }else{
                id = atomicAdd(moved_particles_min_plane, 1);
            }
            //*/

            //and compute the particle position
            int row_count = id / max_count_width;
            int level_count = row_count / max_row;

            Vector3d pos_local = Vector3d(0, 0, 0);
            pos_local.y += level_count*(p_distance*0.80);
            //the 1 or -1 at the end is because the second iter start at the max and it need to go reverse
            pos_local += (plane_unit*p_distance*(row_count - level_count*max_row) + plane_unit_perp*p_distance*(id - row_count*max_count_width))*((near_min) ? 1 : -1);
            //just a simple interleave on y
            if (level_count & 1 != 0) {
                pos_local += (Vector3d(1,0,1)*(p_distance / 2.0f))*((near_min) ? 1 : -1);
            }

            //now I need to find the first possible position
            //it depends if we are close to the min of to the max
            Vector3d pos_f = (near_min) ? min : max;

            //and for the height we need to find the column
            Vector3d pos_temp = (pos_f + pos_local);

            //now the problem is that the column id wontains the height befoore any particle movement;
            //so from the id I have here I need to know the corresponding id before any particle movement
            //the easiest way is to notivce that anything before the first plane and after the secodn plane have been moved
            //anything else is still the same
            if (near_min){
                //0 is the min plane
                if (plane_unit.dot(pos_temp) < plane_unit.dot(data.damp_planes[0])){
                    pos_temp -= (movement*data.getKernelRadius());
                }
            }else{
                //1 is the max plane
                if (plane_unit.dot(pos_temp) > plane_unit.dot(data.damp_planes[1])) {
                    pos_temp -= (movement*data.getKernelRadius());
                }
            }

            pos_temp= pos_temp / data.getKernelRadius() + data.gridOffset;
            pos_temp.toFloor();

            //read the actual height
            int column_id = pos_temp.x + pos_temp.z*CELL_ROW_LENGTH;
            pos_f.y =  column_max_height[column_id] + p_distance;


            pos_f += pos_local;


            particleSet->pos[i] = pos_f;
            particleSet->vel[i] = Vector3d(0,0,0);
            particleSet->kappa[i] = 0;
            particleSet->kappaV[i] = 0;

        }
    }
}

#define SHOW_MESSAGES_IN_CUDA_FUNCTIONS
void move_simulation_cuda(SPH::DFSPHCData& data, Vector3d movement) {
    data.damp_planes_count = 0;
    //compute the movement on the position and the axis
    Vector3d mov_pos = movement*data.getKernelRadius();
    Vector3d mov_axis = (movement.abs()) / movement.norm();

    //we store the min and max before the movement of the solid particles
    get_min_max_pos_kernel << <1, 1 >> > (data.boundaries_data->gpu_ptr, data.bmin, data.bmax, data.particleRadius);
    gpuErrchk(hipDeviceSynchronize());

#ifdef SHOW_MESSAGES_IN_CUDA_FUNCTIONS
    std::cout << "test min_max: " << data.bmin->x << " " << data.bmin->y << " " << data.bmin->z << " " << data.bmax->x << " " << data.bmax->y << " " << data.bmax->z << std::endl;
#endif
    //move the boundaries
    //we need to move the positions
    SPH::UnifiedParticleSet* particleSet = data.boundaries_data;
    {
        std::chrono::steady_clock::time_point start = std::chrono::steady_clock::now();


        unsigned int numParticles = particleSet->numParticles;
        int numBlocks = (numParticles + BLOCKSIZE - 1) / BLOCKSIZE;

        //move the particles
        apply_delta_to_buffer_kernel<< <numBlocks, BLOCKSIZE >> > (particleSet->pos, mov_pos, numParticles);
        gpuErrchk(hipDeviceSynchronize());



#ifdef SHOW_MESSAGES_IN_CUDA_FUNCTIONS
        std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
        float time = std::chrono::duration_cast<std::chrono::nanoseconds> (end - start).count() / 1000000.0f;
        std::cout << "time to move solid particles simu: " << time << " ms" << std::endl;
#endif
    }

    //and now the fluid
    particleSet = data.fluid_data;
    {
        //I'll need the information of whih cell contains which particles
        particleSet->initNeighborsSearchData(data, false);


        std::chrono::steady_clock::time_point start = std::chrono::steady_clock::now();

        //first I need the highest particle for each cell
        static RealCuda* column_max_height = NULL;
        if (column_max_height == NULL) {
            hipMallocManaged(&(column_max_height), CELL_ROW_LENGTH*CELL_ROW_LENGTH * sizeof(RealCuda));
        }
        {
            int numBlocks = (CELL_ROW_LENGTH*CELL_ROW_LENGTH + BLOCKSIZE - 1) / BLOCKSIZE;
            find_column_max_height_kernel << <numBlocks, BLOCKSIZE >> > (particleSet->gpu_ptr, column_max_height);
            gpuErrchk(hipDeviceSynchronize());
        }




        //for the fluid I don't want to "move"the fluid, I have to rmv some particles and
        //add others to change the simulation area of the fluid
        //the particles that I'll remove are the ones in the second layer when a linear index is used
        //to find the second layer just take the first particle and you add 1to the cell id on the desired direction
        unsigned int numParticles = particleSet->numParticles;
        int numBlocks = (numParticles + BLOCKSIZE - 1) / BLOCKSIZE;

        //to remove the particles the easiest way is to attribute a huge id to the particles I want to rmv and them to
        //sort the particles but that id followed by lowering the particle number
        static int* count_rmv_particles = NULL;
        static int* count_possible_particles = NULL;
        if (count_rmv_particles == NULL) {
            hipMallocManaged(&(count_rmv_particles), sizeof(int));
            hipMallocManaged(&(count_possible_particles), sizeof(int));
        }
        gpuErrchk(hipMemset(count_rmv_particles, 0, sizeof(int)));
        gpuErrchk(hipMemset(count_possible_particles, 0, sizeof(int)));

        //this flag tjhe particles that need tobe moved and store the index of the particles that are in the target row
        //also apply the movement to the border rows
        remove_particle_layer_kernel << <numBlocks, BLOCKSIZE >> > (particleSet->gpu_ptr, movement, data.bmin, data.bmax, data.getKernelRadius(),
                                                                    data.gridOffset, count_rmv_particles, count_possible_particles);
        gpuErrchk(hipDeviceSynchronize());

        std::cout << "count particle delta: (moved particles, possible particles)" << *count_rmv_particles <<"  "<< *count_possible_particles<< std::endl;
        std::chrono::steady_clock::time_point tp1 = std::chrono::steady_clock::now();

        //compute the positions of the 2 planes where there is a junction
        //the first of the two planes need to be the source one
        //calc the postion of the jonction planes
        //we updata the min max so that it now considers the new borders
        get_min_max_pos_kernel << <1, 1 >> > (data.boundaries_data->gpu_ptr, data.bmin, data.bmax, data.particleRadius);
        gpuErrchk(hipDeviceSynchronize());
#ifdef SHOW_MESSAGES_IN_CUDA_FUNCTIONS
        std::cout << "test min_max_2: " << data.bmin->x << " " << data.bmin->y << " " << data.bmin->z << " " << data.bmax->x << " " << data.bmax->y << " " << data.bmax->z << std::endl;
#endif

        //min plane
        RealCuda min_plane_precision = data.particleRadius / 1000;
        Vector3d plane = (*data.bmin)*mov_axis;
        plane /= data.getKernelRadius();
        plane.toFloor();
        plane += (movement.abs() == movement)?movement:(movement.abs()*2);
        plane *= data.getKernelRadius();
        //we need to prevent going to close to 0,0,0
        if (plane.norm() < min_plane_precision) {
            plane = mov_axis*min_plane_precision;
        }
        data.damp_planes[data.damp_planes_count++] = plane;

        //max plane
        plane = (*data.bmax)*mov_axis;
        plane /= data.getKernelRadius();
        plane.toFloor();
        plane -= (movement.abs() == movement)?movement:0;
        plane *= data.getKernelRadius();
        //we need to prevent going to close to 0,0,0
        if (plane.norm() < min_plane_precision) {
            plane = mov_axis*min_plane_precision;
        }
        data.damp_planes[data.damp_planes_count++] = plane;

        //always save the source
        if (movement.abs() == movement) {
            plane= data.damp_planes[data.damp_planes_count - 2];
        }

        //now modify the position of the particles that need to be moved in the new layers
        //if there are more particle that neeed to be moved than available positions
        //I'll put the additional particles in the junction plance on the side where particles have been removed
        gpuErrchk(hipMemset(count_rmv_particles, 0, sizeof(int)));
        adapt_inserted_particles_position_kernel << <numBlocks, BLOCKSIZE >> > (particleSet->gpu_ptr, count_rmv_particles, count_possible_particles,
                                                                                mov_pos, plane);
        gpuErrchk(hipDeviceSynchronize());


        std::chrono::steady_clock::time_point tp2 = std::chrono::steady_clock::now();





        //trigger the damping mechanism
        data.damp_borders = false;
        data.damp_borders_steps_count = 5;

        //add_border_to_damp_planes_cuda(data);


        //transate the particles that are too close to the jonction planes
        gpuErrchk(hipMemset(count_rmv_particles, 0, sizeof(int)));
        gpuErrchk(hipMemset(count_possible_particles, 0, sizeof(int)));
        data.destructor_activated = false;
        translate_borderline_particles_kernel << <numBlocks, BLOCKSIZE >> > (data, particleSet->gpu_ptr, column_max_height,
                                                                             count_rmv_particles, count_possible_particles, movement);
        gpuErrchk(hipDeviceSynchronize());
        data.destructor_activated = true;

#ifdef SHOW_MESSAGES_IN_CUDA_FUNCTIONS
        std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
        float time = std::chrono::duration_cast<std::chrono::nanoseconds> (tp1 - start).count() / 1000000.0f;
        float time_1 = std::chrono::duration_cast<std::chrono::nanoseconds> (tp2 - tp1).count() / 1000000.0f;
        float time_2 = std::chrono::duration_cast<std::chrono::nanoseconds> (end - tp2).count() / 1000000.0f;
        std::cout << "time to move fluid simu: " << time + time_1 + time_2 << " ms  (" << time << "  " << time_1 << "  " << time_2 << ")" << std::endl;
#endif



    }

    //we can now update the offset on the grid
    data.gridOffset-=movement;

    //and we need ot updatethe neighbor structure for the static particles
    //I'll take the easy way and just rerun the neighbor computation
    //there shoudl eb a faster way but it will be enougth for now
    data.boundaries_data->initNeighborsSearchData(data, false);
}

void add_border_to_damp_planes_cuda(SPH::DFSPHCData& data) {

    get_min_max_pos_kernel << <1, 1 >> > (data.boundaries_data->gpu_ptr, data.bmin, data.bmax, data.particleRadius);
    gpuErrchk(hipDeviceSynchronize());


    RealCuda min_plane_precision = data.particleRadius / 1000;
    data.damp_planes[data.damp_planes_count ++] = Vector3d((abs(data.bmin->x) > min_plane_precision) ? data.bmin->x : min_plane_precision, 0, 0);
    data.damp_planes[data.damp_planes_count ++] = Vector3d((abs(data.bmax->x) > min_plane_precision) ? data.bmax->x : min_plane_precision, 0, 0);
    data.damp_planes[data.damp_planes_count ++] = Vector3d(0, 0, (abs(data.bmin->z) > min_plane_precision) ? data.bmin->z : min_plane_precision);
    data.damp_planes[data.damp_planes_count ++] = Vector3d(0, 0, (abs(data.bmax->z) > min_plane_precision) ? data.bmax->z : min_plane_precision);
    //data.damp_planes[data.damp_planes_count ++] = Vector3d(0, (abs(data.bmin->y) > min_plane_precision) ? data.bmin->y : min_plane_precision, 0);

}

//the logic will be I'll get the 5 highest particles and then keep the median
//this mean the actual height willbbe slightly higher but it's a good tradeoff
//the problem with this method is that it can't handle realy low valumes of fluid...
///TODO find a better way ... maybe just keeping the highest is fine since I'll take the median of every columns anyway ...
__global__ void find_splashless_column_max_height_kernel(SPH::UnifiedParticleSet* particleSet, RealCuda* column_max_height) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= CELL_ROW_LENGTH*CELL_ROW_LENGTH) { return; }

    int z = i / CELL_ROW_LENGTH;
    int x = i - z*CELL_ROW_LENGTH;

    //this array store the highest heights for the column
    //later values are higher
    RealCuda max_height[5] = { -2, -2, -2, -2, -2 };
    int count_actual_values = 0;

    for (int y = CELL_ROW_LENGTH - 1; y >= 0; --y) {
        int cell_id = COMPUTE_CELL_INDEX(x, y, z);
        if (particleSet->neighborsDataSet->cell_start_end[cell_id + 1] != particleSet->neighborsDataSet->cell_start_end[cell_id]) {
            unsigned int end = particleSet->neighborsDataSet->cell_start_end[cell_id + 1];
            for (unsigned int cur_particle = particleSet->neighborsDataSet->cell_start_end[cell_id]; cur_particle < end; ++cur_particle) {
                unsigned int j = particleSet->neighborsDataSet->p_id_sorted[cur_particle];
                count_actual_values++;
                RealCuda cur_height = particleSet->pos[j].y;
                int is_superior = -1;
                //so I need to find the right cell of the max array
                //the boolean will indicate the id of the last cell for which the new height was superior
                for (int k = 0; k < 5; ++k) {
                    if (cur_height> max_height[k]) {
                        is_superior = k;
                    }
                }
                if (is_superior > -1) {
                    //Now I need to propagate the values in the array to make place for the new one
                    for (int k = 0; k < is_superior; ++k) {
                        max_height[k] = max_height[k+1];
                    }
                    max_height[is_superior] = cur_height;
                }
            }
            break;
        }
    }

    //and we keep the median value only if there are enougth particles in the column (so that the result is relatively correct)
    column_max_height[i] = (count_actual_values>4)?max_height[2]:-2;

}

__global__ void tag_particles_above_limit_hight_kernel(SPH::UnifiedParticleSet* particleSet, RealCuda target_height, int* count_flagged_particles) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= particleSet->numParticles) { return; }

    //put the particles that will be removed at the end
    if (particleSet->pos[i].y > target_height) {
        particleSet->neighborsDataSet->cell_id[i] = 30000000;
        atomicAdd(count_flagged_particles, 1);
    }
}

__global__ void place_additional_particles_right_above_kernel(SPH::DFSPHCData data, SPH::UnifiedParticleSet* particleSet, RealCuda* column_max_height,
                                                              int count_new_particles, int border_range, int* count_created_particles) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= count_new_particles) { return; }

    Vector3d min = *data.bmin;
    Vector3d max = *data.bmax;
    RealCuda p_distance = data.particleRadius * 2;
    //I need to know the width I have
    Vector3d width = (max)-(min);
    width.toAbs();
    Vector3d max_count_width = width / p_distance;
    max_count_width.toFloor();
    //idk why but with that computation it's missing one particle so I'll add it
    max_count_width+=1;



    //and compute the particle position
    int row_count = id / max_count_width.x;
    int level_count = row_count / max_count_width.z;

    Vector3d pos_local = Vector3d(0, 0, 0);
    pos_local.y += level_count*(p_distance*0.80);
    pos_local.x += (id - row_count*max_count_width.x)*p_distance;
    pos_local.z += (row_count - level_count*max_count_width.z)*p_distance;
    //just a simple interleave on y
    if (level_count & 1 != 0) {
        pos_local += Vector3d(1, 0, 1)*(p_distance / 2.0f);
    }

    //now I need to find the first possible position
    //it depends if we are close to the min of to the max
    Vector3d pos_f = min;

    //and for the height we need to find the column
    Vector3d pos_temp = (pos_f + pos_local);
    pos_temp = pos_temp / data.getKernelRadius() + data.gridOffset;
    pos_temp.toFloor();

    //now if required check if the particle is near enougth from the border
    int effective_id = 0;
    if (border_range > 0) {
        min = min / data.getKernelRadius() + data.gridOffset + border_range;
        min.toFloor();
        max = max / data.getKernelRadius() + data.gridOffset - border_range;
        max.toFloor();
        //
        if (!(pos_temp.x<min.x || pos_temp.z<min.z || pos_temp.x>max.x || pos_temp.z>max.z)) {
            return;
        }
        effective_id=atomicAdd(count_created_particles, 1);
    }
    else {
        effective_id = id;
    }


    //read the actual height
    int column_id = pos_temp.x + pos_temp.z*CELL_ROW_LENGTH;
    pos_f.y = column_max_height[column_id] + p_distance;

    pos_f += pos_local;

    int global_id = effective_id + particleSet->numParticles;
    particleSet->pos[global_id] = pos_f;
    particleSet->vel[global_id] = Vector3d(0, 0, 0);
    particleSet->kappa[global_id] = 0;
    particleSet->kappaV[global_id] = 0;
}


void control_fluid_height_cuda(SPH::DFSPHCData& data, RealCuda target_height) {
#ifdef SHOW_MESSAGES_IN_CUDA_FUNCTIONS
    std::cout << "start fluid level control" << std::endl;
#endif 

    SPH::UnifiedParticleSet* particleSet = data.fluid_data;


    //we will need the neighbors data to know where the particles are
    particleSet->initNeighborsSearchData(data, false);



    //so first i need to kow the fluid height
    //the main problem is that I don't want to consider splash particles
    //so I need a special kernel for that
    //first I need the highest particle for each cell
    static RealCuda* column_max_height = NULL;
    if (column_max_height == NULL) {
        hipMallocManaged(&(column_max_height), CELL_ROW_LENGTH*CELL_ROW_LENGTH * sizeof(RealCuda));
    }
    {
        int numBlocks = (CELL_ROW_LENGTH*CELL_ROW_LENGTH + BLOCKSIZE - 1) / BLOCKSIZE;
        //find_column_max_height_kernel << <numBlocks, BLOCKSIZE >> > (particleSet->gpu_ptr, column_max_height);
        find_splashless_column_max_height_kernel << <numBlocks, BLOCKSIZE >> > (particleSet->gpu_ptr, column_max_height);
        gpuErrchk(hipDeviceSynchronize());
    }

    //now I keep the avg of all the cells containing enought particles
    //technicaly i'd prefer the median but it would require way more computations
    //also doing it on the gpu would be better but F it for now
    RealCuda global_height = 0;
    int count_existing_columns = 0;
    for (int i = 0; i < CELL_ROW_LENGTH*CELL_ROW_LENGTH; ++i) {
        if (column_max_height[i] > 0) {
            global_height += column_max_height[i];
            count_existing_columns++;
        }
    }

    global_height /= count_existing_columns;
#ifdef SHOW_MESSAGES_IN_CUDA_FUNCTIONS
    std::cout << "global height detected: " << global_height << "  over column count " << count_existing_columns << std::endl;
#endif 

    //I'll take an error margin of 5 cm for now
    if (abs(global_height - target_height) < 0.05) {
        return;
    }

    //now we have 2 possible cases
    //either not enougth particles, or too many

    if (global_height > target_height) {
        //so we have to many particles
        //to rmv them, I'll flag the particles above the limit
        static int* tagged_particles_count = NULL;
        if (tagged_particles_count == NULL) {
            hipMallocManaged(&(tagged_particles_count),sizeof(int));
        }
        *tagged_particles_count = 0;

        unsigned int numParticles = particleSet->numParticles;
        int numBlocks = (numParticles + BLOCKSIZE - 1) / BLOCKSIZE;

        //tag the particles and count them
        tag_particles_above_limit_hight_kernel << <numBlocks, BLOCKSIZE >> > (particleSet->gpu_ptr, target_height, tagged_particles_count);
        gpuErrchk(hipDeviceSynchronize());

        //now use the same process as when creating the neighbors structure to put the particles to be removed at the end
        hipcub::DeviceRadixSort::SortPairs(particleSet->neighborsDataSet->d_temp_storage_pair_sort, particleSet->neighborsDataSet->temp_storage_bytes_pair_sort,
                                        particleSet->neighborsDataSet->cell_id, particleSet->neighborsDataSet->cell_id_sorted,
                                        particleSet->neighborsDataSet->p_id, particleSet->neighborsDataSet->p_id_sorted, particleSet->numParticles);
        gpuErrchk(hipDeviceSynchronize());
        cuda_sortData(*particleSet, particleSet->neighborsDataSet->p_id_sorted);
        gpuErrchk(hipDeviceSynchronize());

        //and now you can update the number of particles
        int new_num_particles = particleSet->numParticles - *tagged_particles_count;
        particleSet->update_active_particle_number(new_num_particles);
#ifdef SHOW_MESSAGES_IN_CUDA_FUNCTIONS
        std::cout << "new number of particles: " << particleSet->numParticles << std::endl;
#endif

    }
    else {
        //here we are missing fluid particles
        //Ahahahah... ok there is no way in hell I have a correct solution for that ...
        //but let's build smth
        //so let's supose that there are no objects near the borders of the fluid
        //and I'll add the particles there sright above the existing particles

        //so first I need to have the min max and the max height for each column (the actual one even taking the plash into consideration
        get_min_max_pos_kernel << <1, 1 >> > (data.boundaries_data->gpu_ptr, data.bmin, data.bmax, data.particleRadius);
        gpuErrchk(hipDeviceSynchronize());

        {
            int numBlocks = (CELL_ROW_LENGTH*CELL_ROW_LENGTH + BLOCKSIZE - 1) / BLOCKSIZE;
            find_column_max_height_kernel << <numBlocks, BLOCKSIZE >> > (particleSet->gpu_ptr, column_max_height);
            gpuErrchk(hipDeviceSynchronize());
        }



        //so now add particles near the border (let's say in the 2 column near the fluid border
        //untill you reach the desired liquid level there
        //note, if there are no rigid bodies in the simulation I can add the fluid particles everywhere

        //count the number of new particles
        Vector3d min = *data.bmin;
        Vector3d max = *data.bmax;
        RealCuda p_distance = data.particleRadius * 2;
        //I need to know the width I have
        Vector3d width = (max)-(min);
        Vector3d max_count_width = width / p_distance;

        //the 0.8 is because the particles will be interleaved and slightly compresses to be closer to a fluid at rest
        max_count_width.y=(target_height - global_height) / (p_distance);
        max_count_width.toFloor();
        //idk why but with that computation it's missing one particle so I'll add it
        max_count_width += 1;


        int count_new_particles = max_count_width.x*max_count_width.y*max_count_width.z;


        //check that we don't go over the maximum number of particles ...
        if ((particleSet->numParticles + count_new_particles) > particleSet->numParticlesMax) {
            count_new_particles = particleSet->numParticlesMax - particleSet->numParticles;
        }
#ifdef SHOW_MESSAGES_IN_CUDA_FUNCTIONS
        std::cout << "num particles to be added: " << count_new_particles << std::endl;
#endif


        int numBlocks= (count_new_particles + BLOCKSIZE - 1) / BLOCKSIZE;
        data.destructor_activated = false;
        int border_range = 2;

        static int* count_created_particles = NULL;
        if (count_created_particles == NULL) {
            hipMallocManaged(&(count_created_particles), sizeof(int));
        }
        *count_created_particles = 0;
        //and place the particles in the simulation
        place_additional_particles_right_above_kernel << <numBlocks, BLOCKSIZE >> > (data, particleSet->gpu_ptr, column_max_height,
                                                                                     count_new_particles, border_range, (border_range>0) ? count_created_particles : NULL);


        gpuErrchk(hipDeviceSynchronize());
        data.destructor_activated = true;

        //and now you can update the number of particles
        int new_num_particles = particleSet->numParticles + ((border_range>0)? (*count_created_particles) :count_new_particles);
        particleSet->update_active_particle_number(new_num_particles);
#ifdef SHOW_MESSAGES_IN_CUDA_FUNCTIONS
        std::cout << "new number of particles: " << particleSet->numParticles << std::endl;
#endif
    }



}


Vector3d get_simulation_center_cuda(SPH::DFSPHCData& data){
    //get the min and max
    get_min_max_pos_kernel << <1, 1 >> > (data.boundaries_data->gpu_ptr, data.bmin, data.bmax, data.particleRadius);
    gpuErrchk(hipDeviceSynchronize());

    //std::cout<<"get_simulation_center_cuda min max: "<<
    //           data.bmin->x<<"  "<<data.bmin->z<<"  "<<data.bmax->x<<"  "<<data.bmax->z<<std::endl;

    //and computethe center
    return ((*data.bmax)+(*data.bmin))/2;
}



__global__ void compute_dynamic_body_particle_mass_kernel(SPH::DFSPHCData data, SPH::UnifiedParticleSet* particleSet) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= particleSet->numParticles) { return; }

    Real delta = 0;//data.W_zero;

    RealCuda radius_sq = data.m_kernel_precomp.getRadius();
    Vector3d pos = particleSet->pos[i];
    Vector3d pos_cell = (pos / radius_sq) + data.gridOffset; //on that line the radius is not yet squared
    pos_cell.toFloor();
    int x = pos_cell.x;
    int y = pos_cell.y;
    int z = pos_cell.z;
    radius_sq *= radius_sq;


    //since this version use the std index to be able to iterate on 3 successive cells
    //I can do the -1 at the start on x.
    //one thing: it x=0 then we can only iterate 2 cells at a time
    unsigned int successive_cells_count = (x > 0) ? 3 : 2;
    x = (x > 0) ? x - 1 : x;


    const SPH::UnifiedParticleSet& body = *particleSet;
    for (int k = -1; k < 2; ++k) {
        for (int m = -1; m < 2; ++m) {
            unsigned int cur_cell_id = COMPUTE_CELL_INDEX(x, y + k, z + m);
            unsigned int end = body.neighborsDataSet->cell_start_end[cur_cell_id + successive_cells_count];
            for (unsigned int cur_particle = body.neighborsDataSet->cell_start_end[cur_cell_id]; cur_particle < end; ++cur_particle) {
                unsigned int j = body.neighborsDataSet->p_id_sorted[cur_particle];
                if ((pos - body.pos[j]).squaredNorm() < radius_sq) {
                    if (i != j) { delta += data.W(pos - body.pos[j]); }
                }
            }
        }
    }


    const Real volume = 1.0 / delta;
    particleSet->mass[i] = particleSet->density0 * volume;
}

void compute_UnifiedParticleSet_particles_mass_cuda(SPH::DFSPHCData& data, SPH::UnifiedParticleSet& container){
    int numBlocks = (container.numParticles + BLOCKSIZE - 1) / BLOCKSIZE;

    container.initNeighborsSearchData(data, false);


    data.destructor_activated = false;
    compute_dynamic_body_particle_mass_kernel << <numBlocks, BLOCKSIZE >> > (data, container.gpu_ptr);
    gpuErrchk(hipDeviceSynchronize());
    data.destructor_activated = true;
}

void cuda_neighborsSearch(SPH::DFSPHCData& data) {

    //std::chrono::steady_clock::time_point begin_global = std::chrono::steady_clock::now();
    static unsigned int time_count = 0;
    float time_global;
    static float time_avg_global = 0;
    time_count++;

    /*
    if (time_count<5) {
        cuda_shuffleData(data.fluid_data[0]);
        std::cout << "randomizing particle order" << std::endl;
    }
    //*/

    bool need_sort = true;//((time_count%15)==0);

    if (need_sort){
        std::cout<<"doing full neighbor search"<<std::endl;
    }

    bool old_fluid_aggregated=data.is_fluid_aggregated;
    hipError_t cudaStatus;
    if (true){
        if (need_sort&&data.is_fluid_aggregated){
            data.is_fluid_aggregated=false;
            data.neighborsDataSetGroupedDynamicBodies->numParticles-=data.fluid_data->numParticles;
        }

        //*
        std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();
        //*/

        //first let's generate the cell start end for the dynamic bodies
#ifdef GROUP_DYNAMIC_BODIES_NEIGHBORS_SEARCH
        cuda_initNeighborsSearchDataSetGroupedDynamicBodies(data);
#else
        for (int i = 0; i < data.numDynamicBodies; ++i) {
            SPH::UnifiedParticleSet& body = data.vector_dynamic_bodies_data[i];
            body.initNeighborsSearchData(data.m_kernel_precomp.getRadius(), false);
        }
#endif


        std::chrono::steady_clock::time_point middle = std::chrono::steady_clock::now();

        //no need to ever do it forthe boundaries since they don't ever move

        //now update the cell start end of the fluid particles
        if ((!data.is_fluid_aggregated)||data.numDynamicBodies<1){

            //since it the init iter I'll sort both even if it's the boundaries
            static int step_count = 0;
            step_count++;

            data.fluid_data->initNeighborsSearchData(data, need_sort);


            cudaStatus = hipDeviceSynchronize();
            if (cudaStatus != hipSuccess) {
                fprintf(stderr, "before neighbors search: %d\n", (int)cudaStatus);
                exit(1598);
            }


        }

        //*

        std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
        float time0;
        float time1;
        static float time_avg = 0;
        time0 = std::chrono::duration_cast<std::chrono::nanoseconds> (middle - begin).count() / 1000000.0f;
        time1 = std::chrono::duration_cast<std::chrono::nanoseconds> (end - middle).count() / 1000000.0f;

        time_avg += time0+time1;
        //printf("Time to generate cell start end: %f ms (%f,%f)   avg: %f ms \n", time0+time1,time0,time1, time_avg / time_count);

        if (time_count > 150) {
            time_avg = 0;
        }
        //*/


    }
    //and we can now do the actual search of the neaighbor for eahc fluid particle
    if (true)
    {
        //*
        float time;
        static float time_avg = 0;
        std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();
        //*/

        //cuda way
        int numBlocks = (data.fluid_data[0].numParticles + BLOCKSIZE - 1) / BLOCKSIZE;

        //*
        DFSPH_neighborsSearch_kernel << <numBlocks, BLOCKSIZE >> > (data, data.fluid_data_cuda);

        if (data.boundaries_data->has_factor_computation) {
            DFSPH_neighborsSearch_kernel << <numBlocks, BLOCKSIZE >> > (data, data.boundaries_data_cuda);
        }
        //*/
        /*
        //this test show that even just computing the neighbors for the fluid particle
        //with a basic method take more time than building the whole structure
        DFSPH_neighborsSearchBasic_kernel << <numBlocks, BLOCKSIZE >> > (data.numFluidParticles,
            data.m_kernel_precomp.getRadius(),
            data.fluid_data_cuda,
            data.boundaries_data_cuda,
            data.vector_dynamic_bodies_data_cuda, data.numDynamicBodies);
        //*/

        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            std::cerr << "cuda neighbors search failed: " << (int)cudaStatus << std::endl;
            exit(1598);
        }

        //*
        std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
        time = std::chrono::duration_cast<std::chrono::nanoseconds> (end - begin).count() / 1000000.0f;

        time_avg += time;
        //printf("Time to generate neighbors buffers: %f ms   avg: %f ms \n", time, time_avg / time_count);

        if (time_count > 150) {
            time_avg = 0;
            time_count = 0;
        }
        //*/



        /*
        {
            //a simple check to know the max nbr of neighbors
            static int absolute_max = 0;
            int max = 0;

            static int absolute_max_d[3] = { 0 };
            int max_d[3] = { 0 };



            for (int j = 0; j < data.fluid_data->getNumberOfNeighbourgs(j); j++)
            {
                //check the global value
                int count_neighbors = 0;
                for (int k = 0; k < 3; ++k) {
                    count_neighbors += data.fluid_data->getNumberOfNeighbourgs(j, k);
                }
                if (count_neighbors > max)max = count_neighbors;

                //chekc the max for each category
                for (unsigned int k = 0; k < 3; ++k) {
                    if ((int)data.fluid_data->getNumberOfNeighbourgs(j,k) > max_d[k])max_d[k] = data.fluid_data->getNumberOfNeighbourgs(j,k);
                }

            }
            if (max>absolute_max)absolute_max = max;
            for (unsigned int k = 0; k < 3; ++k) {
                if (max_d[k]>absolute_max_d[k])absolute_max_d[k] = max_d[k];
            }
            printf("max nbr of neighbors %d  (%d) \n", absolute_max, max);
            printf("max nbr of neighbors %d  (%d)      absolute max  fluid // boundaries // bodies   %d // %d // %d\n",
            absolute_max, max, absolute_max_d[0], absolute_max_d[1], absolute_max_d[2]);
        }


        //*/
    }

    //reactive the aggragation if we desactivated it because a sort was required
    if (need_sort&&old_fluid_aggregated){
        data.is_fluid_aggregated=true;
        data.neighborsDataSetGroupedDynamicBodies->numParticles+=data.fluid_data->numParticles;
    }

    /*
    std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
    time_global = std::chrono::duration_cast<std::chrono::nanoseconds> (end - begin_global).count() / 1000000.0f;

    time_avg_global += time_global;
    printf("time taken by the neighbor function: %f ms   avg: %f ms \n", time_global, time_avg_global / time_count);
    //*/
}



void cuda_initNeighborsSearchDataSet(SPH::UnifiedParticleSet& particleSet, SPH::NeighborsSearchDataSet& dataSet,
                                     SPH::DFSPHCData& data, bool sortBuffers){



    //com the id
    cuda_neighborsSearchInternal_sortParticlesId(particleSet.pos, data.getKernelRadius(), data.gridOffset, dataSet.numParticles,
                                                 &dataSet.d_temp_storage_pair_sort, dataSet.temp_storage_bytes_pair_sort, dataSet.cell_id, dataSet.cell_id_sorted,
                                                 dataSet.p_id, dataSet.p_id_sorted);



    //since it the init iter I'll sort both even if it's the boundaries
    if (sortBuffers) {
        cuda_sortData(particleSet, dataSet.p_id_sorted);
    }



    //and now I cna compute the start and end of each cell :)
    cuda_neighborsSearchInternal_computeCellStartEnd(dataSet.numParticles, dataSet.cell_id_sorted, dataSet.hist,
                                                     &dataSet.d_temp_storage_cumul_hist, dataSet.temp_storage_bytes_cumul_hist, dataSet.cell_start_end);




}


__global__ void DFSPH_fill_aggregated_pos_buffer_kernel(SPH::DFSPHCData data, unsigned int num_particles) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= num_particles) { return; }

    if (data.is_fluid_aggregated){
        if (i<data.fluid_data_cuda->numParticles){

            //writte de pos
            data.posBufferGroupedDynamicBodies[i]=data.fluid_data_cuda->pos[i];

            return;
        }
    }

    //find the current dynamic body
    int count_particles_previous_bodies=(data.is_fluid_aggregated)?data.fluid_data_cuda->numParticles:0;
    int body_id=0;
    while((count_particles_previous_bodies+data.vector_dynamic_bodies_data_cuda[body_id].numParticles)<i ){
        count_particles_previous_bodies+=data.vector_dynamic_bodies_data_cuda[body_id].numParticles;
        body_id++;
    }

    //writte de pos
    data.posBufferGroupedDynamicBodies[i]=data.vector_dynamic_bodies_data_cuda[body_id].pos[i-count_particles_previous_bodies];
}

void cuda_initNeighborsSearchDataSetGroupedDynamicBodies(SPH::DFSPHCData& data){
    if (data.numDynamicBodies<1){
        return;
    }

    SPH::NeighborsSearchDataSet& dataSet=*(data.neighborsDataSetGroupedDynamicBodies);


    // now fill itr
    int numBlocks = (dataSet.numParticles + BLOCKSIZE - 1) / BLOCKSIZE;
    DFSPH_fill_aggregated_pos_buffer_kernel<< <numBlocks, BLOCKSIZE >> > (data, dataSet.numParticles);
    gpuErrchk(hipDeviceSynchronize());

    //and now we can do the neighbor search
    //com the id
    cuda_neighborsSearchInternal_sortParticlesId(data.posBufferGroupedDynamicBodies, data.getKernelRadius(), data.gridOffset, dataSet.numParticles,
                                                 &dataSet.d_temp_storage_pair_sort, dataSet.temp_storage_bytes_pair_sort, dataSet.cell_id, dataSet.cell_id_sorted,
                                                 dataSet.p_id, dataSet.p_id_sorted);



    //and now I cna compute the start and end of each cell :)
    cuda_neighborsSearchInternal_computeCellStartEnd(dataSet.numParticles, dataSet.cell_id_sorted, dataSet.hist,
                                                     &dataSet.d_temp_storage_cumul_hist, dataSet.temp_storage_bytes_cumul_hist, dataSet.cell_start_end);



}


void cuda_renderFluid(SPH::DFSPHCData& data) {
    cuda_opengl_renderParticleSet(*data.fluid_data->renderingData, data.fluid_data[0].numParticles);
}



void cuda_renderBoundaries(SPH::DFSPHCData& data, bool renderWalls) {
    if (renderWalls) {
        cuda_opengl_renderParticleSet(*(data.boundaries_data->renderingData), data.boundaries_data->numParticles);
    }

    for (int i = 0; i < data.numDynamicBodies; ++i) {
        SPH::UnifiedParticleSet& body= data.vector_dynamic_bodies_data[i];
        cuda_opengl_renderParticleSet(*body.renderingData, body.numParticles);
    }
}

/*
THE NEXT FUNCTIONS ARE FOR THE RENDERING
*/


void cuda_opengl_initParticleRendering(ParticleSetRenderingData& renderingData, unsigned int numParticles,
                                       Vector3d** pos, Vector3d** vel) {
    glGenVertexArrays(1, &renderingData.vao); // Cr�er le VAO
    glBindVertexArray(renderingData.vao); // Lier le VAO pour l'utiliser


    glGenBuffers(1, &renderingData.pos_buffer);
    // selectionne le buffer pour l'initialiser
    glBindBuffer(GL_ARRAY_BUFFER, renderingData.pos_buffer);
    // dimensionne le buffer actif sur array_buffer, l'alloue et l'initialise avec les positions des sommets de l'objet
    glBufferData(GL_ARRAY_BUFFER,
                 /* length */	numParticles * sizeof(Vector3d),
                 /* data */      NULL,
                 /* usage */     GL_DYNAMIC_DRAW);
    //set it to the attribute
    glEnableVertexAttribArray(0);
    glVertexAttribPointer(0, 3, GL_FORMAT, GL_FALSE, 0, 0);

    glGenBuffers(1, &renderingData.vel_buffer);
    // selectionne le buffer pour l'initialiser
    glBindBuffer(GL_ARRAY_BUFFER, renderingData.vel_buffer);
    // dimensionne le buffer actif sur array_buffer, l'alloue et l'initialise avec les positions des sommets de l'objet
    glBufferData(GL_ARRAY_BUFFER,
                 /* length */	numParticles * sizeof(Vector3d),
                 /* data */      NULL,
                 /* usage */     GL_DYNAMIC_DRAW);
    //set it to the attribute
    glEnableVertexAttribArray(1);
    glVertexAttribPointer(1, 3, GL_FORMAT, GL_FALSE, 0, 0);

    // nettoyage
    glBindVertexArray(0);
    glBindBuffer(GL_ARRAY_BUFFER, 0);

    // Registration with CUDA.
    gpuErrchk(hipGraphicsGLRegisterBuffer(&renderingData.pos, renderingData.pos_buffer, hipGraphicsRegisterFlagsNone));
    gpuErrchk(hipGraphicsGLRegisterBuffer(&renderingData.vel, renderingData.vel_buffer, hipGraphicsRegisterFlagsNone));

    //link the pos and vel buffer to cuda
    gpuErrchk(hipGraphicsMapResources(1, &renderingData.pos, 0));
    gpuErrchk(hipGraphicsMapResources(1, &renderingData.vel, 0));

    //set the openglbuffer for direct use in cuda
    Vector3d* vboPtr = NULL;
    size_t size = 0;

    // pos
    gpuErrchk(hipGraphicsResourceGetMappedPointer((void**)&vboPtr, &size, renderingData.pos));//get cuda ptr
    *pos = vboPtr;

    // vel
    gpuErrchk(hipGraphicsResourceGetMappedPointer((void**)&vboPtr, &size, renderingData.vel));//get cuda ptr
    *vel = vboPtr;

}

void cuda_opengl_releaseParticleRendering(ParticleSetRenderingData& renderingData) {
    //unlink the pos and vel buffer from cuda
    gpuErrchk(hipGraphicsUnmapResources(1, &(renderingData.pos), 0));
    gpuErrchk(hipGraphicsUnmapResources(1, &(renderingData.vel), 0));

    //delete the opengl buffers
    glDeleteBuffers(1, &renderingData.vel_buffer);
    glDeleteBuffers(1, &renderingData.pos_buffer);
    glDeleteVertexArrays(1, &renderingData.vao);
}

void cuda_opengl_renderParticleSet(ParticleSetRenderingData& renderingData, unsigned int numParticles) {

    //unlink the pos and vel buffer from cuda
    gpuErrchk(hipGraphicsUnmapResources(1, &(renderingData.pos), 0));
    gpuErrchk(hipGraphicsUnmapResources(1, &(renderingData.vel), 0));

    //Actual opengl rendering
    // link the vao
    glBindVertexArray(renderingData.vao);

    glBindBuffer(GL_ARRAY_BUFFER, 0);

    //show it
    glDrawArrays(GL_POINTS, 0, numParticles);

    // unlink the vao
    glBindVertexArray(0);

    //link the pos and vel buffer to cuda
    gpuErrchk(hipGraphicsMapResources(1, &renderingData.pos, 0));
    gpuErrchk(hipGraphicsMapResources(1, &renderingData.vel, 0));

}






/*
THE NEXT FUNCTIONS ARE FOR THE MEMORY ALLOCATION
*/

void allocate_DFSPHCData_base_cuda(SPH::DFSPHCData& data) {
    if (data.damp_planes == NULL) {
        hipMallocManaged(&(data.damp_planes), sizeof(Vector3d) * 10);
    }
    if (data.bmin == NULL) {

        hipMallocManaged(&(data.bmin), sizeof(Vector3d));
        hipMallocManaged(&(data.bmax), sizeof(Vector3d));
    }
}



void allocate_UnifiedParticleSet_cuda(SPH::UnifiedParticleSet& container) {

    //hipMalloc(&(container.pos), container.numParticles * sizeof(Vector3d)); //use opengl buffer with cuda interop
    //hipMalloc(&(container.vel), container.numParticles * sizeof(Vector3d)); //use opengl buffer with cuda interop
    hipMalloc(&(container.mass), container.numParticlesMax * sizeof(RealCuda));


    if (container.has_factor_computation) {
        //*
        hipMallocManaged(&(container.numberOfNeighbourgs), container.numParticlesMax * 3 * sizeof(int));
        hipMallocManaged(&(container.neighbourgs), container.numParticlesMax * MAX_NEIGHBOURS * sizeof(int));

        hipMalloc(&(container.density), container.numParticlesMax * sizeof(RealCuda));
        hipMalloc(&(container.factor), container.numParticlesMax * sizeof(RealCuda));
        hipMalloc(&(container.densityAdv), container.numParticlesMax * sizeof(RealCuda));

        if (container.velocity_impacted_by_fluid_solver) {
            hipMalloc(&(container.acc), container.numParticlesMax * sizeof(Vector3d));
            hipMalloc(&(container.kappa), container.numParticlesMax * sizeof(RealCuda));
            hipMalloc(&(container.kappaV), container.numParticlesMax * sizeof(RealCuda));

            //I need the allocate the memory cub need to compute the reduction
            //I need the avg pointer because cub require it (but i'll clear after the cub call)
            RealCuda* avg_density_err = NULL;
            hipMalloc(&(avg_density_err), sizeof(RealCuda));

            container.d_temp_storage=NULL;
            container.temp_storage_bytes=0;
            hipcub::DeviceReduce::Sum(container.d_temp_storage, container.temp_storage_bytes,
                                   container.densityAdv, avg_density_err, container.numParticlesMax);
            // Allocate temporary storage
            hipMalloc(&(container.d_temp_storage), container.temp_storage_bytes);

            hipFree(avg_density_err);
        }
        //*/

    }

    if (container.is_dynamic_object) {
        hipMalloc(&(container.pos0), container.numParticlesMax * sizeof(Vector3d));
        hipMalloc(&(container.F), container.numParticlesMax * sizeof(Vector3d));
    }

    gpuErrchk(hipDeviceSynchronize());
}

void release_UnifiedParticleSet_cuda(SPH::UnifiedParticleSet& container) {

    //hipMalloc(&(container.pos), container.numParticles * sizeof(Vector3d)); //use opengl buffer with cuda interop
    //hipMalloc(&(container.vel), container.numParticles * sizeof(Vector3d)); //use opengl buffer with cuda interop
    hipFree(container.mass); container.mass = NULL;


    if (container.has_factor_computation) {
        //*
        hipFree(container.numberOfNeighbourgs); container.numberOfNeighbourgs = NULL;
        hipFree(container.neighbourgs); container.neighbourgs = NULL;

        hipFree(container.density); container.density = NULL;
        hipFree(container.factor); container.factor = NULL;
        hipFree(container.densityAdv); container.densityAdv = NULL;

        if (container.velocity_impacted_by_fluid_solver) {
            hipFree(container.acc); container.acc = NULL;
            hipFree(container.kappa); container.kappa = NULL;
            hipFree(container.kappaV); container.kappaV = NULL;

            hipFree(container.d_temp_storage); container.d_temp_storage = NULL;
            container.temp_storage_bytes = 0;
        }
        //*/

    }

    if (container.is_dynamic_object) {
        hipFree(container.F); container.F = NULL;
    }

}


void load_UnifiedParticleSet_cuda(SPH::UnifiedParticleSet& container, Vector3d* pos, Vector3d* vel, RealCuda* mass) {
    gpuErrchk(hipMemcpy(container.pos, pos, container.numParticles * sizeof(Vector3d), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(container.vel, vel, container.numParticles * sizeof(Vector3d), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(container.mass, mass, container.numParticles * sizeof(RealCuda), hipMemcpyHostToDevice));

    if (container.is_dynamic_object) {
        int numBlocks = (container.numParticles + BLOCKSIZE - 1) / BLOCKSIZE;
        gpuErrchk(hipMemcpy(container.pos0, pos, container.numParticles * sizeof(Vector3d), hipMemcpyHostToDevice));
        DFSPH_setVector3dBufferToZero_kernel << <numBlocks, BLOCKSIZE >> > (container.F, container.numParticles);
    }

    if (container.has_factor_computation) {

        if (container.velocity_impacted_by_fluid_solver) {
            gpuErrchk(hipMemset(container.kappa, 0, container.numParticles * sizeof(RealCuda)));
            gpuErrchk(hipMemset(container.kappaV, 0, container.numParticles * sizeof(RealCuda)));
        }
    }

}

void read_UnifiedParticleSet_cuda(SPH::UnifiedParticleSet& container, Vector3d* pos, Vector3d* vel, RealCuda* mass, Vector3d* pos0) {
    if (pos != NULL) {
        gpuErrchk(hipMemcpy(pos, container.pos, container.numParticles * sizeof(Vector3d), hipMemcpyDeviceToHost));
    }

    if (vel != NULL) {
        gpuErrchk(hipMemcpy(vel, container.vel, container.numParticles * sizeof(Vector3d), hipMemcpyDeviceToHost));
    }

    if (mass != NULL) {
        gpuErrchk(hipMemcpy(mass, container.mass,  container.numParticles * sizeof(RealCuda), hipMemcpyDeviceToHost));
    }

    if (container.is_dynamic_object&&pos0 != NULL) {
        gpuErrchk(hipMemcpy(pos0, container.pos0, container.numParticles * sizeof(Vector3d), hipMemcpyDeviceToHost));
    }
}

void read_rigid_body_force_cuda(SPH::UnifiedParticleSet& container) {
    if (container.is_dynamic_object) {
        if (container.F_cpu == NULL) {
            container.F_cpu = new Vector3d[container.numParticles];
        }

        gpuErrchk(hipMemcpy(container.F_cpu, container.F, container.numParticles * sizeof(Vector3d), hipMemcpyDeviceToHost));
    }
}


__global__ void compute_fluid_impact_on_dynamic_body_kernel(SPH::UnifiedParticleSet* container, Vector3d rb_position,
                                                            Vector3d* force, Vector3d* moment) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= container->numParticles) { return; }

    Vector3d F,M;

    F=container->F[i];
    M=(container->pos[i]-rb_position).cross(F);

    atomicAdd(&(force->x),F.x);
    atomicAdd(&(force->y),F.y);
    atomicAdd(&(force->z),F.z);
    atomicAdd(&(moment->x),M.x);
    atomicAdd(&(moment->y),M.y);
    atomicAdd(&(moment->z),M.z);
}

void compute_fluid_impact_on_dynamic_body_cuda(SPH::UnifiedParticleSet& container, Vector3d& force, Vector3d& moment){
    static Vector3d* force_cuda = NULL;
    static Vector3d* moment_cuda = NULL;
    if (force_cuda == NULL) {
        hipMallocManaged(&(force_cuda),sizeof(Vector3d));
        hipMallocManaged(&(moment_cuda),sizeof(Vector3d));
    }
    *force_cuda=Vector3d(0,0,0);
    *moment_cuda=Vector3d(0,0,0);

    int numBlocks = (container.numParticles + BLOCKSIZE - 1) / BLOCKSIZE;
    compute_fluid_impact_on_dynamic_body_kernel << <numBlocks, BLOCKSIZE >> > (container.gpu_ptr,
                                                                               container.rigidBody_cpu->position, force_cuda, moment_cuda);
    gpuErrchk(hipDeviceSynchronize());

    force=*force_cuda;
    moment=*moment_cuda;
}

__global__ void compute_fluid_boyancy_on_dynamic_body_kernel(SPH::UnifiedParticleSet* container, Vector3d* force, Vector3d* pt_appli) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= container->numParticles) { return; }



    //I use the abs just in case for some reason the vertical force is negative ...
    //By this I mena that the y component also contains the y component of the drag. but there
    //is no way to extract the actual boyancy, soand approximation will have to do
    RealCuda boyancy=container->F[i].y;
    RealCuda boyancy_abs=abs(boyancy);
    if(boyancy_abs>0){
        Vector3d pt=container->pos[i]*boyancy_abs;

        //in the x componant I'll store the total abs
        atomicAdd(&(force->x),boyancy_abs);
        atomicAdd(&(force->y),boyancy);
        atomicAdd(&(pt_appli->x),pt.x);
        atomicAdd(&(pt_appli->y),pt.y);
        atomicAdd(&(pt_appli->z),pt.z);
    }
}

void compute_fluid_Boyancy_on_dynamic_body_cuda(SPH::UnifiedParticleSet& container, Vector3d& force, Vector3d& pt_appli){
    static Vector3d* force_cuda = NULL;
    static Vector3d* pt_cuda = NULL;
    if (force_cuda == NULL) {
        hipMallocManaged(&(force_cuda),sizeof(Vector3d));
        hipMallocManaged(&(pt_cuda),sizeof(Vector3d));
    }
    *force_cuda=Vector3d(0,0,0);
    *pt_cuda=Vector3d(0,0,0);

    int numBlocks = (container.numParticles + BLOCKSIZE - 1) / BLOCKSIZE;
    compute_fluid_boyancy_on_dynamic_body_kernel << <numBlocks, BLOCKSIZE >> > (container.gpu_ptr, force_cuda, pt_cuda);
    gpuErrchk(hipDeviceSynchronize());

    force=*force_cuda;
    //if the sum of the force is non zero
    if(abs(force.y)>0){
        pt_appli=*pt_cuda;

        //now compute the avg to get the actual point
        pt_appli= pt_appli/force.x;
        //and clear the x component
        force.x=0;
    }else{
        force=Vector3d(0,0,0);
        pt_appli=Vector3d(0,0,0);
    }
}

void allocate_and_copy_UnifiedParticleSet_vector_cuda(SPH::UnifiedParticleSet** out_vector, SPH::UnifiedParticleSet* in_vector, int numSets) {

    gpuErrchk(hipMalloc(out_vector, numSets * sizeof(SPH::UnifiedParticleSet)));

    //now set the gpu_ptr in eahc object so that it points to the right place
    for (int i = 0; i < numSets; ++i) {
        in_vector[i].gpu_ptr = *out_vector + i;
    }



    //before being able to fill the gpu array we need to make a copy of the data structure since
    //we will have to change the neighborsdataset from the cpu to the gpu
    //*
    SPH::UnifiedParticleSet* temp;
    temp = new SPH::UnifiedParticleSet[numSets];
    std::copy(in_vector, in_vector + numSets, temp);

    for (int i = 0; i < numSets; ++i) {
        SPH::UnifiedParticleSet& body = temp[i];

        //we need to toggle the flag that prevent the destructor from beeing called on release
        //since it's the cpu version that clear the memory buffers that are common to the two structures
        body.releaseDataOnDestruction = false;

        //duplicate the neighbor dataset to the gpu
        bool copy_neighbor_struct=true;

#ifdef GROUP_DYNAMIC_BODIES_NEIGHBORS_SEARCH
        //copy_neighbor_struct=false;
#endif

        if (copy_neighbor_struct){
            gpuErrchk(hipMalloc(&(body.neighborsDataSet), sizeof(SPH::NeighborsSearchDataSet)));

            gpuErrchk(hipMemcpy(body.neighborsDataSet, in_vector[i].neighborsDataSet,
                                 sizeof(SPH::NeighborsSearchDataSet), hipMemcpyHostToDevice));
        }else{
            body.neighborsDataSet=NULL;
        }

    }
    //*/


    gpuErrchk(hipMemcpy(*out_vector, temp, numSets * sizeof(SPH::UnifiedParticleSet), hipMemcpyHostToDevice));


    //Now I have to update the pointer of the cpu set so that it point to the gpu structure
    delete[] temp;



}



void allocate_grouped_neighbors_struct_cuda(SPH::DFSPHCData& data){
    int numParticles=0;

    if (data.is_fluid_aggregated){
        numParticles+=data.fluid_data->numParticles;
    }

    for(int i=0;i<data.numDynamicBodies;++i){
        numParticles+= data.vector_dynamic_bodies_data[i].numParticles;
    }

    //allocate the dataset
    if (data.neighborsDataSetGroupedDynamicBodies==NULL){
        data.neighborsDataSetGroupedDynamicBodies=new SPH::NeighborsSearchDataSet(numParticles,numParticles);

        //duplicate the neighbor dataset to the gpu
        gpuErrchk(hipMalloc(&(data.neighborsDataSetGroupedDynamicBodies_cuda), sizeof(SPH::NeighborsSearchDataSet)));

        gpuErrchk(hipMemcpy(data.neighborsDataSetGroupedDynamicBodies_cuda, data.neighborsDataSetGroupedDynamicBodies,
                             sizeof(SPH::NeighborsSearchDataSet), hipMemcpyHostToDevice));
    }

    //now it's like the normal neighbor search excapt that we have to iterate on all the solid particles
    //instead of just one buffer
    //the easiest way is to build a new pos array that contains all the solid particles
    if (data.posBufferGroupedDynamicBodies==NULL){
        hipMalloc(&(data.posBufferGroupedDynamicBodies), numParticles * sizeof(Vector3d));
    }

}

void update_neighborsSearchBuffers_UnifiedParticleSet_vector_cuda(SPH::UnifiedParticleSet** out_vector, SPH::UnifiedParticleSet* in_vector, int numSets) {
    SPH::UnifiedParticleSet* temp;
    temp = new SPH::UnifiedParticleSet[numSets];

    gpuErrchk(hipMemcpy(temp, *out_vector, numSets * sizeof(SPH::UnifiedParticleSet), hipMemcpyDeviceToHost));

    for (int i = 0; i < numSets; ++i) {
        SPH::UnifiedParticleSet& body = temp[i];

        //we need to toggle the flag that prevent the destructor from beeing called on release
        //since it's the cpu version that clear the memory buffers that are common to the two structures
        body.releaseDataOnDestruction = false;

        //update the neighbor dataset to the cpu
        gpuErrchk(hipMemcpy(body.neighborsDataSet, in_vector[i].neighborsDataSet,
                             sizeof(SPH::NeighborsSearchDataSet), hipMemcpyHostToDevice));

    }

    gpuErrchk(hipMemcpy(*out_vector, temp, numSets * sizeof(SPH::UnifiedParticleSet), hipMemcpyHostToDevice));


    delete[] temp;
}



void release_UnifiedParticleSet_vector_cuda(SPH::UnifiedParticleSet** vector, int numSets) {
    //to be able to release the internal buffer I need firt to copy everything back to the cpu
    //then release the internal buffers
    //then release the UnifiedParticleSet
    //*
    SPH::UnifiedParticleSet* temp;
    temp = new SPH::UnifiedParticleSet[numSets];


    gpuErrchk(hipMemcpy(temp, *vector, numSets * sizeof(SPH::UnifiedParticleSet), hipMemcpyDeviceToHost));

    for (int i = 0; i < numSets; ++i) {
        hipFree(temp[i].neighborsDataSet); temp[i].neighborsDataSet = NULL;
    }

    hipFree(*vector); *vector = NULL;
}



void release_cudaPtr_cuda(void** ptr) {
    hipFree(*ptr); *ptr = NULL;
}


template<class T> __global__ void cuda_setBufferToValue_kernel(T* buff, T value, unsigned int buff_size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= buff_size) { return; }

    buff[i] = value;
}

__global__ void cuda_updateParticleCount_kernel(SPH::UnifiedParticleSet* container, unsigned int numParticles) {
    //that kernel wil only ever use one thread so I sould noteven need that
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= 1) { return; }

    container->numParticles = numParticles;
    container->neighborsDataSet->numParticles = numParticles;
}



void update_active_particle_number_cuda(SPH::UnifiedParticleSet& container) {
    //And now I need to update the particle count in the gpu structures
    //the easiest way is to use a kernel with just one thread used
    //the other way would be to copy the data back to the cpu then update the value before sending it back to the cpu
    cuda_updateParticleCount_kernel << <1, 1 >> > (container.gpu_ptr, container.numParticles);
}

void add_particles_cuda(SPH::UnifiedParticleSet& container, int num_additional_particles, const Vector3d* pos, const Vector3d* vel) {
    //can't use memeset for the mass so I have to make a kernel for the set
    int numBlocks = (num_additional_particles + BLOCKSIZE - 1) / BLOCKSIZE;
    cuda_setBufferToValue_kernel<RealCuda> << <numBlocks, BLOCKSIZE >> > (container.mass,
                                                                          container.m_V*container.density0, container.numParticles+num_additional_particles);



    gpuErrchk(hipMemcpy(container.pos + container.numParticles, pos, num_additional_particles * sizeof(Vector3d), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(container.vel + container.numParticles, vel, num_additional_particles * sizeof(Vector3d), hipMemcpyHostToDevice));


    gpuErrchk(hipMemset(container.kappa + container.numParticles, 0, num_additional_particles * sizeof(RealCuda)));
    gpuErrchk(hipMemset(container.kappaV + container.numParticles, 0, num_additional_particles * sizeof(RealCuda)));

    //update the particle count
    container.update_active_particle_number(container.numParticles + num_additional_particles);


    hipError_t cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        std::cerr << "add_particles_cuda failed: " << (int)cudaStatus << std::endl;
        exit(1598);
    }


}

template<class T> void set_buffer_to_value(T* buff, T val, int size) {
    //can't use memeset for the mass so I have to make a kernel for the  set
    int numBlocks = (size + BLOCKSIZE - 1) / BLOCKSIZE;
    cuda_setBufferToValue_kernel<T> << <numBlocks, BLOCKSIZE >> > (buff, val, size);

    hipError_t cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        std::cerr << "set_buffer_to_value failed: " << (int)cudaStatus << std::endl;
        exit(1598);
    }
}


void allocate_precomputed_kernel_managed(SPH::PrecomputedCubicKernelPerso& kernel, bool minimize_managed) {

    if (minimize_managed) {
        hipMalloc(&(kernel.m_W), kernel.m_resolution * sizeof(RealCuda));
        hipMalloc(&(kernel.m_gradW), (kernel.m_resolution + 1) * sizeof(RealCuda));
    }
    else {
        fprintf(stderr, "trying to use managed buffers for the kernels\n");
        exit(1256);
        //hipMallocManaged(&(kernel.m_W), kernel.m_resolution * sizeof(RealCuda));
        //hipMallocManaged(&(kernel.m_gradW), (kernel.m_resolution + 1) * sizeof(RealCuda));
    }
}


void init_precomputed_kernel_from_values(SPH::PrecomputedCubicKernelPerso& kernel, RealCuda* w, RealCuda* grad_W) {
    hipError_t cudaStatus;
    //W
    cudaStatus = hipMemcpy(kernel.m_W,
                            w,
                            kernel.m_resolution * sizeof(RealCuda),
                            hipMemcpyHostToDevice);

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "precomputed initialization of W from data failed: %d\n", (int)cudaStatus);
        exit(1598);
    }

    //grad W
    cudaStatus = hipMemcpy(kernel.m_gradW,
                            grad_W,
                            (kernel.m_resolution + 1) * sizeof(RealCuda),
                            hipMemcpyHostToDevice);

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "precomputed initialization of grad W from data failed: %d\n", (int)cudaStatus);
        exit(1598);
    }

}


void allocate_neighbors_search_data_set(SPH::NeighborsSearchDataSet& dataSet) {



    //allocatethe mme for fluid particles
    hipMallocManaged(&(dataSet.cell_id), dataSet.numParticlesMax * sizeof(unsigned int));
    hipMallocManaged(&(dataSet.cell_id_sorted), dataSet.numParticlesMax * sizeof(unsigned int));
    hipMallocManaged(&(dataSet.local_id), dataSet.numParticlesMax * sizeof(unsigned int));
    hipMallocManaged(&(dataSet.p_id), dataSet.numParticlesMax * sizeof(unsigned int));
    hipMallocManaged(&(dataSet.hist), (CELL_COUNT + 1) * sizeof(unsigned int));

    hipMallocManaged(&(dataSet.p_id_sorted), dataSet.numParticlesMax * sizeof(unsigned int));
    hipMallocManaged(&(dataSet.cell_start_end), (CELL_COUNT + 1) * sizeof(unsigned int));

    hipMalloc(&(dataSet.intermediate_buffer_v3d), dataSet.numParticlesMax * sizeof(Vector3d));
    hipMalloc(&(dataSet.intermediate_buffer_real), dataSet.numParticlesMax * sizeof(RealCuda));


    //reset the particle id
    {
        int numBlocks = (dataSet.numParticles + BLOCKSIZE - 1) / BLOCKSIZE;
        DFSPH_setBufferValueToItself_kernel << <numBlocks, BLOCKSIZE >> > (dataSet.p_id, dataSet.numParticlesMax);
        DFSPH_setBufferValueToItself_kernel << <numBlocks, BLOCKSIZE >> > (dataSet.p_id_sorted, dataSet.numParticlesMax);
    }

    hipError_t cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "allocation neighbors structure failed: %d\n", (int)cudaStatus);
        exit(1598);
    }

    //init variables for cub calls
    dataSet.temp_storage_bytes_pair_sort = 0;
    dataSet.d_temp_storage_pair_sort = NULL;
    hipcub::DeviceRadixSort::SortPairs(dataSet.d_temp_storage_pair_sort, dataSet.temp_storage_bytes_pair_sort,
                                    dataSet.cell_id, dataSet.cell_id_sorted, dataSet.p_id, dataSet.p_id_sorted, dataSet.numParticlesMax);
    gpuErrchk(hipDeviceSynchronize());
    hipMalloc(&(dataSet.d_temp_storage_pair_sort), dataSet.temp_storage_bytes_pair_sort);

    dataSet.temp_storage_bytes_cumul_hist = 0;
    dataSet.d_temp_storage_cumul_hist = NULL;
    hipcub::DeviceScan::ExclusiveSum(dataSet.d_temp_storage_cumul_hist, dataSet.temp_storage_bytes_cumul_hist,
                                  dataSet.hist, dataSet.cell_start_end, (CELL_COUNT + 1));
    gpuErrchk(hipDeviceSynchronize());
    hipMalloc(&(dataSet.d_temp_storage_cumul_hist), dataSet.temp_storage_bytes_cumul_hist);


    std::cout << "neighbors struct num byte allocated cub (numParticlesMax pair_sort cumul_hist)" << dataSet.numParticlesMax << "  " <<
                 dataSet.temp_storage_bytes_pair_sort << "  " << dataSet.temp_storage_bytes_cumul_hist << std::endl;

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "allocation neighbors structure cub part failed: %d\n", (int)cudaStatus);
        exit(1598);
    }

    dataSet.internal_buffers_allocated = true;
}


void release_neighbors_search_data_set(SPH::NeighborsSearchDataSet& dataSet, bool keep_result_buffers) {
    //allocatethe mme for fluid particles
    hipFree(dataSet.cell_id); dataSet.cell_id = NULL;
    hipFree(dataSet.local_id); dataSet.local_id = NULL;
    hipFree(dataSet.p_id); dataSet.p_id = NULL;
    hipFree(dataSet.cell_id_sorted); dataSet.cell_id_sorted = NULL;
    hipFree(dataSet.hist); dataSet.hist = NULL;

    //init variables for cub calls
    hipFree(dataSet.d_temp_storage_pair_sort); dataSet.d_temp_storage_pair_sort = NULL;
    dataSet.temp_storage_bytes_pair_sort = 0;
    hipFree(dataSet.d_temp_storage_cumul_hist); dataSet.d_temp_storage_cumul_hist = NULL;
    dataSet.temp_storage_bytes_cumul_hist = 0;


    hipFree(dataSet.intermediate_buffer_v3d); dataSet.intermediate_buffer_v3d = NULL;
    hipFree(dataSet.intermediate_buffer_real); dataSet.intermediate_buffer_real = NULL;

    dataSet.internal_buffers_allocated = false;

    if (!keep_result_buffers) {
        hipFree(dataSet.p_id_sorted); dataSet.p_id_sorted = NULL;
        hipFree(dataSet.cell_start_end); dataSet.cell_start_end = NULL;
    }
}























/*
AFTER THIS ARE ONLY THE TEST FUNCTION TO HAVE CUDA WORKING ...
*/


inline __host__ __device__ float3 make_float3(float s)
{
    return make_float3(s, s, s);
}

inline __host__ __device__ float4 make_float4(float s)
{
    return make_float4(s, s, s, s);
}

inline __host__ __device__ float4 operator*(float4& a, RealCuda b)
{
    return make_float4(a.x * b, a.y * b, a.z * b, 0);
}

inline __host__ __device__ float3 operator*(float3& a, RealCuda b)
{
    return make_float3(a.x * b, a.y * b, a.z * b);
}

inline __host__ __device__ void operator+=(float4 &a, float4 b)
{
    a.x += b.x;
    a.y += b.y;
    a.z += b.z;
}

inline __host__ __device__ void operator+=(float3 &a, float3 b)
{
    a.x += b.x;
    a.y += b.y;
    a.z += b.z;
}



template<typename T>
__global__ void test_vector_type_kernel(T* v1, T* v2, RealCuda factor, int count_elem) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= count_elem) { return; }

    v1[i]+=v2[i]*factor;
}

void compare_vector3_struct_speed(){
    RealCuda factor=0.001;
    int count_elem= 1000000;
    Vector3d* v1_v3d;
    Vector3d* v2_v3d;
    float3* v1_f3;
    float3* v2_f3;
    float4* v1_f4;
    float4* v2_f4;
    Vector3d* v1_v3d_2=new Vector3d[count_elem];
    Vector3d* v2_v3d_2=new Vector3d[count_elem];
    float3* v1_f3_2=new float3[count_elem];
    float3* v2_f3_2=new float3[count_elem];
    float4* v1_f4_2=new float4[count_elem];
    float4* v2_f4_2=new float4[count_elem];
    hipMalloc(&(v1_v3d), count_elem * sizeof(Vector3d));
    hipMalloc(&(v2_v3d), count_elem * sizeof(Vector3d));

    hipMalloc(&(v1_f3), count_elem * sizeof(float3));
    hipMalloc(&(v2_f3), count_elem * sizeof(float3));

    hipMalloc(&(v1_f4), count_elem * sizeof(float4));
    hipMalloc(&(v2_f4), count_elem * sizeof(float4));

    for (int i=0;i<count_elem;++i){
        v1_v3d_2[i]=i;
        v2_v3d_2[i]=i;
        v1_f3_2[i]=make_float3(i);
        v2_f3_2[i]=make_float3(i);
        v1_f4_2[i]=make_float4(i);
        v2_f4_2[i]=make_float4(i);
    }

    gpuErrchk(hipMemcpy(v1_v3d, v1_v3d_2,count_elem * sizeof(Vector3d),hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(v2_v3d, v2_v3d_2,count_elem * sizeof(Vector3d),hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(v1_f3, v1_f3_2,count_elem * sizeof(float3),hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(v2_f3, v2_f3_2,count_elem * sizeof(float3),hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(v1_f4, v1_f4_2,count_elem * sizeof(float4),hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(v2_f4, v2_f4_2,count_elem * sizeof(float4),hipMemcpyHostToDevice));

    int numBlocks = (count_elem + BLOCKSIZE - 1) / BLOCKSIZE;
    gpuErrchk(hipDeviceSynchronize());

    float avg0=0;
    float avg1=0;
    float avg2=0;

    int iter=10;
    for (int i=0;i<iter;++i){
        std::chrono::steady_clock::time_point t0 = std::chrono::steady_clock::now();
        test_vector_type_kernel<float3> << <numBlocks, BLOCKSIZE >> > (v1_f3, v2_f3, factor, count_elem);
        gpuErrchk(hipDeviceSynchronize());

        std::chrono::steady_clock::time_point t1 = std::chrono::steady_clock::now();
        test_vector_type_kernel<Vector3d> << <numBlocks, BLOCKSIZE >> > (v1_v3d, v2_v3d, factor, count_elem);
        gpuErrchk(hipDeviceSynchronize());

        std::chrono::steady_clock::time_point t2 = std::chrono::steady_clock::now();
        test_vector_type_kernel<float4> << <numBlocks, BLOCKSIZE >> > (v1_f4, v2_f4, factor, count_elem);
        gpuErrchk(hipDeviceSynchronize());

        std::chrono::steady_clock::time_point t3 = std::chrono::steady_clock::now();

        float time_0 = std::chrono::duration_cast<std::chrono::nanoseconds> (t1 - t0).count() / 1000000.0f;
        float time_1 = std::chrono::duration_cast<std::chrono::nanoseconds> (t2 - t1).count() / 1000000.0f;
        float time_2 = std::chrono::duration_cast<std::chrono::nanoseconds> (t3 - t2).count() / 1000000.0f;

        printf("comparison between vector data struct  (float3, Vector3d, float4): %f   %f   %f\n", time_0, time_1, time_2);

        if (iter>0){
            avg0+=time_0;
            avg1+=time_1;
            avg2+=time_2;
        }
    }
    iter--;

    printf("comparison between vector data struct Global (float3, Vector3d, float4): %f   %f   %f\n",
           avg0/iter, avg1/iter, avg2/iter);

}







hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}
//*
__global__ void addKernel(Vector3d* vect)
{
    int i = threadIdx.x;
    vect[i].z = vect[i].x + vect[i].y;
}

__global__ void setVectkernel(Vector3d& vect)
{
    vect.x = 5;
    vect.y = 6;
    vect.z = 7;
}
//*/
int test_cuda()
{
    //DFSPHCData* data;
    std::cout << "start cuda test basic" << std::endl;

    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };
    //*
    Vector3d* vect;
    hipMallocManaged(&vect, arraySize * sizeof(Vector3d));
    for (int i = 0; i < arraySize; ++i) {
        vect[i].x = a[i];
        vect[i].y = b[i];
    }
    //*/*

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }


    printf("macro val: %d, %d, %d\n", __CUDACC_VER_MAJOR__, __CUDACC_VER_MINOR__, __CUDACC_VER_BUILD__);

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
           c[0], c[1], c[2], c[3], c[4]);

    for (int i = 0; i < arraySize; ++i) {
        c[i] = 0;
    }


    // Launch a kernel on the GPU with one thread for each element.
    addKernel << <1, arraySize >> > (vect);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    printf("with vects {1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
           (int)(vect[0].z), (int)(vect[1].z), (int)(vect[2].z), (int)(vect[3].z), (int)(vect[4].z));

    hipFree(vect);



    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    /*
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipDeviceReset failed!");
    return 1;
    }
    //*/

    printf("Finished test cuda\n");


    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel << <1, size >> > (dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus;
}
